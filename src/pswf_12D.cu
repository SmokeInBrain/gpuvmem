#include "hip/hip_runtime.h"
#include "pswf_12D.cuh"

__host__ void PSWF_12D::buildKernel(float amp, float x0, float y0, float sigma_x, float sigma_y)
{
        this->setKernelMemory();
        float x, y;
        float val;
        for(int i=0; i<this->m; i++) {
                for(int j=0; j<this->n; j++) {
                        y = (i-this->support_y)*sigma_y;
                        x = (j-this->support_x)*sigma_x;
                        val = pswf_12D(amp, x, y, x0, y0, sigma_x, sigma_y, this->w1);
                        this->kernel[this->n * i + j] = val;
                }
        }
        this->copyKerneltoGPU();

};

__host__ float PSWF_12D::GCF(float amp, float x, float y, float x0, float y0, float sigma_x, float sigma_y, float w, float alpha)
{
        float val = pswf_12D(amp, x, y, x0, y0, sigma_x, sigma_y, w);
        return 1.0f/val;
}


namespace {
CKernel* CreateCKernel()
{
        return new PSWF_12D;
}

const std::string name = "PSWF";
const bool RegisteredPSWF = registerCreationFunction<CKernel, std::string>(name, CreateCKernel);
};
