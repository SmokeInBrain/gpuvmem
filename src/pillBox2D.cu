#include "hip/hip_runtime.h"
#include "pillBox2D.cuh"

__host__ __device__ void PillBox2D::constructKernel(float amp, float x0, float y0, float sigma_x, float sigma_y)
{
        float limit_x = this->M * sigma_x / 2;
        float limit_y = this->N * sigma_y / 2;

        float x, y;
        for(int i=0; i<this->M; i++) {
                for(int j=0; j<this->N; j++) {
                        y = (i-this->support_y)*sigma_y;
                        x = (j-this->support_x)*sigma_x;
                        this->kernel[this->N*i+j] = pillBox2D(amp, x, y, limit_x, limit_y);
                }
        }
};

namespace {
CKernel* CreateCKernel()
{
        return new PillBox2D;
}
const int CKERNELID = 0;
const bool RegisteredCKernel = Singleton<CKernelFactory>::Instance().RegisterCKernel(CKERNELID, CreateCKernel);
};
