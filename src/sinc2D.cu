#include "hip/hip_runtime.h"
#include "sinc2D.cuh"

__host__ __device__ void Sinc2D::constructKernel(float amp, float x0, float y0, float sigma_x, float sigma_y)
{
        float x, y;
        for(int i=0; i<this->m; i++) {
                for(int j=0; j<this->n; j++) {
                        y = (i-this->support_y)*sigma_y;
                        x = (j-this->support_x)*sigma_x;
                        this->kernel[this->n*i+j] = sinc2D(amp, x, y, x0, y0, sigma_x, sigma_y, this->w1);
                }
        }

};

__host__ __device__ float Sinc2D::GCF_fn(float amp, float nu, float w)
{
        if(fabs(nu) < w)
                return amp*1.0f;
        else
                return 0.0f;
};

__device__ float Sinc2D::constructGCF(float amp, float x0, float y0, float sigma_x, float sigma_y, float w, int M, int N)
{
        const int i = threadIdx.y + blockDim.y * blockIdx.y;
        const int j = threadIdx.x + blockDim.x * blockIdx.x;

        float x = (j - x0) * sigma_x;
        float y = (i - y0) * sigma_y;

        float radius = distance(x, 0, y, 0);
        float val = GCF_fn(amp, radius, w);

        return val;

};

namespace {
CKernel* CreateCKernel()
{
        return new Sinc2D;
}
const int CKERNELID = 3;
const bool RegisteredCKernel = Singleton<CKernelFactory>::Instance().RegisterCKernel(CKERNELID, CreateCKernel);
};
