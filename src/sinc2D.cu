#include "hip/hip_runtime.h"
#include "sinc2D.cuh"

__host__ __device__ void Sinc2D::constructKernel(float amp, float x0, float y0, float sigma_x, float sigma_y)
{
        float x, y;
        for(int i=0; i<this->m; i++) {
                for(int j=0; j<this->n; j++) {
                        y = (i-this->support_y)*sigma_y;
                        x = (j-this->support_x)*sigma_x;
                        this->kernel[this->n*i+j] = sinc2D(amp, x, y, x0, y0, sigma_x, sigma_y, this->w1);
                }
        }

};

namespace {
CKernel* CreateCKernel()
{
        return new Sinc2D;
}
const int CKERNELID = 3;
const bool RegisteredCKernel = Singleton<CKernelFactory>::Instance().RegisterCKernel(CKERNELID, CreateCKernel);
};
