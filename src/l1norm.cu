#include "l1norm.cuh"

extern long M, N;
extern int image_count;
extern float * penalizators;
extern int nPenalizators;

L1norm::L1norm(){
};

float L1norm::calcFi(float *p)
{
        float result = 0.0;
        this->set_fivalue(L1Norm(p, device_S, penalization_factor, mod, order, imageIndex));
        result = (penalization_factor)*( this->get_fivalue() );
        return result;
}
void L1norm::calcGi(float *p, float *xi)
{
        DL1Norm(p, device_DS, penalization_factor, mod, order, imageIndex);
};


void L1norm::restartDGi()
{
        checkCudaErrors(hipMemset(device_DS, 0, sizeof(float)*M*N));
};

void L1norm::addToDphi(float *device_dphi)
{
        linkAddToDPhi(device_dphi, device_DS, imageToAdd);
};

void L1norm::configure(int penalizatorIndex, int imageIndex, int imageToAdd)
{
        this->imageIndex = imageIndex;
        this->order = order;
        this->mod = mod;
        this->imageToAdd = imageToAdd;

        if(imageIndex > image_count -1 || imageToAdd > image_count -1)
        {
                printf("There is no image for the provided index (L1 Norm)\n");
                exit(-1);
        }

        if(penalizatorIndex != -1)
        {
                if(penalizatorIndex > (nPenalizators - 1) || penalizatorIndex < 0)
                {
                        printf("invalid index for penalizator (L1 Norm)\n");
                        exit(-1);
                }else{
                        this->penalization_factor = penalizators[penalizatorIndex];
                }
        }

        checkCudaErrors(hipMalloc((void**)&device_S, sizeof(float)*M*N));
        checkCudaErrors(hipMemset(device_S, 0, sizeof(float)*M*N));

        checkCudaErrors(hipMalloc((void**)&device_DS, sizeof(float)*M*N));
        checkCudaErrors(hipMemset(device_DS, 0, sizeof(float)*M*N));

};

void L1norm::setSandDs(float *S, float *Ds)
{
        hipFree(this->device_S);
        hipFree(this->device_DS);
        this->device_S = S;
        this->device_DS = Ds;
};

namespace {
Fi* CreateL1norm()
{
        return new L1norm;
}
const int L1normId = 6;
const bool RegisteredL1norm = Singleton<FiFactory>::Instance().RegisterFi(L1normId, CreateL1norm);
};
