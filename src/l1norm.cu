#include "l1norm.cuh"

extern long M, N;
extern int image_count;
extern float * penalizators;
extern int nPenalizators;

L1norm::L1norm(){
        this->name = "L1 Norm";
};

L1norm::L1norm(float epsilon){
        this->name = "L1 Norm";
        this->epsilon = epsilon;
};

float L1norm::getEpsilon(){
        return this->epsilon;
};

float L1norm::calcFi(float *p)
{
        float result = 0.0;
        this->set_fivalue(L1Norm(p, device_S, penalization_factor, this->epsilon, mod, order, imageIndex));
        result = (penalization_factor)*( this->get_fivalue() );
        return result;
}
void L1norm::calcGi(float *p, float *xi)
{
        DL1Norm(p, device_DS, penalization_factor, this->epsilon, mod, order, imageIndex);
};

void L1norm::restartDGi()
{
        checkCudaErrors(hipMemset(device_DS, 0, sizeof(float)*M*N));
};

void L1norm::addToDphi(float *device_dphi)
{
        linkAddToDPhi(device_dphi, device_DS, imageToAdd);
};

void L1norm::setSandDs(float *S, float *Ds)
{
        hipFree(this->device_S);
        hipFree(this->device_DS);
        this->device_S = S;
        this->device_DS = Ds;
};

void L1norm::setEpsilon(float epsilon)
{
        this->epsilon = epsilon;
};

namespace {
Fi* CreateL1norm()
{
        return new L1norm;
}
const std::string name = "L1-Norm";
const bool RegisteredL1norm = registerCreationFunction<Fi, std::string>(name, CreateL1norm);
};
