/* -------------------------------------------------------------------------
   Copyright (C) 2016-2017  Miguel Carcamo, Pablo Roman, Simon Casassus,
   Victor Moral, Fernando Rannou - miguel.carcamo@usach.cl

   This program includes Numerical Recipes (NR) based routines whose
   copyright is held by the NR authors. If NR routines are included,
   you are required to comply with the licensing set forth there.

   Part of the program also relies on an an ANSI C library for multi-stream
   random number generation from the related Prentice-Hall textbook
   Discrete-Event Simulation: A First Course by Steve Park and Larry Leemis,
   for more information please contact leemis@math.wm.edu

   Additionally, this program uses some NVIDIA routines whose copyright is held
   by NVIDIA end user license agreement (EULA).

   For the original parts of this code, the following license applies:

   This program is free software: you can redistribute it and/or modify
   it under the terms of the GNU General Public License as published by
   the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.

   This program is distributed in the hope that it will be useful,
   but WITHOUT ANY WARRANTY; without even the implied warranty of
   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU General Public License for more details.

   You should have received a copy of the GNU General Public License
   along with this program. If not, see <http://www.gnu.org/licenses/>.
 * -------------------------------------------------------------------------
 */

#include "frprmn.cuh"
#include "directioncosines.cuh"
#include "pillBox2D.cuh"
#include "ellipticalGaussian2D.cuh"
#include "gaussianSinc2D.cuh"
#include "gaussian2D.cuh"
#include "sinc2D.cuh"
#include "pswf_02D.cuh"
#include "pswf_12D.cuh"
#include <time.h>

int num_gpus;

inline bool IsAppBuiltAs64()
{
  #if defined(__x86_64) || defined(AMD64) || defined(_M_AMD64)
        return 1;
  #else
        return 0;
  #endif
}

void optimizationOrder(Optimizator *optimizator, Image *image){
        optimizator->setImage(image);
        optimizator->setFlag(0);
        optimizator->optimize();
        /*optimizator->setFlag(1);
        optimizator->optimize();
        optimizator->setFlag(2);
        optimizator->optimize();
        optimizator->setFlag(3);
        optimizator->optimize();*/
}

__host__ int main(int argc, char **argv) {
        ////CHECK FOR AVAILABLE GPUs
        hipGetDeviceCount(&num_gpus);

        printf("gpuvmem Copyright (C) 2016-2020  Miguel Carcamo, Pablo Roman, Simon Casassus, Victor Moral, Fernando Rannou, Nicolás Muñoz - miguel.carcamo@protonmail.com\n");
        printf("This program comes with ABSOLUTELY NO WARRANTY; for details use option -w\n");
        printf("This is free software, and you are welcome to redistribute it under certain conditions; use option -c for details.\n\n\n");


        if(num_gpus < 1) {
                printf("No CUDA capable devices were detected\n");
                return 1;
        }

        if (!IsAppBuiltAs64()) {
                printf("%s is only supported with on 64-bit OSs and the application must be built as a 64-bit target. Test is being waived.\n", argv[0]);
                exit(EXIT_SUCCESS);
        }

        //// AVAILABLE CLASSES
        enum {MFS}; // Synthesizer
        enum {Chi2, Entropy, Laplacian, QuadraticPenalization, TotalVariation, TotalSquaredVariation, L1Norm}; // Fi
        enum {Gridding}; // Filter
        enum {CG, LBFGS}; // Optimizator
        enum {DefaultObjectiveFunction}; // ObjectiveFunction
        enum {MS}; // Io
        enum {SecondDerivative}; // Error calculation
        enum {pillbox2D, ellipticalGaussian2D, gaussian2D, sinc2D, gaussianSinc2D, pswf_02D, pswf_12D}; // CKernels for gridding

        Synthesizer * sy = Singleton<SynthesizerFactory>::Instance().CreateSynthesizer(MFS);
        Optimizator * cg = Singleton<OptimizatorFactory>::Instance().CreateOptimizator(CG);
        // Choose your antialiasing kernel!
        CKernel * sc = new PillBox2D(1,1);
        //CKernel * sc = new GaussianSinc2D(7, 7);
        //CKernel * sc = new PSWF_12D(3,3);
        //sc->setW1(2.50f);
        //CKernel * sc = Singleton<CKernelFactory>::Instance().CreateCKernel(gaussianSinc2D);
        ObjectiveFunction *of = Singleton<ObjectiveFunctionFactory>::Instance().CreateObjectiveFunction(DefaultObjectiveFunction);
        Io *ioms = Singleton<IoFactory>::Instance().CreateIo(MS); // This is the default Io Class
        sy->setIoHandler(ioms);
        sy->setOrder(&optimizationOrder);
        sy->setGriddingKernel(sc);
        sy->configure(argc, argv);
        cg->setObjectiveFunction(of);
        sy->setOptimizator(cg);

        //Filter *g = Singleton<FilterFactory>::Instance().CreateFilter(Gridding);
        //sy->applyFilter(g); // delete this line for no gridding

        sy->setDevice(); // This routine sends the data to GPU memory
        Fi *chi2 = Singleton<FiFactory>::Instance().CreateFi(Chi2);
        Fi *e = Singleton<FiFactory>::Instance().CreateFi(Entropy);
        Fi *l = Singleton<FiFactory>::Instance().CreateFi(Laplacian);
        chi2->configure(-1, 0, 0); // (penalizatorIndex, ImageIndex, imageToaddDphi)
        e->configure(0, 0, 0);
        l->configure(1, 0, 0);
        //e->setPenalizationFactor(0.01); // If not used -Z (Fi.configure(-1,x,x))
        of->addFi(chi2);
        of->addFi(e);
        of->addFi(l);
        //sy->getImage()->getFunctionMapping()[i].evaluateXt = particularEvaluateXt;
        //sy->getImage()->getFunctionMapping()[i].newP = particularNewP;
        //if the nopositivity flag is on  all images will run with no posivity,
        //otherwise the first image image will be calculated with positivity and all the others without positivity,
        //to modify this, use these sentences, where i corresponds to the index of the image ( particularly, means positivity)
        sy->run();
        std::vector<float> fi_values = of->get_fi_values();
        //for(int it = 0; it < fi_values.size() ; it++)
        //{
        //        std::cout << fi_values[it] << std::endl;
        //}

        //e->setPenalizationFactor(0.00001);
        //l->setPenalizationFactor(0.00005);
        //std::cout << "Trying to run another iteration" << std::endl;
        //sy->run();
        sy->unSetDevice(); // This routine performs memory cleanup and release

        return 0;
}
