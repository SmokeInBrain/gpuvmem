
#include <hip/hip_runtime.h>
template <class T>
__host__ __device__ T EllipticalGaussian2D(T amp, T x, T y, T x0, T y0, T sigma_x, T sigma_y, T angle)
{
        T x_i = x-x0;
        T y_i = y-y0;
        T cos_angle, sin_angle;
        sincos(angle, &sin_angle, &cos_angle);
        T sin_angle_2 = sin(2.0*angle);
        T a = (cos_angle*cos_angle)/(2.0*sigma_x*sigma_x) + (sin_angle*sin_angle)/(2.0*sigma_y*sigma_y);
        T b = sin_angle_2/(2.0*sigma_x*sigma_x) - sin_angle_2/(2.0*sigma_y*sigma_y);
        T c = (sin_angle*sin_angle)/(2.0*sigma_x*sigma_x) + (cos_angle*cos_angle)/(2.0*sigma_y*sigma_y);
        T G = amp*exp(-a*x_i*x_i - b*x_i*y_i - c*y_i*y_i);

        return G;
}

template <class T>
__host__ __device__ T Gaussian2D(T amp, T x, T y, T x0, T y0, T sigma_x, T sigma_y, T w, T alpha)
{
        T x_i = x-x0;
        T y_i = y-y0;

        T num_x = pow(x_i, alpha);
        T num_y = pow(y_i, alpha);

        T den_x = 2.0*pow(w*sigma_x,alpha);
        T den_y = 2.0*pow(w*sigma_y,alpha);

        T val_x = num_x/den_x;
        T val_y = num_y/den_y;
        T G = amp*exp(-val_x-val_y);

        return G;
}

template <class T>
__host__ __device__ T Gaussian1D(T amp, T x, T x0, T sigma, T w, T alpha)
{
        T x_i = x-x0;
        T val = abs(x_i)/(w*sigma);
        T val_alpha = pow(val, alpha);
        T G = amp*exp(-val_alpha);

        return G;
}

template <class T>
__host__ __device__ T Sinc1D(T amp, T x, T x0, T sigma, T w)
{
  T s = amp*sinc((x-x0)/(w*sigma));
  return s;
}

template <class T>
__host__ __device__ T GaussianSinc1D(T amp, T x, T x0, T sigma, T w1, T w2, T alpha)
{
  return amp*Gaussian1D(1.0, x, x0, sigma, w1, alpha)*Sinc1D(1.0, x, x0, sigma, w2);
}


template <class T>
__host__ __device__ T Sinc2D(T amp, T x, T x0, T y, T y0, T sigma_x, T sigma_y, T w)
{
  T s_x = Sinc1D(1.0, x, x0, sigma_x, w);
  T s_y = Sinc1D(1.0, y, y0, sigma_y, w);
  return amp*s_x*s_y;
}

template <class T>
__host__ __device__ T GaussianSinc2D(T amp, T x, T y, T x0, T y0, T sigma_x, T sigma_y, T w1, T w2, T alpha)
{
  T G = Gaussian2D(1.0, x, y, x0, y0, sigma_x, sigma_y, w1, alpha);
  T S = Sinc2D(1.0, x, x0, y, y0, sigma_x, sigma_y, w2);
  return amp*G*S;
}
