#include "laplacian.cuh"

extern long M, N;
extern int image_count;
extern float * penalizators;
extern int nPenalizators;

Laplacian::Laplacian(){
};

float Laplacian::calcFi(float *p)
{
        float result = 0.0;
        result = (penalization_factor)*( laplacian(p, device_S, penalization_factor, mod, order, imageIndex) );
        return result;
}
void Laplacian::calcGi(float *p, float *xi)
{
        DLaplacian(p, device_DS, penalization_factor, mod, order, imageIndex);
};


void Laplacian::restartDGi()
{
        checkCudaErrors(hipMemset(device_DS, 0, sizeof(float)*M*N));
};

void Laplacian::addToDphi(float *device_dphi)
{
        linkAddToDPhi(device_dphi, device_DS, imageToAdd);
};

void Laplacian::configure(int penalizatorIndex, int imageIndex, int imageToAdd)
{
        this->imageIndex = imageIndex;
        this->order = order;
        this->mod = mod;
        this->imageToAdd = imageToAdd;

        if(imageIndex > image_count -1 || imageToAdd > image_count -1)
        {
                printf("There is no image for the provided index (Laplacian)\n");
                exit(-1);
        }

        if(penalizatorIndex != -1)
        {
                if(penalizatorIndex > (nPenalizators - 1) || penalizatorIndex < 0)
                {
                        printf("invalid index for penalizator (laplacian)\n");
                        exit(-1);
                }else{
                        this->penalization_factor = penalizators[penalizatorIndex];
                }
        }

        checkCudaErrors(hipMalloc((void**)&device_S, sizeof(float)*M*N));
        checkCudaErrors(hipMemset(device_S, 0, sizeof(float)*M*N));

        checkCudaErrors(hipMalloc((void**)&device_DS, sizeof(float)*M*N));
        checkCudaErrors(hipMemset(device_DS, 0, sizeof(float)*M*N));

};

void Laplacian::setSandDs(float *S, float *Ds)
{
        hipFree(this->device_S);
        hipFree(this->device_DS);
        this->device_S = S;
        this->device_DS = Ds;
};

namespace {
Fi* CreateLaplacian()
{
        return new Laplacian;
}
const int LaplacianId = 2;
const bool RegisteredLaplacian = Singleton<FiFactory>::Instance().RegisterFi(LaplacianId, CreateLaplacian);
};
