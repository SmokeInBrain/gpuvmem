#include "totalvariation.cuh"

extern long M, N;
extern int image_count;
extern float * penalizators;
extern int nPenalizators;

TVariation::TVariation(){
};

float TVariation::calcFi(float *p)
{
        float result = 0.0;
        result = (penalization_factor)*( totalvariation(p, device_S, penalization_factor, mod, order, imageIndex) );
        return result;
}
void TVariation::calcGi(float *p, float *xi)
{
        DTVariation(p, device_DS, penalization_factor, mod, order, imageIndex);
};


void TVariation::restartDGi()
{
        gpuErrchk(hipMemset(device_DS, 0, sizeof(float)*M*N));
};

void TVariation::addToDphi(float *device_dphi)
{
        linkAddToDPhi(device_dphi, device_DS, 0);
};

void TVariation::configure(int penalizatorIndex, int imageIndex, int imageToAdd)
{
        this->imageIndex = imageIndex;
        this->order = order;
        this->mod = mod;
        this->imageToAdd = imageToAdd;

        if(imageIndex > image_count -1 || imageToAdd > image_count -1)
        {
                printf("There is no image for the provided index (TVariation)\n");
                exit(-1);
        }

        if(penalizatorIndex != -1)
        {
                if(penalizatorIndex > (nPenalizators - 1) || penalizatorIndex < 0)
                {
                        printf("invalid index for penalizator (TVariation)\n");
                        exit(-1);
                }else{
                        this->penalization_factor = penalizators[penalizatorIndex];
                }
        }

        gpuErrchk(hipMalloc((void**)&device_S, sizeof(float)*M*N));
        gpuErrchk(hipMemset(device_S, 0, sizeof(float)*M*N));

        gpuErrchk(hipMalloc((void**)&device_DS, sizeof(float)*M*N));
        gpuErrchk(hipMemset(device_DS, 0, sizeof(float)*M*N));
};

void TVariation::setSandDs(float *S, float *Ds)
{
        hipFree(this->device_S);
        hipFree(this->device_DS);
        this->device_S = S;
        this->device_DS = Ds;
};

namespace {
Fi* CreateTVariation()
{
        return new TVariation;
}
const int TVariationId = 4;
const bool RegisteredTVariation = Singleton<FiFactory>::Instance().RegisterFi(TVariationId, CreateTVariation);
};
