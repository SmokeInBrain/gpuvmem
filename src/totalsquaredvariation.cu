#include "totalsquaredvariation.cuh"

extern long M, N;
extern int image_count;
extern float * penalizators;
extern int nPenalizators;

TotalSquaredVariationP::TotalSquaredVariationP(){
};

float TotalSquaredVariationP::calcFi(float *p)
{
        float result = 0.0;
        result = (penalization_factor)*(TotalSquaredVariation(p, device_S, penalization_factor, mod, order, imageIndex) );
        return result;
}
void TotalSquaredVariationP::calcGi(float *p, float *xi)
{
        DTSVariation(p, device_DS, penalization_factor, mod, order, imageIndex);
};


void TotalSquaredVariationP::restartDGi()
{
        gpuErrchk(hipMemset(device_DS, 0, sizeof(float)*M*N));
};

void TotalSquaredVariationP::addToDphi(float *device_dphi)
{
        linkAddToDPhi(device_dphi, device_DS, imageToAdd);
};

void TotalSquaredVariationP::configure(int penalizatorIndex, int imageIndex, int imageToAdd)
{
        this->imageIndex = imageIndex;
        this->order = order;
        this->mod = mod;
        this->imageToAdd = imageToAdd;

        if(imageIndex > image_count -1 || imageToAdd > image_count -1)
        {
                printf("There is no image for the provided index (SquaredTotalVariationP)\n");
                exit(-1);
        }

        if(penalizatorIndex != -1)
        {
                if(penalizatorIndex > (nPenalizators - 1) || penalizatorIndex < 0)
                {
                        printf("invalid index for penalizator (SquaredTotalVariationP)\n");
                        exit(-1);
                }else{
                        this->penalization_factor = penalizators[penalizatorIndex];
                }
        }

        gpuErrchk(hipMalloc((void**)&device_S, sizeof(float)*M*N));
        gpuErrchk(hipMemset(device_S, 0, sizeof(float)*M*N));

        gpuErrchk(hipMalloc((void**)&device_DS, sizeof(float)*M*N));
        gpuErrchk(hipMemset(device_DS, 0, sizeof(float)*M*N));

};

void TotalSquaredVariationP::setSandDs(float *S, float *Ds)
{
        hipFree(this->device_S);
        hipFree(this->device_DS);
        this->device_S = S;
        this->device_DS = Ds;
};

namespace {
Fi* CreateTotalSquaredVariationP()
{
        return new TotalSquaredVariationP;
}
const int TotalSquaredVariationPId = 5;
const bool RegisteredTotalSquaredVariationP = Singleton<FiFactory>::Instance().RegisterFi(TotalSquaredVariationPId, CreateTotalSquaredVariationP);
};
