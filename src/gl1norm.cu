#include "gl1norm.cuh"

extern long M, N;
extern int image_count;
extern float * penalizators;
extern int nPenalizators;

GL1Norm::GL1Norm(){
        this->name = "G L1-Norm";
        this->prior = NULL;
        this->normalization_factor = 1.0f;
};

GL1Norm::GL1Norm(std::vector<float> prior){
        this->name = "G L1-Norm";
        this->normalization_factor = 1.0f;
        checkCudaErrors(hipMalloc((void**)&this->prior, sizeof(float)*M*N));
        checkCudaErrors(hipMemcpy(this->prior, prior.data(), M*N, hipMemcpyHostToDevice));
};

GL1Norm::GL1Norm(float *prior, float normalization_factor){
        this->name = "G L1-Norm";
        this->prior = prior;
        this->normalization_factor = normalization_factor;
        this->normalizePrior();
};

GL1Norm::GL1Norm(std::vector<float> prior, float normalization_factor){
        this->name = "G L1-Norm";
        this->normalization_factor = normalization_factor;
        checkCudaErrors(hipMalloc((void**)&this->prior, sizeof(float)*M*N));
        checkCudaErrors(hipMemcpy(this->prior, prior.data(), M*N, hipMemcpyHostToDevice));
        this->normalizePrior();
};

GL1Norm::GL1Norm(float *prior){
        this->name = "G L1-Norm";
        this->prior = prior;
        this->normalization_factor = 1.0f;
};

GL1Norm::~GL1Norm() {
        hipFree(this->prior);
};

float GL1Norm::getNormalizationFactor(){
        return this->normalization_factor;
};

void GL1Norm::setNormalizationFactor(float normalization_factor){
        this->normalization_factor = normalization_factor;
};

void GL1Norm::setPrior(float *prior){
        hipFree(this->prior);
        this->prior = prior;
};

float GL1Norm::calcFi(float *p)
{
        float result = 0.0;
        this->set_fivalue(SGEntropy(p, device_S, this->prior, penalization_factor, mod, order, imageIndex));
        result = (penalization_factor)*( this->get_fivalue() );
        return result;
};

void GL1Norm::calcGi(float *p, float *xi)
{
        DGEntropy(p, device_DS, this->prior, penalization_factor, mod, order, imageIndex);
};


void GL1Norm::restartDGi()
{
        checkCudaErrors(hipMemset(device_DS, 0, sizeof(float)*M*N));
};

void GL1Norm::addToDphi(float *device_dphi)
{
        linkAddToDPhi(device_dphi, device_DS, imageToAdd);
};

void GL1Norm::setSandDs(float *S, float *Ds)
{
        hipFree(this->device_S);
        hipFree(this->device_DS);
        this->device_S = S;
        this->device_DS = Ds;
};

void GL1Norm::normalizePrior()
{
        normalizeImage(this->prior, this->normalization_factor);
};

namespace {
Fi* CreateGL1Norm()
{
        return new GL1Norm;
}
const std::string name = "GL1Norm";
const bool RegisteredGL1Norm = registerCreationFunction<Fi, std::string>(name, CreateGL1Norm);
const bool RegisteredGL1NormInt = registerCreationFunction<Fi, int>(0, CreateGL1Norm);
};
