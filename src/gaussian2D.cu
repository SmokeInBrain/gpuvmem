#include "hip/hip_runtime.h"
#include "gaussian2D.cuh"

__host__ __device__ void Gaussian2D::constructKernel(float amp, float x0, float y0, float sigma_x, float sigma_y)
{

        float x, y;
        for(int i=0; i<this->M; i++) {
                for(int j=0; j<this->N; j++) {
                        y = (i-this->support_y)*sigma_y;
                        x = (j-this->support_x)*sigma_x;
                        this->kernel[this->N*i+j] = gaussian2D(amp, x, y, x0, y0, sigma_x, sigma_y, this->w1, this->alpha);
                }
        }
};


namespace {
CKernel* CreateCKernel()
{
        return new Gaussian2D;
}
const int CKERNELID = 2;
const bool RegisteredCKernel = Singleton<CKernelFactory>::Instance().RegisterCKernel(CKERNELID, CreateCKernel);
};
