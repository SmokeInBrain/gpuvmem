#include "hip/hip_runtime.h"
#include "gaussian2D.cuh"

__host__ __device__ float Gaussian2D::run(float amp, float x, float y, float x0, float y0, float sigma_x, float sigma_y)
{
        float value = gaussian2D(amp, x, y, x0, y0, sigma_x, sigma_y, this->w1, this->alpha);
        return value;
};

namespace {
CKernel* CreateCKernel()
{
        return new Gaussian2D;
}
const int CKERNELID = 2;
const bool RegisteredCKernel = Singleton<CKernelFactory>::Instance().RegisterCKernel(CKERNELID, CreateCKernel);
};
