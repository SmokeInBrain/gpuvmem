#include "hip/hip_runtime.h"
#include "synthesizer.cuh"
#include "imageProcessor.cuh"


long M, N, numVisibilities;
int iter=0;

float *device_Image, *device_dphi, *device_dchi2_total, *device_dS, *device_S, beam_noise, beam_bmaj, *device_noise_image, *device_weight_image;
float beam_bmin, b_noise_aux, noise_cut, MINPIX, minpix, lambda, ftol, random_probability = 1.0;
float noise_jypix, fg_scale, final_chi2, final_S, eta, robust_param;
float *host_I, sum_weights, *initial_values, *penalizators;
Telescope *telescope;

dim3 threadsPerBlockNN;
dim3 numBlocksNN;

int threadsVectorReduceNN, blocksVectorReduceNN, nopositivity = 0, verbose_flag = 0, clip_flag = 0, apply_noise = 0, print_images = 0, save_model = 0;
int gridding, it_maximum, status_mod_in;
int multigpu, firstgpu, selected, reg_term, total_visibilities, image_count, nPenalizators, print_errors, nMeasurementSets=0, max_number_vis;
char *output, *mempath, *out_image, *msinput, *msoutput, *inputdat, *modinput;
char *t_telescope;
float nu_0, threshold;
extern int num_gpus;

double ra, dec, crpix1, crpix2, DELTAX, DELTAY, deltau, deltav;

fitsfile *mod_in;

std::vector<MSDataset> datasets;

varsPerGPU *vars_gpu;

Vars variables;

clock_t t;
double start, end;

float noise_min = 1E32;

inline bool IsGPUCapableP2P(hipDeviceProp_t *pProp)
{
  #ifdef _WIN32
        return (bool)(pProp->tccDriver ? true : false);
  #else
        return (bool)(pProp->major >= 2);
  #endif
}

std::vector<std::string> MFS::countAndSeparateStrings(char *input)
{
        char *pt;
        std::vector<std::string> ret;

        int counter = 0;
        pt = strtok(input, ",");
        while(pt!=NULL) {
                std::string s(pt);
                ret.push_back(s);
                pt = strtok (NULL, ",");
        }

        free(pt);
        return ret;
}

void MFS::configure(int argc, char **argv)
{
        if(iohandler == NULL)
        {
                iohandler = Singleton<IoFactory>::Instance().CreateIo(0);
        }

        variables = getOptions(argc, argv);
        msinput = variables.input;
        msoutput = variables.output;
        inputdat = variables.inputdat;
        modinput = variables.modin;
        out_image = variables.output_image;
        selected = variables.select;
        mempath = variables.path;
        it_maximum = variables.it_max;
        total_visibilities = 0;
        b_noise_aux = variables.noise;
        noise_cut = variables.noise_cut;
        random_probability = variables.randoms;
        eta = variables.eta;
        gridding = variables.gridding;
        nu_0 = variables.nu_0;
        robust_param = variables.robust_param;
        threshold = variables.threshold * 5.0;
        std::vector<std::string> string_values;
        std::vector<std::string> s_output_values;
        int n_outputs;


        if(strcmp(msinput, "NULL")!=0) {
                string_values = countAndSeparateStrings(msinput);
                nMeasurementSets = string_values.size();
        }else{
                printf("Datasets files were not provided\n");
                print_help();
                exit(-1);
        }

        if(strcmp(msoutput, "NULL")!=0) {
                s_output_values = countAndSeparateStrings(msoutput);
                n_outputs = s_output_values.size();
        }else{
                printf("Output/s was/were not provided\n");
                print_help();
                exit(-1);
        }

        if(n_outputs != nMeasurementSets) {
                printf("Number of input datasets should be equal to the number of output datasets\n");
                exit(-1);
        }

        if(verbose_flag)
                printf("Number of input datasets %d\n", nMeasurementSets);

        for(int i=0; i< nMeasurementSets; i++) {
                datasets.push_back(MSDataset());
                datasets[i].name = (char*)malloc((string_values[i].length()+1)*sizeof(char));
                datasets[i].oname = (char*)malloc((s_output_values[i].length()+1)*sizeof(char));
                strcpy(datasets[i].name, string_values[i].c_str());
                strcpy(datasets[i].oname, s_output_values[i].c_str());

        }

        string_values.clear();
        s_output_values.clear();

        if(strcmp(variables.initial_values, "NULL")!=0) {
                string_values = countAndSeparateStrings(variables.initial_values);
                image_count = string_values.size();
        }else{
                printf("Initial values for image/s were not provided\n");
                print_help();
                exit(-1);
        }

        if(image_count == 1)
                initial_values = (float*)malloc(sizeof(float)*image_count+1);
        else
                initial_values = (float*)malloc(sizeof(float)*image_count);

        for(int i=0; i< image_count; i++)
                initial_values[i] = atof(string_values[i].c_str());

        string_values.clear();
        if(image_count == 1)
        {
                initial_values[1] = 0.0f;
                image_count++;
                nu_0 = 1.0f;
                imagesChanged = 1;
        }

        if(image_count > 1 && nu_0 == -1)
        {
                print_help();
                printf("for 2 or more images, nu_0 (-F) is mandatory\n");
                exit(-1);
        }



        /*
         *
         * Create directory to save images for each iterations
         */
        struct stat st = {0};
        if(print_images)
                if(stat(mempath, &st) == -1) mkdir(mempath,0700);

        hipDeviceProp_t dprop[num_gpus];

        if(verbose_flag) {
                printf("Number of host CPUs:\t%d\n", omp_get_num_procs());
                printf("Number of CUDA devices:\t%d\n", num_gpus);


                for(int i = 0; i < num_gpus; i++) {
                        hipGetDeviceProperties(&dprop[i], i);

                        printf("> GPU%d = \"%15s\" %s capable of Peer-to-Peer (P2P)\n", i, dprop[i].name, (IsGPUCapableP2P(&dprop[i]) ? "IS " : "NOT"));

                        //printf("   %d: %s\n", i, dprop.name);
                }
                printf("---------------------------\n");
        }

        hipGetDeviceProperties(&dprop[0], 0);
        if(variables.blockSizeX*variables.blockSizeY > dprop[0].maxThreadsPerBlock || variables.blockSizeV > dprop[0].maxThreadsPerBlock) {
                printf("Block size X: %d\n", variables.blockSizeX);
                printf("Block size Y: %d\n", variables.blockSizeY);
                printf("Block size X*Y: %d\n", variables.blockSizeX*variables.blockSizeY);
                printf("Block size V: %d\n", variables.blockSizeV);
                printf("ERROR. The maximum threads per block cannot be greater than %d\n", dprop[0].maxThreadsPerBlock);
                exit(-1);
        }

        if(variables.blockSizeX > dprop[0].maxThreadsDim[0] || variables.blockSizeY > dprop[0].maxThreadsDim[1] || variables.blockSizeV > dprop[0].maxThreadsDim[0]) {
                printf("Block size X: %d\n", variables.blockSizeX);
                printf("Block size Y: %d\n", variables.blockSizeY);
                printf("Block size V: %d\n", variables.blockSizeV);
                printf("ERROR. The size of the blocksize cannot exceed X: %d Y: %d Z: %d\n", dprop[0].maxThreadsDim[0], dprop[0].maxThreadsDim[1], dprop[0].maxThreadsDim[2]);
                exit(-1);
        }

        if(selected > num_gpus || selected < 0) {
                printf("ERROR. THE SELECTED GPU DOESN'T EXIST\n");
                exit(-1);
        }

        readInputDat(inputdat);

        int n_telescopes;
        if(strcmp(t_telescope, "NULL")!=0) {
                string_values = countAndSeparateStrings(t_telescope);
                n_telescopes = string_values.size();
        }else{
                printf("Telescope codes were not provided\n");
                print_help();
                exit(-1);
        }

        if(n_telescopes != nMeasurementSets) {
                printf("Number of telescope codes cannot be different to the number of Measurement Sets\n");
                print_help();
                exit(-1);
        }

        int telescope;
        for(int i=0; i< nMeasurementSets; i++) {

                telescope = atoi(string_values[i].c_str());
                init_beam(telescope, &datasets[i].antenna_diameter, &datasets[i].pb_factor, &datasets[i].pb_cutoff);
        }


        canvasVariables canvas_vars = iohandler->IoreadCanvas(modinput, mod_in, b_noise_aux, status_mod_in, verbose_flag);

        M = canvas_vars.M;
        N = canvas_vars.N;
        DELTAX = canvas_vars.DELTAX;
        DELTAY = canvas_vars.DELTAY;
        ra = canvas_vars.ra;
        dec = canvas_vars.dec;
        crpix1 = canvas_vars.crpix1;
        crpix2 = canvas_vars.crpix2;
        beam_bmaj = canvas_vars.beam_bmaj;
        beam_bmin = canvas_vars.beam_bmin;
        beam_noise = canvas_vars.beam_noise;

        if(verbose_flag)
                printf("Counting data for memory allocation\n");


        for(int d=0; d<nMeasurementSets; d++) {
                if(apply_noise) {
                        iohandler->IoreadMS(datasets[d].name, datasets[d].fields, &datasets[d].data, true, false, random_probability, gridding);
                }else{
                        iohandler->IoreadMS(datasets[d].name, datasets[d].fields, &datasets[d].data, false, false, random_probability, gridding);
                }
        }

        if(verbose_flag) {
                for(int i=0; i<nMeasurementSets; i++) {
                        printf("Dataset %s\n", datasets[i].name);
                        printf("\tNumber of fields = %d\n", datasets[i].data.nfields);
                        printf("\tNumber of frequencies = %d\n", datasets[i].data.total_frequencies);
                        printf("\tNumber of Stokes = %d\n", datasets[i].data.nstokes);
                }
        }

        multigpu = 0;
        firstgpu = 0;
        if(strcmp(variables.multigpu, "NULL")!=0) {
                string_values = countAndSeparateStrings(variables.multigpu);
                multigpu = string_values.size();
                firstgpu = atoi(string_values[0].c_str());
        }
        string_values.clear();

        if(strcmp(variables.penalization_factors, "NULL")!=0) {

                string_values = countAndSeparateStrings(variables.penalization_factors);
                nPenalizators = string_values.size();
                penalizators = (float*)malloc(sizeof(float)*nPenalizators);
                for(int i = 0; i < nPenalizators; i++)
                        penalizators[i] = atof(string_values[i].c_str());

        }else{
                printf("No regularization factors provided\n");
        }
        string_values.clear();

        int max_nfreq = 1;
        if(multigpu < 0 || multigpu > num_gpus) {
                printf("ERROR. NUMBER OF GPUS CANNOT BE NEGATIVE OR GREATER THAN THE NUMBER OF GPUS\n");
                exit(-1);
        }else{
                if(multigpu == 0) {
                        num_gpus = 1;
                        firstgpu = selected;
                }else{
                        for(int d=0; d<nMeasurementSets; d++) {
                                if(datasets[d].data.total_frequencies > max_nfreq)
                                        max_nfreq = datasets[d].data.total_frequencies;
                        }

                        if(max_nfreq == 1) {
                                printf("ONLY ONE FREQUENCY. CHANGING NUMBER OF GPUS TO 1\n");
                                num_gpus = 1;
                        }else{
                                num_gpus = multigpu;
                                omp_set_num_threads(num_gpus);
                        }
                }
        }

        //printf("number of FINAL host CPUs:\t%d\n", omp_get_num_procs());
        if(verbose_flag) {
                printf("Number of CUDA devices and threads: \t%d\n", num_gpus);
        }

        //Check peer access if there is more than 1 GPU
        if(num_gpus > 1) {
                for(int i=firstgpu + 1; i< firstgpu + num_gpus; i++) {
                        hipDeviceProp_t dprop0, dpropX;
                        hipGetDeviceProperties(&dprop0, firstgpu);
                        hipGetDeviceProperties(&dpropX, i);
                        int canAccessPeer0_x, canAccessPeerx_0;
                        hipDeviceCanAccessPeer(&canAccessPeer0_x, firstgpu, i);
                        hipDeviceCanAccessPeer(&canAccessPeerx_0, i, firstgpu);
                        if(verbose_flag) {
                                printf("> Peer-to-Peer (P2P) access from %s (GPU%d) -> %s (GPU%d) : %s\n", dprop0.name, firstgpu, dpropX.name, i, canAccessPeer0_x ? "Yes" : "No");
                                printf("> Peer-to-Peer (P2P) access from %s (GPU%d) -> %s (GPU%d) : %s\n", dpropX.name, i, dprop0.name, firstgpu, canAccessPeerx_0 ? "Yes" : "No");
                        }
                        if(canAccessPeer0_x == 0 || canAccessPeerx_0 == 0) {
                                printf("Two or more SM 2.0 class GPUs are required for %s to run.\n", argv[0]);
                                printf("Support for UVA requires a GPU with SM 2.0 capabilities.\n");
                                printf("Peer to Peer access is not available between GPU%d <-> GPU%d, waiving test.\n", 0, i);
                                exit(EXIT_SUCCESS);
                        }else{
                                hipSetDevice(firstgpu);
                                if(verbose_flag) {
                                        printf("Granting access from %d to %d...\n",firstgpu, i);
                                }
                                hipDeviceEnablePeerAccess(i,0);
                                hipSetDevice(i);
                                if(verbose_flag) {
                                        printf("Granting access from %d to %d...\n", i, firstgpu);
                                }
                                hipDeviceEnablePeerAccess(firstgpu,0);
                                if(verbose_flag) {
                                        printf("Checking GPU %d and GPU %d for UVA capabilities...\n", firstgpu, i);
                                }
                                const bool has_uva = (dprop0.unifiedAddressing && dpropX.unifiedAddressing);
                                if(verbose_flag) {
                                        printf("> %s (GPU%d) supports UVA: %s\n", dprop0.name, firstgpu, (dprop0.unifiedAddressing ? "Yes" : "No"));
                                        printf("> %s (GPU%d) supports UVA: %s\n", dpropX.name, i, (dpropX.unifiedAddressing ? "Yes" : "No"));
                                }
                                if (has_uva) {
                                        if(verbose_flag) {
                                                printf("Both GPUs can support UVA, enabling...\n");
                                        }
                                }
                                else{
                                        printf("At least one of the two GPUs does NOT support UVA, waiving test.\n");
                                        exit(EXIT_SUCCESS);
                                }
                        }
                }

        }

        vars_gpu = (varsPerGPU*)malloc(num_gpus*sizeof(varsPerGPU));


        // REMEMBER TO INITIALIZE GRIDDED VISIBILITIES VECTORS AS M*N ZEROS!
        hipfftComplex cufft_zeroval;
        cufft_zeroval.x = 0.0f;
        cufft_zeroval.x = 0.0f;
        double3 zeros;
        zeros.x = 0.0;
        zeros.y = 0.0;
        zeros.z = 0.0;
        if(gridding) {
                for(int d=0; d<nMeasurementSets; d++) {
                        for(int f=0; f<datasets[d].data.nfields; f++) {
                                for(int i=0; i<datasets[d].data.total_frequencies; i++) {
                                        for(int s=0; s<datasets[d].data.nstokes; s++) {
                                                datasets[d].fields[f].gridded_visibilities[i][s].uvw.resize(M*N, zeros);
                                                datasets[d].fields[f].gridded_visibilities[i][s].weight.resize(M*N, 0.0f);
                                                datasets[d].fields[f].gridded_visibilities[i][s].S.resize(M*N, 0);
                                                datasets[d].fields[f].gridded_visibilities[i][s].Vo.resize(M*N, cufft_zeroval);
                                                datasets[d].fields[f].gridded_visibilities[i][s].Vm.resize(M*N, cufft_zeroval);
                                        }
                                }
                        }
                }
        }



        if(verbose_flag) {
                printf("Reading visibilities and FITS input files...\n");
        }

        this->visibilities = new Visibilities();
        this->visibilities->setMSDataset(datasets);
        this->visibilities->setNDatasets(nMeasurementSets);

        double deltax = RPDEG_D*DELTAX; //radians
        double deltay = RPDEG_D*DELTAY; //radians
        deltau = 1.0 / (M * deltax);
        deltav = 1.0 / (N * deltay);

        if(gridding) {
                printf("Doing gridding\n");
                omp_set_num_threads(gridding);
                for(int d=0; d<nMeasurementSets; d++)
                        do_gridding(datasets[d].fields, &datasets[d].data, deltau, deltav, M, N, robust_param);

                omp_set_num_threads(num_gpus);
        }
}

void MFS::setDevice()
{
        double deltax = RPDEG_D*DELTAX; //radians
        double deltay = RPDEG_D*DELTAY; //radians
        deltau = 1.0 / (M * deltax);
        deltav = 1.0 / (N * deltay);

        if(verbose_flag) {
                printf("MS File Successfully Read\n");
                if(beam_noise == -1) {
                        printf("Beam noise wasn't provided by the user... Calculating...\n");
                }
        }

        for(int d=0; d<nMeasurementSets; d++) {
                sum_weights = calculateNoise(datasets[d].fields, datasets[d].data, &total_visibilities, variables.blockSizeV, gridding);
        }

        this->visibilities->setTotalVisibilities(total_visibilities);

        for(int d=0; d<nMeasurementSets; d++) {
                for(int f=0; f<datasets[d].data.nfields; f++) {
                        if(num_gpus == 1) {
                                hipSetDevice(selected);
                                for(int i=0; i<datasets[d].data.total_frequencies; i++) {
                                        for(int s=0; s<datasets[d].data.nstokes; s++) {
                                                gpuErrchk(hipMalloc(&datasets[d].fields[f].device_visibilities[i][s].uvw,
                                                                     sizeof(double3) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]));
                                                gpuErrchk(hipMalloc(&datasets[d].fields[f].device_visibilities[i][s].Vo,
                                                                     sizeof(hipfftComplex) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]));
                                                gpuErrchk(hipMalloc(&datasets[d].fields[f].device_visibilities[i][s].weight,
                                                                     sizeof(float) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]));
                                                gpuErrchk(hipMalloc(&datasets[d].fields[f].device_visibilities[i][s].Vm,
                                                                     sizeof(hipfftComplex) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]));
                                                gpuErrchk(hipMalloc(&datasets[d].fields[f].device_visibilities[i][s].Vr,
                                                                     sizeof(hipfftComplex) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]));
                                                gpuErrchk(hipMemcpy(datasets[d].fields[f].device_visibilities[i][s].uvw, datasets[d].fields[f].visibilities[i][s].uvw.data(),
                                                                     sizeof(double3) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s],
                                                                     hipMemcpyHostToDevice));

                                                gpuErrchk(hipMemcpy(datasets[d].fields[f].device_visibilities[i][s].weight,
                                                                     datasets[d].fields[f].visibilities[i][s].weight.data(),
                                                                     sizeof(float) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s],
                                                                     hipMemcpyHostToDevice));

                                                gpuErrchk(hipMemcpy(datasets[d].fields[f].device_visibilities[i][s].Vo, datasets[d].fields[f].visibilities[i][s].Vo.data(),
                                                                     sizeof(hipfftComplex) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s],
                                                                     hipMemcpyHostToDevice));

                                                gpuErrchk(hipMemset(datasets[d].fields[f].device_visibilities[i][s].Vr, 0,
                                                                     sizeof(hipfftComplex) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]));
                                                gpuErrchk(hipMemset(datasets[d].fields[f].device_visibilities[i][s].Vm, 0,
                                                                     sizeof(hipfftComplex) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]));
                                        }
                                }

                                gpuErrchk(hipMalloc((void**)&datasets[d].fields[f].atten_image, sizeof(float)*M*N));
                                gpuErrchk(hipMemset(datasets[d].fields[f].atten_image, 0, sizeof(float)*M*N));

                        }else{
                                hipSetDevice(firstgpu);
                                gpuErrchk(hipMalloc((void**)&datasets[d].fields[f].atten_image, sizeof(float)*M*N));
                                gpuErrchk(hipMemset(datasets[d].fields[f].atten_image, 0, sizeof(float)*M*N));
                                for(int i=0; i<datasets[d].data.total_frequencies; i++) {
                                        hipSetDevice((i % num_gpus) + firstgpu);
                                        for(int s=0; s<datasets[d].data.nstokes; s++) {
                                                gpuErrchk(hipMalloc(&datasets[d].fields[f].device_visibilities[i][s].uvw,
                                                                     sizeof(double3) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]));
                                                gpuErrchk(hipMalloc(&datasets[d].fields[f].device_visibilities[i][s].Vo,
                                                                     sizeof(hipfftComplex) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]));
                                                gpuErrchk(hipMalloc(&datasets[d].fields[f].device_visibilities[i][s].weight,
                                                                     sizeof(float) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]));
                                                gpuErrchk(hipMalloc(&datasets[d].fields[f].device_visibilities[i][s].Vm,
                                                                     sizeof(hipfftComplex) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]));
                                                gpuErrchk(hipMalloc(&datasets[d].fields[f].device_visibilities[i][s].Vr,
                                                                     sizeof(hipfftComplex) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]));
                                                gpuErrchk(hipMemcpy(datasets[d].fields[f].device_visibilities[i][s].uvw,datasets[d].fields[f].visibilities[i][s].uvw.data(),
                                                                     sizeof(double3) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s],hipMemcpyHostToDevice));
                                                gpuErrchk(hipMemcpy(datasets[d].fields[f].device_visibilities[i][s].weight,datasets[d].fields[f].visibilities[i][s].weight.data(),
                                                                     sizeof(float) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s],hipMemcpyHostToDevice));
                                                gpuErrchk(hipMemcpy(datasets[d].fields[f].device_visibilities[i][s].Vo,datasets[d].fields[f].visibilities[i][s].Vo.data(),
                                                                     sizeof(hipfftComplex) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s],hipMemcpyHostToDevice));
                                                gpuErrchk(hipMemset(datasets[d].fields[f].device_visibilities[i][s].Vr, 0, sizeof(hipfftComplex) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]));
                                                gpuErrchk(hipMemset(datasets[d].fields[f].device_visibilities[i][s].Vm, 0,sizeof(hipfftComplex) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]));
                                        }
                                }
                        }
                }
        }


        max_number_vis = 0;
        for(int d=0; d<nMeasurementSets; d++) {
                if(datasets[d].data.max_number_visibilities_in_channel_and_stokes > max_number_vis)
                        max_number_vis = datasets[d].data.max_number_visibilities_in_channel_and_stokes;
        }

        if(max_number_vis == 0) {
                printf("Max number of visibilities cannot be zero for image synthesis\n");
                exit(-1);
        }

        this->visibilities->setMaxNumberVis(max_number_vis);

        for(int g=0; g<num_gpus; g++) {
                hipSetDevice((g%num_gpus) + firstgpu);
                gpuErrchk(hipMalloc((void**)&vars_gpu[g].device_dchi2, sizeof(float)*M*N));
                gpuErrchk(hipMemset(vars_gpu[g].device_dchi2, 0, sizeof(float)*M*N));

                gpuErrchk(hipMalloc(&vars_gpu[g].device_chi2, sizeof(float)*max_number_vis));
                gpuErrchk(hipMemset(vars_gpu[g].device_chi2, 0, sizeof(float)*max_number_vis));
        }

        //Declaring block size and number of blocks for Image
        dim3 threads(variables.blockSizeX, variables.blockSizeY);
        dim3 blocks(M/threads.x, N/threads.y);
        threadsPerBlockNN = threads;
        numBlocksNN = blocks;

        noise_jypix = beam_noise / (PI * beam_bmaj * beam_bmin / (4 * log(2) ));

        /////////////////////////////////////////////////////CALCULATE DIRECTION COSINES/////////////////////////////////////////////////
        double raimage = ra * RPDEG_D;
        double decimage = dec * RPDEG_D;

        if(verbose_flag) {
                printf("FITS: Ra: %.16e (rad), dec: %.16e (rad)\n", raimage, decimage);
                printf("FITS: Center pix: (%lf,%lf)\n", crpix1-1, crpix2-1);
        }

        double lobs, mobs, lphs, mphs;
        double dcosines_l_pix_ref, dcosines_m_pix_ref, dcosines_l_pix_phs, dcosines_m_pix_phs;
        for(int d=0; d<nMeasurementSets; d++) {
                if(verbose_flag)
                        printf("Dataset: %s\n", datasets[d].name);
                for(int f=0; f<datasets[d].data.nfields; f++) {

                        direccos(datasets[d].fields[f].ref_ra, datasets[d].fields[f].ref_dec, raimage, decimage, &lobs,  &mobs);
                        direccos(datasets[d].fields[f].phs_ra, datasets[d].fields[f].phs_dec, raimage, decimage, &lphs,  &mphs);

                        dcosines_l_pix_ref = lobs/ -deltax; // Radians to pixels
                        dcosines_m_pix_ref = mobs/fabs(deltay); // Radians to pixels

                        dcosines_l_pix_phs = lphs/ -deltax; // Radians to pixels
                        dcosines_m_pix_phs = mphs/fabs(deltay); // Radians to pixels

                        if(verbose_flag)
                        {
                                printf("Ref: l (pix): %e, m (pix): %e\n", dcosines_l_pix_ref, dcosines_m_pix_ref);
                                printf("Phase: l (pix): %e, m (pix): %e\n", dcosines_l_pix_phs, dcosines_m_pix_phs);

                        }


                        datasets[d].fields[f].ref_xobs = (crpix1 - 1.0f) + dcosines_l_pix_ref;// + 6.0f;
                        datasets[d].fields[f].ref_yobs = (crpix2 - 1.0f) + dcosines_m_pix_ref;// - 7.0f;

                        datasets[d].fields[f].phs_xobs = (crpix1 - 1.0f) + dcosines_l_pix_phs;// + 5.0f;
                        datasets[d].fields[f].phs_yobs = (crpix2 - 1.0f) + dcosines_m_pix_phs;// - 7.0f;


                        if(verbose_flag) {
                                printf("Ref: Field %d - Ra: %.16e (rad), dec: %.16e (rad), x0: %f (pix), y0: %f (pix)\n", f, datasets[d].fields[f].ref_ra, datasets[d].fields[f].ref_dec,
                                       datasets[d].fields[f].ref_xobs, datasets[d].fields[f].ref_yobs);
                                printf("Phase: Field %d - Ra: %.16e (rad), dec: %.16e (rad), x0: %f (pix), y0: %f (pix)\n", f, datasets[d].fields[f].phs_ra, datasets[d].fields[f].phs_dec,
                                       datasets[d].fields[f].phs_xobs, datasets[d].fields[f].phs_yobs);
                        }

                        if(datasets[d].fields[f].ref_xobs < 0 || datasets[d].fields[f].ref_xobs >= M || datasets[d].fields[f].ref_xobs < 0 || datasets[d].fields[f].ref_yobs >= N) {
                                printf("Dataset: %s\n", datasets[d].name);
                                printf("Pointing reference center (%f,%f) is outside the range of the image\n", datasets[d].fields[f].ref_xobs, datasets[d].fields[f].ref_yobs);
                                goToError();
                        }

                        if(datasets[d].fields[f].phs_xobs < 0 || datasets[d].fields[f].phs_xobs >= M || datasets[d].fields[f].phs_xobs < 0 || datasets[d].fields[f].phs_yobs >= N) {
                                printf("Dataset: %s\n", datasets[d].name);
                                printf("Pointing phase center (%f,%f) is outside the range of the image\n", datasets[d].fields[f].phs_xobs, datasets[d].fields[f].phs_yobs);
                                goToError();
                        }
                }
        }
        ////////////////////////////////////////////////////////MAKE STARTING IMAGE////////////////////////////////////////////////////////

        host_I = (float*)malloc(M*N*sizeof(float)*image_count);

        for(int i=0; i<M; i++) {
                for(int j=0; j<N; j++) {
                        for(int k=0; k<image_count; k++) {
                                host_I[N*M*k+N*i+j] = initial_values[k];
                        }
                }
        }

        ////////////////////////////////////////////////CUDA MEMORY ALLOCATION FOR DEVICE///////////////////////////////////////////////////

        for(int g=0; g<num_gpus; g++) {
                hipSetDevice((g%num_gpus) + firstgpu);
                gpuErrchk(hipMalloc((void**)&vars_gpu[g].device_V, sizeof(hipfftComplex)*M*N));
                gpuErrchk(hipMalloc((void**)&vars_gpu[g].device_I_nu, sizeof(hipfftComplex)*M*N));
        }


        hipSetDevice(firstgpu);

        gpuErrchk(hipMalloc((void**)&device_Image, sizeof(float)*M*N*image_count));
        gpuErrchk(hipMemset(device_Image, 0, sizeof(float)*M*N*image_count));

        gpuErrchk(hipMemcpy(device_Image, host_I, sizeof(float)*N*M*image_count, hipMemcpyHostToDevice));

        gpuErrchk(hipMalloc((void**)&device_noise_image, sizeof(float)*M*N));
        gpuErrchk(hipMemset(device_noise_image, 0, sizeof(float)*M*N));

        gpuErrchk(hipMalloc((void**)&device_weight_image, sizeof(float)*M*N));
        gpuErrchk(hipMemset(device_weight_image, 0, sizeof(float)*M*N));



        for(int g=0; g<num_gpus; g++) {
                hipSetDevice((g%num_gpus) + firstgpu);
                gpuErrchk(hipMemset(vars_gpu[g].device_V, 0, sizeof(hipfftComplex)*M*N));
                gpuErrchk(hipMemset(vars_gpu[g].device_I_nu, 0, sizeof(hipfftComplex)*M*N));

        }

        /////////// MAKING IMAGE OBJECT /////////////
        image = new Image(device_Image, image_count);
        imageMap *functionPtr = (imageMap*)malloc(sizeof(imageMap)*image_count);
        image->setFunctionMapping(functionPtr);

        for(int i = 0; i < image_count; i++)
        {
                if(nopositivity)
                {
                        functionPtr[i].evaluateXt = defaultEvaluateXt;
                        functionPtr[i].newP = defaultNewP;
                }else{
                        if(!i)
                        {
                                functionPtr[i].evaluateXt = particularEvaluateXt;
                                functionPtr[i].newP = particularNewP;
                        }else{
                                functionPtr[i].evaluateXt = defaultEvaluateXt;
                                functionPtr[i].newP = defaultNewP;
                        }
                }
        }


        initFFT(vars_gpu, M, N, firstgpu, num_gpus);

        //Time is taken from first kernel
        t = clock();
        start = omp_get_wtime();
        for(int d=0; d<nMeasurementSets; d++) {
                for(int f=0; f < datasets[d].data.nfields; f++) {
                        if(num_gpus == 1) {
                                hipSetDevice(selected);
                                for(int i=0; i<datasets[d].data.total_frequencies; i++) {
                                        for(int s=0; s<datasets[d].data.nstokes; s++) {
                                                hermitianSymmetry << < datasets[d].fields[f].device_visibilities[i][s].numBlocksUV,
                                                        datasets[d].fields[f].device_visibilities[i][s].threadsPerBlockUV >> >
                                                (datasets[d].fields[f].device_visibilities[i][s].uvw, datasets[d].fields[f].device_visibilities[i][s].Vo, datasets[d].fields[f].nu[i], datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]);
                                                gpuErrchk(hipDeviceSynchronize());
                                        }
                                }

                        }else{
                            #pragma omp parallel for schedule(static,1)
                                for (int i = 0; i < datasets[d].data.total_frequencies; i++)
                                {
                                        unsigned int j = omp_get_thread_num();
                                        //unsigned int num_cpu_threads = omp_get_num_threads();
                                        // set and check the CUDA device for this CPU thread
                                        int gpu_id = -1;
                                        hipSetDevice((i%num_gpus) + firstgpu); // "% num_gpus" allows more CPU threads than GPU devices
                                        hipGetDevice(&gpu_id);
                                        for(int s=0; s<datasets[d].data.nstokes; s++) {
                                                hermitianSymmetry << < datasets[d].fields[f].device_visibilities[i][s].numBlocksUV,
                                                        datasets[d].fields[f].device_visibilities[i][s].threadsPerBlockUV >> >
                                                (datasets[d].fields[f].device_visibilities[i][s].uvw, datasets[d].fields[f].device_visibilities[i][s].Vo, datasets[d].fields[f].nu[i], datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]);
                                                gpuErrchk(hipDeviceSynchronize());
                                        }
                                }


                        }
                }

                if(num_gpus == 1) {
                        hipSetDevice(selected);
                        for(int f=0; f<datasets[d].data.nfields; f++) {
                                for(int i=0; i<datasets[d].data.total_frequencies; i++) {
                                        if(datasets[d].fields[f].numVisibilitiesPerFreq[i] > 0) {
                                                total_attenuation<<<numBlocksNN, threadsPerBlockNN>>>(datasets[d].fields[f].atten_image, datasets[d].antenna_diameter, datasets[d].pb_factor, datasets[d].pb_cutoff, datasets[d].fields[f].nu[i], datasets[d].fields[f].ref_xobs, datasets[d].fields[f].ref_yobs, DELTAX, DELTAY, N);
                                                gpuErrchk(hipDeviceSynchronize());
                                        }
                                }
                        }
                }else{
                        for(int f=0; f<datasets[d].data.nfields; f++) {
                                #pragma omp parallel for schedule(static,1)
                                for (int i = 0; i < datasets[d].data.total_frequencies; i++)
                                {
                                        unsigned int j = omp_get_thread_num();
                                        //unsigned int num_cpu_threads = omp_get_num_threads();
                                        // set and check the CUDA device for this CPU thread
                                        int gpu_id = -1;
                                        hipSetDevice((i%num_gpus) + firstgpu); // "% num_gpus" allows more CPU threads than GPU devices
                                        hipGetDevice(&gpu_id);
                                        if(datasets[d].fields[f].numVisibilitiesPerFreq[i] > 0) {
                                                #pragma omp critical
                                                {
                                                        total_attenuation<<<numBlocksNN, threadsPerBlockNN>>>(datasets[d].fields[f].atten_image, datasets[d].antenna_diameter, datasets[d].pb_factor, datasets[d].pb_cutoff, datasets[d].fields[f].nu[i], datasets[d].fields[f].ref_xobs, datasets[d].fields[f].ref_yobs, DELTAX, DELTAY, N);
                                                        gpuErrchk(hipDeviceSynchronize());
                                                }
                                        }
                                }
                        }
                }

                for(int f=0; f<datasets[d].data.nfields; f++) {
                        if(datasets[d].fields[f].valid_frequencies > 0) {
                                if(num_gpus == 1) {
                                        hipSetDevice(selected);
                                        mean_attenuation<<<numBlocksNN, threadsPerBlockNN>>>(datasets[d].fields[f].atten_image, datasets[d].fields[f].valid_frequencies, N);
                                        gpuErrchk(hipDeviceSynchronize());
                                }else{
                                        hipSetDevice(firstgpu);
                                        mean_attenuation<<<numBlocksNN, threadsPerBlockNN>>>(datasets[d].fields[f].atten_image, datasets[d].fields[f].valid_frequencies, N);
                                        gpuErrchk(hipDeviceSynchronize());
                                }
                                if(print_images) {
                                        std::string atten_name =  "dataset_" + std::to_string(d) + "_atten";
                                        iohandler->IoPrintImageIteration(datasets[d].fields[f].atten_image, mod_in, mempath, atten_name.c_str(), "", f, 0, 1.0, M, N);
                                }
                        }
                }
        }



        if(num_gpus == 1) {
                hipSetDevice(selected);
        }else{
                hipSetDevice(firstgpu);
        }

        for(int d=0; d<nMeasurementSets; d++) {
                for(int f=0; f<datasets[d].data.nfields; f++) {
                        weight_image<<<numBlocksNN, threadsPerBlockNN>>>(device_weight_image, datasets[d].fields[f].atten_image, noise_jypix, N);
                        gpuErrchk(hipDeviceSynchronize());
                }
        }

        noise_image<<<numBlocksNN, threadsPerBlockNN>>>(device_noise_image, device_weight_image, noise_jypix, N);
        gpuErrchk(hipDeviceSynchronize());
        if(print_images)
                iohandler->IoPrintImage(device_noise_image, mod_in, mempath, "noise.fits", "", 0, 0, 1.0, M, N);


        float *host_noise_image = (float*)malloc(M*N*sizeof(float));
        gpuErrchk(hipMemcpy2D(host_noise_image, sizeof(float), device_noise_image, sizeof(float), sizeof(float), M*N, hipMemcpyDeviceToHost));
        float noise_min = *std::min_element(host_noise_image,host_noise_image+(M*N));

        fg_scale = noise_min;
        noise_cut = noise_cut * noise_min;
        if(verbose_flag) {
                printf("fg_scale = %e\n", fg_scale);
                printf("noise (Jy/pix) = %e\n", noise_jypix);
        }
        free(host_noise_image);
        hipFree(device_weight_image);
        for(int d=0; d<nMeasurementSets; d++) {
                for(int f=0; f<datasets[d].data.nfields; f++) {
                        hipFree(datasets[d].fields[f].atten_image);
                }
        }
};

void MFS::run()
{
        //printf("\n\nStarting Fletcher Reeves Polak Ribiere method (Conj. Grad.)\n\n");
        printf("\n\nStarting Optimizator\n");
        optimizator->getObjectiveFuntion()->setIo(iohandler);
        optimizator->getObjectiveFuntion()->setPrintImages(print_images);
        //optimizator->getObjectiveFuntion()->setIoOrderIterations(IoOrderIterations);
        if(this->Order == NULL) {
                if(imagesChanged)
                {
                        optimizator->setImage(image);
                        optimizator->minimizate();
                }else if(image_count == 2) {
                        optimizator->setImage(image);
                        optimizator->setFlag(0);
                        optimizator->minimizate();
                        optimizator->setFlag(1);
                        optimizator->minimizate();
                        optimizator->setFlag(2);
                        optimizator->minimizate();
                        optimizator->setFlag(3);
                        optimizator->minimizate();
                }
        }else{
                (this->Order)(optimizator, image);
        }

        t = clock() - t;
        end = omp_get_wtime();
        printf("Minimization ended successfully\n\n");
        printf("Iterations: %d\n", iter);
        printf("chi2: %f\n", final_chi2);
        printf("0.5*chi2: %f\n", 0.5*final_chi2);
        printf("Total visibilities: %d\n", total_visibilities);
        printf("Reduced-chi2 (Num visibilities): %f\n", (0.5*final_chi2)/total_visibilities);
        printf("Reduced-chi2 (Weights sum): %f\n", (0.5*final_chi2)/sum_weights);
        printf("S: %f\n", final_S);
        if(reg_term != 1) {
                printf("Normalized S: %f\n", final_S/(M*N));
        }else{
                printf("Normalized S: %f\n", final_S/(M*M*N*N));
        }
        printf("lambda*S: %f\n\n", lambda*final_S);
        double time_taken = ((double)t)/CLOCKS_PER_SEC;
        double wall_time = end-start;
        printf("Total CPU time: %lf\n", time_taken);
        printf("Wall time: %lf\n\n\n", wall_time);

        if(strcmp(variables.ofile,"NULL") != 0) {
                FILE *outfile = fopen(variables.ofile, "w");
                if (outfile == NULL)
                {
                        printf("Error opening output file!\n");
                        goToError();
                }

                fprintf(outfile, "Iterations: %d\n", iter);
                fprintf(outfile, "chi2: %f\n", final_chi2);
                fprintf(outfile, "0.5*chi2: %f\n", 0.5*final_chi2);
                fprintf(outfile, "Total visibilities: %d\n", total_visibilities);
                fprintf(outfile, "Reduced-chi2 (Num visibilities): %f\n", (0.5*final_chi2)/total_visibilities);
                fprintf(outfile, "Reduced-chi2 (Weights sum): %f\n", (0.5*final_chi2)/sum_weights);
                fprintf(outfile, "S: %f\n", final_S);
                if(reg_term != 1) {
                        fprintf(outfile, "Normalized S: %f\n", final_S/(M*N));
                }else{
                        fprintf(outfile, "Normalized S: %f\n", final_S/(M*M*N*N));
                }
                fprintf(outfile, "lambda*S: %f\n", lambda*final_S);
                fprintf(outfile, "Wall time: %lf", wall_time);
                fclose(outfile);
        }
        //Pass residuals to host
        printf("Saving final image to disk\n");
        if(IoOrderEnd == NULL) {
                iohandler->IoPrintImage(image->getImage(), mod_in, "", out_image, "JY/PIXEL", iter, 0, fg_scale, M, N);
                iohandler->IoPrintImage(image->getImage(), mod_in, "", "alpha.fits", "", iter, 1, 1.0, M, N);
        }else{
                (IoOrderEnd)(image->getImage(), iohandler);
        }

        if(print_errors) /* flag for print error image */
        {
                if(this->error == NULL)
                {
                        this->error = Singleton<ErrorFactory>::Instance().CreateError(0);
                }
                /* code to calculate error */
                /* make void * params */
                printf("Calculating Error Images\n");
                this->error->calculateErrorImage(this->image, this->visibilities);
                if(IoOrderError == NULL) {
                        iohandler->IoPrintImage(image->getErrorImage(), mod_in, "", "error_Inu_0.fits", "JY/PIXEL", iter, 0, 1.0, M, N);
                        iohandler->IoPrintImage(image->getErrorImage(), mod_in, "", "error_alpha.fits", "", iter, 1, 1.0, M, N);
                }else{
                        (IoOrderError)(image->getErrorImage(), iohandler);
                }

        }

        if(!gridding)
        {
                //Saving residuals to disk
                for(int d=0; d<nMeasurementSets; d++) {
                        residualsToHost(datasets[d].fields, datasets[d].data, num_gpus, firstgpu);
                }
        }else{
                double deltax = RPDEG_D*DELTAX; //radians
                double deltay = RPDEG_D*DELTAY; //radians
                deltau = 1.0 / (M * deltax);
                deltav = 1.0 / (N * deltay);

                printf("Visibilities are gridded, we will need to de-grid to save them in a Measurement Set File\n");
                omp_set_num_threads(gridding);
                for(int d=0; d<nMeasurementSets; d++)
                        degridding(datasets[d].fields, datasets[d].data, deltau, deltav, num_gpus, firstgpu, variables.blockSizeV, M, N);

                omp_set_num_threads(num_gpus);

                for(int d=0; d<nMeasurementSets; d++)
                        residualsToHost(datasets[d].fields, datasets[d].data, num_gpus, firstgpu);

        }

        printf("Saving residuals to MS...\n");
        for(int d=0; d<nMeasurementSets; d++) {
                iohandler->IowriteMS(datasets[d].name, datasets[d].oname, "DATA", datasets[d].fields, datasets[d].data, random_probability, false, false, false, verbose_flag);

                if(save_model)
                        iohandler->IowriteMS(datasets[d].name, datasets[d].oname, "MODEL", datasets[d].fields, datasets[d].data, random_probability, true, false, false, verbose_flag);

        }

        printf("Residuals saved.\n");


};

void MFS::unSetDevice()
{
        //Free device and host memory
        printf("Freeing device memory\n");
        if(num_gpus == 1) {
                hipSetDevice(selected);
        }else{
                hipSetDevice(firstgpu);
        }

        for(int d=0; d<nMeasurementSets; d++) {
                for(int f=0; f<datasets[d].data.nfields; f++) {
                        for(int i=0; i<datasets[d].data.total_frequencies; i++) {

                                if(num_gpus > 1) {
                                        hipSetDevice((i%num_gpus) + firstgpu);
                                }
                                for(int s=0; s<datasets[d].data.nstokes; s++) {
                                        hipFree(datasets[d].fields[f].device_visibilities[i][s].uvw);
                                        hipFree(datasets[d].fields[f].device_visibilities[i][s].weight);
                                        hipFree(datasets[d].fields[f].device_visibilities[i][s].Vr);
                                        hipFree(datasets[d].fields[f].device_visibilities[i][s].Vm);
                                        hipFree(datasets[d].fields[f].device_visibilities[i][s].Vo);
                                }

                        }
                }
        }

        printf("Freeing cuFFT plans\n");
        for(int g=0; g<num_gpus; g++) {
                hipSetDevice((g%num_gpus) + firstgpu);
                hipfftDestroy(vars_gpu[g].plan);
        }

        printf("Freeing host memory\n");
        for(int d=0; d<nMeasurementSets; d++) {
                for(int f=0; f<datasets[d].data.nfields; f++) {
                        for(int i=0; i<datasets[d].data.total_frequencies; i++) {
                                for(int s=0; s<datasets[d].data.nstokes; s++) {
                                        if (datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s] > 0) {
                                                datasets[d].fields[f].visibilities[i][s].uvw.clear();
                                                datasets[d].fields[f].visibilities[i][s].weight.clear();
                                                datasets[d].fields[f].visibilities[i][s].Vo.clear();
                                                datasets[d].fields[f].visibilities[i][s].Vm.clear();
                                        }
                                }
                        }
                }
        }



        hipFree(device_Image);

        for(int g=0; g<num_gpus; g++) {
                hipSetDevice((g%num_gpus) + firstgpu);
                hipFree(vars_gpu[g].device_V);
                hipFree(vars_gpu[g].device_I_nu);
        }


        hipSetDevice(firstgpu);


        hipFree(device_noise_image);

        hipFree(device_dphi);
        hipFree(device_dchi2_total);
        hipFree(device_dS);

        hipFree(device_S);

        //Disabling UVA
        if(num_gpus > 1) {
                for(int i=firstgpu+1; i<num_gpus+firstgpu; i++) {
                        hipSetDevice(firstgpu);
                        hipDeviceDisablePeerAccess(i);
                        hipSetDevice(i);
                        hipDeviceDisablePeerAccess(firstgpu);
                }

                for(int i=0; i<num_gpus; i++ ) {
                        hipSetDevice((i%num_gpus) + firstgpu);
                        hipDeviceReset();
                }
        }
        free(host_I);
        free(msinput);
        free(t_telescope);
        free(msoutput);
        free(modinput);

        for(int i=0; i< nMeasurementSets; i++) {
                free(datasets[i].name);
                free(datasets[i].oname);
        }

        iohandler->IocloseCanvas(mod_in);
};

namespace {
Synthesizer* CreateMFS()
{
        return new MFS;
}
const int MFSID = 0;
const bool RegisteredMFS = Singleton<SynthesizerFactory>::Instance().RegisterSynthesizer(MFSID, CreateMFS);
};
