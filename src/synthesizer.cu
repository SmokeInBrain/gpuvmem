#include "hip/hip_runtime.h"
#include "synthesizer.cuh"
#include "imageProcessor.cuh"


long M, N, numVisibilities;
int iter=0;

float *device_Image, *device_dphi, *device_dchi2_total, *device_dS, *device_S, beam_noise, beam_bmaj, *device_noise_image, *device_weight_image, *device_distance_image;
float beam_bmin, b_noise_aux, noise_cut, MINPIX, minpix, lambda, ftol, random_probability = 1.0;
float noise_jypix, fg_scale, final_chi2, final_S, eta, robust_param;
float *host_I, sum_weights, *penalizators;

dim3 threadsPerBlockNN;
dim3 numBlocksNN;

int threadsVectorReduceNN, blocksVectorReduceNN, nopositivity = 0, verbose_flag = 0, clip_flag = 0, apply_noise = 0, print_images = 0, save_model_input = 0;
int gridding, it_maximum, status_mod_in;
int multigpu, firstgpu, selected, reg_term, total_visibilities, image_count, nPenalizators, print_errors, nMeasurementSets=0, max_number_vis;
char *output, *mempath, *out_image, *msinput, *msoutput, *inputdat, *modinput;
float nu_0, threshold;
extern int num_gpus;

double ra, dec, crpix1, crpix2, DELTAX, DELTAY, deltau, deltav;

fitsfile *mod_in;

std::vector<float> initial_values;
std::vector<MSDataset> datasets;

varsPerGPU *vars_gpu;

Vars variables;

clock_t t;
double start, end;

float noise_min = 1E32;

inline bool IsGPUCapableP2P(hipDeviceProp_t *pProp)
{
  #ifdef _WIN32
        return (bool)(pProp->tccDriver ? true : false);
  #else
        return (bool)(pProp->major >= 2);
  #endif
}

std::vector<std::string> MFS::countAndSeparateStrings(char *input)
{
        char *pt;
        std::vector<std::string> ret;

        int counter = 0;
        pt = strtok(input, ",");
        while(pt!=NULL) {
                std::string s(pt);
                ret.push_back(s);
                pt = strtok (NULL, ",");
        }

        free(pt);
        return ret;
}

void MFS::configure(int argc, char **argv)
{
        if(iohandler == NULL)
        {
                iohandler = Singleton<IoFactory>::Instance().CreateIo(0);
        }

        variables = getOptions(argc, argv);
        msinput = variables.input;
        msoutput = variables.output;
        inputdat = variables.inputdat;
        modinput = variables.modin;
        out_image = variables.output_image;
        mempath = variables.path;
        it_maximum = variables.it_max;
        total_visibilities = 0;
        b_noise_aux = variables.noise;
        noise_cut = variables.noise_cut;
        random_probability = variables.randoms;
        eta = variables.eta;
        gridding = variables.gridding;
        nu_0 = variables.nu_0;
        robust_param = variables.robust_param;
        threshold = variables.threshold * 5.0;
        std::vector<std::string> string_values;
        std::vector<std::string> s_output_values;
        int n_outputs;


        if(strcmp(msinput, "NULL")!=0) {
                string_values = countAndSeparateStrings(msinput);
                nMeasurementSets = string_values.size();
        }else{
                printf("Datasets files were not provided\n");
                print_help();
                exit(-1);
        }

        if(strcmp(msoutput, "NULL")!=0) {
                s_output_values = countAndSeparateStrings(msoutput);
                n_outputs = s_output_values.size();
        }else{
                printf("Output/s was/were not provided\n");
                print_help();
                exit(-1);
        }

        if(n_outputs != nMeasurementSets) {
                printf("Number of input datasets should be equal to the number of output datasets\n");
                exit(-1);
        }

        if(verbose_flag)
                printf("Number of input datasets %d\n", nMeasurementSets);

        for(int i=0; i< nMeasurementSets; i++) {
                datasets.push_back(MSDataset());
                datasets[i].name = (char*)malloc((string_values[i].length()+1)*sizeof(char));
                datasets[i].oname = (char*)malloc((s_output_values[i].length()+1)*sizeof(char));
                strcpy(datasets[i].name, string_values[i].c_str());
                strcpy(datasets[i].oname, s_output_values[i].c_str());

        }

        string_values.clear();
        s_output_values.clear();

        if(strcmp(variables.initial_values, "NULL")!=0) {
                string_values = countAndSeparateStrings(variables.initial_values);
                image_count = string_values.size();
        }else{
                printf("Initial values for image/s were not provided\n");
                print_help();
                exit(-1);
        }

        for(int i=0; i< image_count; i++)
                initial_values.push_back(atof(string_values[i].c_str()));

        string_values.clear();
        if(image_count == 1)
        {
                initial_values.push_back(0.0f);
                image_count++;
                imagesChanged = 1;
        }

        /*
         *
         * Create directory to save images for each iterations
         */
        struct stat st = {0};
        if(print_images)
                if(stat(mempath, &st) == -1) mkdir(mempath,0700);

        hipDeviceProp_t dprop[num_gpus];

        if(verbose_flag) {
                printf("Number of host CPUs:\t%d\n", omp_get_num_procs());
                printf("Number of CUDA devices:\t%d\n", num_gpus);

                for(int i = 0; i < num_gpus; i++) {
                        hipGetDeviceProperties(&dprop[i], i);
                        printf("> GPU%d = \"%15s\" %s capable of Peer-to-Peer (P2P)\n", i, dprop[i].name, (IsGPUCapableP2P(&dprop[i]) ? "IS " : "NOT"));
                }
                printf("---------------------------\n");
        }

        hipGetDeviceProperties(&dprop[0], 0);
        if(variables.blockSizeX*variables.blockSizeY > dprop[0].maxThreadsPerBlock || variables.blockSizeV > dprop[0].maxThreadsPerBlock) {
                printf("Block size X: %d\n", variables.blockSizeX);
                printf("Block size Y: %d\n", variables.blockSizeY);
                printf("Block size X*Y: %d\n", variables.blockSizeX*variables.blockSizeY);
                printf("Block size V: %d\n", variables.blockSizeV);
                printf("ERROR. The maximum threads per block cannot be greater than %d\n", dprop[0].maxThreadsPerBlock);
                exit(-1);
        }

        if(variables.blockSizeX > dprop[0].maxThreadsDim[0] || variables.blockSizeY > dprop[0].maxThreadsDim[1] || variables.blockSizeV > dprop[0].maxThreadsDim[0]) {
                printf("Block size X: %d\n", variables.blockSizeX);
                printf("Block size Y: %d\n", variables.blockSizeY);
                printf("Block size V: %d\n", variables.blockSizeV);
                printf("ERROR. The size of the blocksize cannot exceed X: %d Y: %d Z: %d\n", dprop[0].maxThreadsDim[0], dprop[0].maxThreadsDim[1], dprop[0].maxThreadsDim[2]);
                exit(-1);
        }

        if(selected > num_gpus || selected < 0) {
                printf("ERROR. THE SELECTED GPU DOESN'T EXIST\n");
                exit(-1);
        }

        readInputDat(inputdat);
        canvasVariables canvas_vars = iohandler->IoreadCanvas(modinput, mod_in, b_noise_aux, status_mod_in, verbose_flag);

        M = canvas_vars.M;
        N = canvas_vars.N;
        DELTAX = canvas_vars.DELTAX;
        DELTAY = canvas_vars.DELTAY;
        ra = canvas_vars.ra;
        dec = canvas_vars.dec;
        crpix1 = canvas_vars.crpix1;
        crpix2 = canvas_vars.crpix2;
        beam_bmaj = canvas_vars.beam_bmaj;
        beam_bmin = canvas_vars.beam_bmin;
        beam_noise = canvas_vars.beam_noise;

        if(verbose_flag)
                printf("Reading data from MSs\n");

        std::vector<float> ms_ref_freqs;
        std::vector<float> ms_max_freqs;
        std::vector<float> ms_max_blength;
        std::vector<float> ms_min_blength;
        std::vector<float> ms_uvmax_wavelength;
        for(int d=0; d<nMeasurementSets; d++) {
                if(apply_noise) {
                        iohandler->IoreadMS(datasets[d].name, datasets[d].antennas, datasets[d].fields, &datasets[d].data, true, false, random_probability, gridding);
                }else{
                        iohandler->IoreadMS(datasets[d].name, datasets[d].antennas, datasets[d].fields, &datasets[d].data, false, false, random_probability, gridding);
                }
                ms_ref_freqs.push_back(datasets[d].data.ref_freq);
                ms_max_freqs.push_back(datasets[d].data.max_freq);
                ms_max_blength.push_back(datasets[d].data.max_blength);
                ms_min_blength.push_back(datasets[d].data.min_blength);
                ms_uvmax_wavelength.push_back(datasets[d].data.uvmax_wavelength);
                printf("Dataset %d: %s - Antenna diameter: %.3f metres\n", d, datasets[d].name, datasets[d].antennas[0].antenna_diameter);
        }

        /*
           Calculating theoretical resolution
         */
        float max_freq = *max_element(ms_max_freqs.begin(), ms_max_freqs.end());
        float max_blength = *max_element(ms_max_blength.begin(), ms_max_blength.end());
        float min_wlength = freq_to_wavelength(max_freq);
        float max_resolution = (min_wlength/(4*max_blength))/RPARCSEC;
        double max_uvmax_wavelength = *max_element(ms_uvmax_wavelength.begin(), ms_uvmax_wavelength.end()) + 1E-5;
        printf("The maximum theoretical resolution of this/these dataset/s is ~%f arcsec\n", max_resolution);

        if(nu_0 < 0) {
                printf("Reference frequency not provided. It will be calculated as the median of all the arrays of frequencies.\n");
                nu_0 = median(ms_ref_freqs);
        }
        printf("Reference frequency: %e Hz\n", nu_0);
        double deltau_theo = 2.0*max_uvmax_wavelength/(M-1);
        double deltax_theo = 1.0/(M*deltau_theo)/RPARCSEC;
        printf("The pixel size has to be less or equal to %lf arcsec\n", deltax_theo);
        printf("Actual pixel size is %lf arcsec\n", DELTAX*3600.0);



        if(verbose_flag) {
                for(int i=0; i<nMeasurementSets; i++) {
                        printf("Dataset %d: %s\n", i, datasets[i].name);
                        printf("\tNumber of fields = %d\n", datasets[i].data.nfields);
                        printf("\tNumber of frequencies = %d\n", datasets[i].data.total_frequencies);
                        printf("\tNumber of Stokes = %d\n", datasets[i].data.nstokes);
                }
        }

        multigpu = 0;
        firstgpu = 0;
        int count_gpus;

        string_values = countAndSeparateStrings(variables.gpus);
        count_gpus = string_values.size();

        if(count_gpus == 1) {
                multigpu = 0;
                selected = atoi(string_values[0].c_str());
        }else{
                multigpu = count_gpus;
                firstgpu = atoi(string_values[0].c_str());
        }


        string_values.clear();

        if(strcmp(variables.penalization_factors, "NULL")!=0) {

                string_values = countAndSeparateStrings(variables.penalization_factors);
                nPenalizators = string_values.size();
                penalizators = (float*)malloc(sizeof(float)*nPenalizators);
                for(int i = 0; i < nPenalizators; i++)
                        penalizators[i] = atof(string_values[i].c_str());

        }else{
                printf("No regularization factors provided\n");
        }
        string_values.clear();

        int max_nfreq = 1;
        if(multigpu < 0 || multigpu > num_gpus) {
                printf("ERROR. NUMBER OF GPUS CANNOT BE NEGATIVE OR GREATER THAN THE NUMBER OF GPUS\n");
                exit(-1);
        }else{
                if(multigpu == 0) {
                        num_gpus = 1;
                        firstgpu = selected;
                }else{
                        for(int d=0; d<nMeasurementSets; d++) {
                                if(datasets[d].data.total_frequencies > max_nfreq)
                                        max_nfreq = datasets[d].data.total_frequencies;
                        }

                        if(max_nfreq == 1) {
                                printf("ONLY ONE FREQUENCY. CHANGING NUMBER OF GPUS TO 1\n");
                                num_gpus = 1;
                        }else{
                                num_gpus = multigpu;
                                omp_set_num_threads(num_gpus);
                        }
                }
        }

        //printf("number of FINAL host CPUs:\t%d\n", omp_get_num_procs());
        if(verbose_flag) {
                printf("Number of CUDA devices and threads: %d\n", num_gpus);
        }

        //Check peer access if there is more than 1 GPU
        if(num_gpus > 1) {
                for(int i=firstgpu + 1; i< firstgpu + num_gpus; i++) {
                        hipDeviceProp_t dprop0, dpropX;
                        hipGetDeviceProperties(&dprop0, firstgpu);
                        hipGetDeviceProperties(&dpropX, i);
                        int canAccessPeer0_x, canAccessPeerx_0;
                        hipDeviceCanAccessPeer(&canAccessPeer0_x, firstgpu, i);
                        hipDeviceCanAccessPeer(&canAccessPeerx_0, i, firstgpu);
                        if(verbose_flag) {
                                printf("> Peer-to-Peer (P2P) access from %s (GPU%d) -> %s (GPU%d) : %s\n", dprop0.name, firstgpu, dpropX.name, i, canAccessPeer0_x ? "Yes" : "No");
                                printf("> Peer-to-Peer (P2P) access from %s (GPU%d) -> %s (GPU%d) : %s\n", dpropX.name, i, dprop0.name, firstgpu, canAccessPeerx_0 ? "Yes" : "No");
                        }
                        if(canAccessPeer0_x == 0 || canAccessPeerx_0 == 0) {
                                printf("Two or more SM 2.0 class GPUs are required for %s to run.\n", argv[0]);
                                printf("Support for UVA requires a GPU with SM 2.0 capabilities.\n");
                                printf("Peer to Peer access is not available between GPU%d <-> GPU%d, waiving test.\n", 0, i);
                                exit(EXIT_SUCCESS);
                        }else{
                                hipSetDevice(firstgpu);
                                if(verbose_flag) {
                                        printf("Granting access from %d to %d...\n",firstgpu, i);
                                }
                                hipDeviceEnablePeerAccess(i,0);
                                hipSetDevice(i);
                                if(verbose_flag) {
                                        printf("Granting access from %d to %d...\n", i, firstgpu);
                                }
                                hipDeviceEnablePeerAccess(firstgpu,0);
                                if(verbose_flag) {
                                        printf("Checking GPU %d and GPU %d for UVA capabilities...\n", firstgpu, i);
                                }
                                const bool has_uva = (dprop0.unifiedAddressing && dpropX.unifiedAddressing);
                                if(verbose_flag) {
                                        printf("> %s (GPU%d) supports UVA: %s\n", dprop0.name, firstgpu, (dprop0.unifiedAddressing ? "Yes" : "No"));
                                        printf("> %s (GPU%d) supports UVA: %s\n", dpropX.name, i, (dpropX.unifiedAddressing ? "Yes" : "No"));
                                }
                                if (has_uva) {
                                        if(verbose_flag) {
                                                printf("Both GPUs can support UVA, enabling...\n");
                                        }
                                }
                                else{
                                        printf("At least one of the two GPUs does NOT support UVA, waiving test.\n");
                                        exit(EXIT_SUCCESS);
                                }
                        }
                }

        }

        vars_gpu = (varsPerGPU*)malloc(num_gpus*sizeof(varsPerGPU));

        this->visibilities = new Visibilities();
        this->visibilities->setMSDataset(datasets);
        this->visibilities->setNDatasets(nMeasurementSets);

        double deltax = RPDEG_D*DELTAX; //radians
        double deltay = RPDEG_D*DELTAY; //radians
        deltau = 1.0 / (M * deltax);
        deltav = 1.0 / (N * deltay);
        printf("Constructing Antialiasing Kernel\n");
        ckernel->constructKernel(1.0f, 0.0f, 0.0f, fabsf(deltau), fabsf(deltav));
        printf("Using an antialiasing kernel of size (%d, %d) and support (%d, %d)\n", ckernel->getm(), ckernel->getn(), ckernel->getSupportX(), ckernel->getSupportY());
        if(gridding) {
                printf("Doing gridding\n");
                omp_set_num_threads(gridding);
                for(int d=0; d<nMeasurementSets; d++)
                        do_gridding(datasets[d].fields, &datasets[d].data, deltau, deltav, M, N, robust_param, this->ckernel);

                omp_set_num_threads(num_gpus);
        }
}

void MFS::setDevice()
{
        double deltax = RPDEG_D*DELTAX; //radians
        double deltay = RPDEG_D*DELTAY; //radians
        deltau = 1.0 / (M * deltax);
        deltav = 1.0 / (N * deltay);

        if(verbose_flag) {
                printf("MS File Successfully Read\n");
                if(beam_noise == -1) {
                        printf("Beam noise wasn't provided by the user... Calculating...\n");
                }
        }


        sum_weights = calculateNoise(datasets, &total_visibilities, variables.blockSizeV, gridding);


        this->visibilities->setTotalVisibilities(total_visibilities);

        for(int d=0; d<nMeasurementSets; d++) {
                for(int f=0; f<datasets[d].data.nfields; f++) {
                        if(num_gpus == 1) {
                                hipSetDevice(selected);
                                for(int i=0; i<datasets[d].data.total_frequencies; i++) {
                                        for(int s=0; s<datasets[d].data.nstokes; s++) {
                                                checkCudaErrors(hipMalloc(&datasets[d].fields[f].device_visibilities[i][s].uvw,
                                                                           sizeof(double3) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]));
                                                checkCudaErrors(hipMalloc(&datasets[d].fields[f].device_visibilities[i][s].Vo,
                                                                           sizeof(hipfftComplex) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]));
                                                checkCudaErrors(hipMalloc(&datasets[d].fields[f].device_visibilities[i][s].weight,
                                                                           sizeof(float) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]));
                                                checkCudaErrors(hipMalloc(&datasets[d].fields[f].device_visibilities[i][s].Vm,
                                                                           sizeof(hipfftComplex) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]));
                                                checkCudaErrors(hipMalloc(&datasets[d].fields[f].device_visibilities[i][s].Vr,
                                                                           sizeof(hipfftComplex) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]));
                                                checkCudaErrors(hipMemcpy(datasets[d].fields[f].device_visibilities[i][s].uvw, datasets[d].fields[f].visibilities[i][s].uvw.data(),
                                                                           sizeof(double3) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s],
                                                                           hipMemcpyHostToDevice));

                                                checkCudaErrors(hipMemcpy(datasets[d].fields[f].device_visibilities[i][s].weight,
                                                                           datasets[d].fields[f].visibilities[i][s].weight.data(),
                                                                           sizeof(float) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s],
                                                                           hipMemcpyHostToDevice));

                                                checkCudaErrors(hipMemcpy(datasets[d].fields[f].device_visibilities[i][s].Vo, datasets[d].fields[f].visibilities[i][s].Vo.data(),
                                                                           sizeof(hipfftComplex) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s],
                                                                           hipMemcpyHostToDevice));

                                                checkCudaErrors(hipMemset(datasets[d].fields[f].device_visibilities[i][s].Vr, 0,
                                                                           sizeof(hipfftComplex) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]));
                                                checkCudaErrors(hipMemset(datasets[d].fields[f].device_visibilities[i][s].Vm, 0,
                                                                           sizeof(hipfftComplex) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]));
                                        }
                                }

                                checkCudaErrors(hipMalloc((void**)&datasets[d].fields[f].atten_image, sizeof(float)*M*N));
                                checkCudaErrors(hipMemset(datasets[d].fields[f].atten_image, 0, sizeof(float)*M*N));

                        }else{
                                hipSetDevice(firstgpu);
                                checkCudaErrors(hipMalloc((void**)&datasets[d].fields[f].atten_image, sizeof(float)*M*N));
                                checkCudaErrors(hipMemset(datasets[d].fields[f].atten_image, 0, sizeof(float)*M*N));
                                for(int i=0; i<datasets[d].data.total_frequencies; i++) {
                                        hipSetDevice((i % num_gpus) + firstgpu);
                                        for(int s=0; s<datasets[d].data.nstokes; s++) {
                                                checkCudaErrors(hipMalloc(&datasets[d].fields[f].device_visibilities[i][s].uvw,
                                                                           sizeof(double3) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]));
                                                checkCudaErrors(hipMalloc(&datasets[d].fields[f].device_visibilities[i][s].Vo,
                                                                           sizeof(hipfftComplex) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]));
                                                checkCudaErrors(hipMalloc(&datasets[d].fields[f].device_visibilities[i][s].weight,
                                                                           sizeof(float) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]));
                                                checkCudaErrors(hipMalloc(&datasets[d].fields[f].device_visibilities[i][s].Vm,
                                                                           sizeof(hipfftComplex) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]));
                                                checkCudaErrors(hipMalloc(&datasets[d].fields[f].device_visibilities[i][s].Vr,
                                                                           sizeof(hipfftComplex) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]));
                                                checkCudaErrors(hipMemcpy(datasets[d].fields[f].device_visibilities[i][s].uvw,datasets[d].fields[f].visibilities[i][s].uvw.data(),
                                                                           sizeof(double3) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s],hipMemcpyHostToDevice));
                                                checkCudaErrors(hipMemcpy(datasets[d].fields[f].device_visibilities[i][s].weight,datasets[d].fields[f].visibilities[i][s].weight.data(),
                                                                           sizeof(float) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s],hipMemcpyHostToDevice));
                                                checkCudaErrors(hipMemcpy(datasets[d].fields[f].device_visibilities[i][s].Vo,datasets[d].fields[f].visibilities[i][s].Vo.data(),
                                                                           sizeof(hipfftComplex) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s],hipMemcpyHostToDevice));
                                                checkCudaErrors(hipMemset(datasets[d].fields[f].device_visibilities[i][s].Vr, 0, sizeof(hipfftComplex) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]));
                                                checkCudaErrors(hipMemset(datasets[d].fields[f].device_visibilities[i][s].Vm, 0,sizeof(hipfftComplex) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]));
                                        }
                                }
                        }
                }
        }


        max_number_vis = 0;
        for(int d=0; d<nMeasurementSets; d++) {
                if(datasets[d].data.max_number_visibilities_in_channel_and_stokes > max_number_vis)
                        max_number_vis = datasets[d].data.max_number_visibilities_in_channel_and_stokes;
        }

        if(max_number_vis == 0) {
                printf("Max number of visibilities cannot be zero for image synthesis\n");
                exit(-1);
        }

        this->visibilities->setMaxNumberVis(max_number_vis);

        for(int g=0; g<num_gpus; g++) {
                hipSetDevice((g%num_gpus) + firstgpu);
                checkCudaErrors(hipMalloc((void**)&vars_gpu[g].device_dchi2, sizeof(float)*M*N));
                checkCudaErrors(hipMemset(vars_gpu[g].device_dchi2, 0, sizeof(float)*M*N));

                checkCudaErrors(hipMalloc(&vars_gpu[g].device_chi2, sizeof(float)*max_number_vis));
                checkCudaErrors(hipMemset(vars_gpu[g].device_chi2, 0, sizeof(float)*max_number_vis));
        }

        //Declaring block size and number of blocks for Image
        threadsPerBlockNN.x = variables.blockSizeX;
        threadsPerBlockNN.y = variables.blockSizeY;

        numBlocksNN.x = iDivUp(M, threadsPerBlockNN.x);
        numBlocksNN.y = iDivUp(N, threadsPerBlockNN.y);

        noise_jypix = beam_noise / (PI * beam_bmaj * beam_bmin / (4 * log(2) ));

        /////////////////////////////////////////////////////CALCULATE DIRECTION COSINES/////////////////////////////////////////////////
        double raimage = ra * RPDEG_D;
        double decimage = dec * RPDEG_D;

        if(verbose_flag) {
                printf("FITS: Ra: %.16e (rad), dec: %.16e (rad)\n", raimage, decimage);
                printf("FITS: Center pix: (%lf,%lf)\n", crpix1-1, crpix2-1);
        }

        double lobs, mobs, lphs, mphs;
        double dcosines_l_pix_ref, dcosines_m_pix_ref, dcosines_l_pix_phs, dcosines_m_pix_phs;
        for(int d=0; d<nMeasurementSets; d++) {
                if(verbose_flag)
                        printf("Dataset: %s\n", datasets[d].name);
                for(int f=0; f<datasets[d].data.nfields; f++) {

                        direccos(datasets[d].fields[f].ref_ra, datasets[d].fields[f].ref_dec, raimage, decimage, &lobs,  &mobs);
                        direccos(datasets[d].fields[f].phs_ra, datasets[d].fields[f].phs_dec, raimage, decimage, &lphs,  &mphs);

                        dcosines_l_pix_ref = lobs/ -deltax; // Radians to pixels
                        dcosines_m_pix_ref = mobs/fabs(deltay); // Radians to pixels

                        dcosines_l_pix_phs = lphs/ -deltax; // Radians to pixels
                        dcosines_m_pix_phs = mphs/fabs(deltay); // Radians to pixels

                        if(verbose_flag)
                        {
                                printf("Ref: l (pix): %e, m (pix): %e\n", dcosines_l_pix_ref, dcosines_m_pix_ref);
                                printf("Phase: l (pix): %e, m (pix): %e\n", dcosines_l_pix_phs, dcosines_m_pix_phs);

                        }


                        datasets[d].fields[f].ref_xobs = (crpix1 - 1.0f) + dcosines_l_pix_ref;// + 6.0f;
                        datasets[d].fields[f].ref_yobs = (crpix2 - 1.0f) + dcosines_m_pix_ref;// - 7.0f;

                        datasets[d].fields[f].phs_xobs = (crpix1 - 1.0f) + dcosines_l_pix_phs;// + 5.0f;
                        datasets[d].fields[f].phs_yobs = (crpix2 - 1.0f) + dcosines_m_pix_phs;// - 7.0f;


                        if(verbose_flag) {
                                printf("Ref: Field %d - Ra: %.16e (rad), dec: %.16e (rad), x0: %f (pix), y0: %f (pix)\n", f, datasets[d].fields[f].ref_ra, datasets[d].fields[f].ref_dec,
                                       datasets[d].fields[f].ref_xobs, datasets[d].fields[f].ref_yobs);
                                printf("Phase: Field %d - Ra: %.16e (rad), dec: %.16e (rad), x0: %f (pix), y0: %f (pix)\n", f, datasets[d].fields[f].phs_ra, datasets[d].fields[f].phs_dec,
                                       datasets[d].fields[f].phs_xobs, datasets[d].fields[f].phs_yobs);
                        }

                        if(datasets[d].fields[f].ref_xobs < 0 || datasets[d].fields[f].ref_xobs >= M || datasets[d].fields[f].ref_xobs < 0 || datasets[d].fields[f].ref_yobs >= N) {
                                printf("Dataset: %s\n", datasets[d].name);
                                printf("Pointing reference center (%f,%f) is outside the range of the image\n", datasets[d].fields[f].ref_xobs, datasets[d].fields[f].ref_yobs);
                                goToError();
                        }

                        if(datasets[d].fields[f].phs_xobs < 0 || datasets[d].fields[f].phs_xobs >= M || datasets[d].fields[f].phs_xobs < 0 || datasets[d].fields[f].phs_yobs >= N) {
                                printf("Dataset: %s\n", datasets[d].name);
                                printf("Pointing phase center (%f,%f) is outside the range of the image\n", datasets[d].fields[f].phs_xobs, datasets[d].fields[f].phs_yobs);
                                goToError();
                        }
                }
        }
        ////////////////////////////////////////////////////////MAKE STARTING IMAGE////////////////////////////////////////////////////////

        host_I = (float*)malloc(M*N*sizeof(float)*image_count);

        for(int i=0; i<M; i++) {
                for(int j=0; j<N; j++) {
                        for(int k=0; k<image_count; k++) {
                                host_I[N*M*k+N*i+j] = initial_values[k];
                        }
                }
        }

        ////////////////////////////////////////////////CUDA MEMORY ALLOCATION FOR DEVICE///////////////////////////////////////////////////

        for(int g=0; g<num_gpus; g++) {
                hipSetDevice((g%num_gpus) + firstgpu);
                checkCudaErrors(hipMalloc((void**)&vars_gpu[g].device_V, sizeof(hipfftComplex)*M*N));
                checkCudaErrors(hipMalloc((void**)&vars_gpu[g].device_I_nu, sizeof(hipfftComplex)*M*N));
        }


        hipSetDevice(firstgpu);

        checkCudaErrors(hipMalloc((void**)&device_Image, sizeof(float)*M*N*image_count));
        checkCudaErrors(hipMemset(device_Image, 0, sizeof(float)*M*N*image_count));

        checkCudaErrors(hipMemcpy(device_Image, host_I, sizeof(float)*N*M*image_count, hipMemcpyHostToDevice));

        checkCudaErrors(hipMalloc((void**)&device_noise_image, sizeof(float)*M*N));
        checkCudaErrors(hipMemset(device_noise_image, 0, sizeof(float)*M*N));

        checkCudaErrors(hipMalloc((void**)&device_weight_image, sizeof(float)*M*N));
        checkCudaErrors(hipMemset(device_weight_image, 0, sizeof(float)*M*N));

        //checkCudaErrors(hipMalloc((void**)&device_distance_image, sizeof(float)*M*N));



        for(int g=0; g<num_gpus; g++) {
                hipSetDevice((g%num_gpus) + firstgpu);
                checkCudaErrors(hipMemset(vars_gpu[g].device_V, 0, sizeof(hipfftComplex)*M*N));
                checkCudaErrors(hipMemset(vars_gpu[g].device_I_nu, 0, sizeof(hipfftComplex)*M*N));

        }

        /////////// MAKING IMAGE OBJECT /////////////
        image = new Image(device_Image, image_count);
        imageMap *functionPtr = (imageMap*)malloc(sizeof(imageMap)*image_count);
        image->setFunctionMapping(functionPtr);

        for(int i = 0; i < image_count; i++)
        {
                if(nopositivity)
                {
                        functionPtr[i].evaluateXt = defaultEvaluateXt;
                        functionPtr[i].newP = defaultNewP;
                }else{
                        if(!i)
                        {
                                functionPtr[i].evaluateXt = particularEvaluateXt;
                                functionPtr[i].newP = particularNewP;
                        }else{
                                functionPtr[i].evaluateXt = defaultEvaluateXt;
                                functionPtr[i].newP = defaultNewP;
                        }
                }
        }


        initFFT(vars_gpu, M, N, firstgpu, num_gpus);

        //Time is taken from first kernel
        t = clock();
        start = omp_get_wtime();
        for(int d=0; d<nMeasurementSets; d++) {
                for(int f=0; f < datasets[d].data.nfields; f++) {
                        if(num_gpus == 1) {
                                hipSetDevice(selected);
                                for(int i=0; i<datasets[d].data.total_frequencies; i++) {
                                        for(int s=0; s<datasets[d].data.nstokes; s++) {
                                                hermitianSymmetry << < datasets[d].fields[f].device_visibilities[i][s].numBlocksUV,
                                                        datasets[d].fields[f].device_visibilities[i][s].threadsPerBlockUV >> >
                                                (datasets[d].fields[f].device_visibilities[i][s].uvw, datasets[d].fields[f].device_visibilities[i][s].Vo, datasets[d].fields[f].nu[i], datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]);
                                                checkCudaErrors(hipDeviceSynchronize());
                                        }
                                }

                        }else{
                            #pragma omp parallel for schedule(static,1)
                                for (int i = 0; i < datasets[d].data.total_frequencies; i++)
                                {
                                        unsigned int j = omp_get_thread_num();
                                        //unsigned int num_cpu_threads = omp_get_num_threads();
                                        // set and check the CUDA device for this CPU thread
                                        int gpu_id = -1;
                                        hipSetDevice((i%num_gpus) + firstgpu); // "% num_gpus" allows more CPU threads than GPU devices
                                        hipGetDevice(&gpu_id);
                                        for(int s=0; s<datasets[d].data.nstokes; s++) {
                                                hermitianSymmetry << < datasets[d].fields[f].device_visibilities[i][s].numBlocksUV,
                                                        datasets[d].fields[f].device_visibilities[i][s].threadsPerBlockUV >> >
                                                (datasets[d].fields[f].device_visibilities[i][s].uvw, datasets[d].fields[f].device_visibilities[i][s].Vo, datasets[d].fields[f].nu[i], datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]);
                                                checkCudaErrors(hipDeviceSynchronize());
                                        }
                                }


                        }
                }

                if(num_gpus == 1) {
                        hipSetDevice(selected);
                        for(int f=0; f<datasets[d].data.nfields; f++) {
                                for(int i=0; i<datasets[d].data.total_frequencies; i++) {
                                        if(datasets[d].fields[f].numVisibilitiesPerFreq[i] > 0) {
                                                total_attenuation<<<numBlocksNN, threadsPerBlockNN>>>(datasets[d].fields[f].atten_image, datasets[d].antennas[0].antenna_diameter, datasets[d].antennas[0].pb_factor, datasets[d].antennas[0].pb_cutoff, datasets[d].fields[f].nu[i], datasets[d].fields[f].ref_xobs, datasets[d].fields[f].ref_yobs, DELTAX, DELTAY, N, datasets[d].antennas[0].primary_beam);
                                                checkCudaErrors(hipDeviceSynchronize());
                                        }
                                }
                        }
                }else{
                        for(int f=0; f<datasets[d].data.nfields; f++) {
                                #pragma omp parallel for schedule(static,1)
                                for (int i = 0; i < datasets[d].data.total_frequencies; i++)
                                {
                                        unsigned int j = omp_get_thread_num();
                                        //unsigned int num_cpu_threads = omp_get_num_threads();
                                        // set and check the CUDA device for this CPU thread
                                        int gpu_id = -1;
                                        hipSetDevice((i%num_gpus) + firstgpu); // "% num_gpus" allows more CPU threads than GPU devices
                                        hipGetDevice(&gpu_id);
                                        if(datasets[d].fields[f].numVisibilitiesPerFreq[i] > 0) {
                                                #pragma omp critical
                                                {
                                                        total_attenuation<<<numBlocksNN, threadsPerBlockNN>>>(datasets[d].fields[f].atten_image, datasets[d].antennas[0].antenna_diameter, datasets[d].antennas[0].pb_factor, datasets[d].antennas[0].pb_cutoff, datasets[d].fields[f].nu[i], datasets[d].fields[f].ref_xobs, datasets[d].fields[f].ref_yobs, DELTAX, DELTAY, N, datasets[d].antennas[0].primary_beam);
                                                        checkCudaErrors(hipDeviceSynchronize());
                                                }
                                        }
                                }
                        }
                }

                for(int f=0; f<datasets[d].data.nfields; f++) {
                        if(datasets[d].fields[f].valid_frequencies > 0) {
                                if(num_gpus == 1) {
                                        hipSetDevice(selected);
                                        mean_attenuation<<<numBlocksNN, threadsPerBlockNN>>>(datasets[d].fields[f].atten_image, datasets[d].fields[f].valid_frequencies, N);
                                        checkCudaErrors(hipDeviceSynchronize());
                                }else{
                                        hipSetDevice(firstgpu);
                                        mean_attenuation<<<numBlocksNN, threadsPerBlockNN>>>(datasets[d].fields[f].atten_image, datasets[d].fields[f].valid_frequencies, N);
                                        checkCudaErrors(hipDeviceSynchronize());
                                }
                                if(print_images) {
                                        std::string atten_name =  "dataset_" + std::to_string(d) + "_atten";
                                        iohandler->IoPrintImageIteration(datasets[d].fields[f].atten_image, mod_in, mempath, atten_name.c_str(), "", f, 0, 1.0, M, N, true);
                                }
                        }
                }
        }



        if(num_gpus == 1) {
                hipSetDevice(selected);
        }else{
                hipSetDevice(firstgpu);
        }

        for(int d=0; d<nMeasurementSets; d++) {
                for(int f=0; f<datasets[d].data.nfields; f++) {
                        weight_image<<<numBlocksNN, threadsPerBlockNN>>>(device_weight_image, datasets[d].fields[f].atten_image, noise_jypix, N);
                        checkCudaErrors(hipDeviceSynchronize());

                        //distance_image<<<numBlocksNN, threadsPerBlockNN>>>(device_distance_image, datasets[d].fields[f].ref_xobs, datasets[d].fields[f].ref_yobs, 4.5e-05, DELTAX, DELTAY, N);
                        //checkCudaErrors(hipDeviceSynchronize());
                }
        }

        noise_image<<<numBlocksNN, threadsPerBlockNN>>>(device_noise_image, device_weight_image, noise_jypix, N);
        checkCudaErrors(hipDeviceSynchronize());
        if(print_images) {
                iohandler->IoPrintImage(device_noise_image, mod_in, mempath, "noise.fits", "", 0, 0, 1.0, M, N, true);
                //iohandler->IoPrintImage(device_distance_image, mod_in, mempath, "distance.fits", "", 0, 0, 1.0, M, N);
        }

        float *host_noise_image = (float*)malloc(M*N*sizeof(float));
        checkCudaErrors(hipMemcpy2D(host_noise_image, sizeof(float), device_noise_image, sizeof(float), sizeof(float), M*N, hipMemcpyDeviceToHost));
        float noise_min = *std::min_element(host_noise_image,host_noise_image+(M*N));

        fg_scale = noise_min;
        noise_cut = noise_cut * noise_min;
        if(verbose_flag) {
                printf("fg_scale = %e\n", fg_scale);
                printf("noise (Jy/pix) = %e\n", noise_jypix);
        }

        //checkCudaErrors(hipMemcpy2D(device_noise_image, sizeof(float), device_distance_image, sizeof(float), sizeof(float), M*N, hipMemcpyDeviceToDevice));

        free(host_noise_image);
        hipFree(device_weight_image);
        //hipFree(device_distance_image);
        for(int d=0; d<nMeasurementSets; d++) {
                for(int f=0; f<datasets[d].data.nfields; f++) {
                        hipFree(datasets[d].fields[f].atten_image);
                }
        }
};

void MFS::clearRun()
{
        for(int d=0; d<nMeasurementSets; d++) {
                for(int f=0; f<datasets[d].data.nfields; f++) {
                        if(num_gpus == 1) {
                                hipSetDevice(selected);
                                for(int i=0; i<datasets[d].data.total_frequencies; i++) {
                                        for(int s=0; s<datasets[d].data.nstokes; s++) {
                                                checkCudaErrors(hipMemset(datasets[d].fields[f].device_visibilities[i][s].Vr, 0,
                                                                           sizeof(hipfftComplex) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]));
                                                checkCudaErrors(hipMemset(datasets[d].fields[f].device_visibilities[i][s].Vm, 0,
                                                                           sizeof(hipfftComplex) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]));
                                        }
                                }

                        }else{
                                hipSetDevice(firstgpu);
                                for(int i=0; i<datasets[d].data.total_frequencies; i++) {
                                        hipSetDevice((i % num_gpus) + firstgpu);
                                        for(int s=0; s<datasets[d].data.nstokes; s++) {
                                                checkCudaErrors(hipMemset(datasets[d].fields[f].device_visibilities[i][s].Vr, 0, sizeof(hipfftComplex) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]));
                                                checkCudaErrors(hipMemset(datasets[d].fields[f].device_visibilities[i][s].Vm, 0,sizeof(hipfftComplex) * datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s]));
                                        }
                                }
                        }
                }
        }
        for(int g=0; g<num_gpus; g++) {
                hipSetDevice((g%num_gpus) + firstgpu);
                checkCudaErrors(hipMalloc((void**)&vars_gpu[g].device_V, sizeof(hipfftComplex)*M*N));
                checkCudaErrors(hipMalloc((void**)&vars_gpu[g].device_I_nu, sizeof(hipfftComplex)*M*N));
        }

        checkCudaErrors(hipMemcpy(device_Image, host_I, sizeof(float)*N*M*image_count, hipMemcpyHostToDevice));

}
void MFS::run()
{
        printf("\n\nStarting Optimizator\n");
        optimizator->getObjectiveFuntion()->setIo(iohandler);
        optimizator->getObjectiveFuntion()->setPrintImages(print_images);

        if(this->Order == NULL) {
                if(imagesChanged)
                {
                        optimizator->setImage(image);
                        optimizator->optimize();
                }else if(image_count == 2) {
                        optimizator->setImage(image);
                        optimizator->setFlag(0);
                        optimizator->optimize();
                        optimizator->setFlag(1);
                        optimizator->optimize();
                        optimizator->setFlag(2);
                        optimizator->optimize();
                        optimizator->setFlag(3);
                        optimizator->optimize();
                }
        }else{
                (this->Order)(optimizator, image);
        }

        t = clock() - t;
        end = omp_get_wtime();
        printf("Minimization ended successfully\n\n");
        printf("Iterations: %d\n", iter);
        printf("chi2: %f\n", final_chi2);
        printf("0.5*chi2: %f\n", 0.5*final_chi2);
        printf("Total visibilities: %d\n", total_visibilities);
        printf("Reduced-chi2 (Num visibilities): %f\n", (0.5*final_chi2)/total_visibilities);
        printf("Reduced-chi2 (Weights sum): %f\n", (0.5*final_chi2)/sum_weights);
        printf("S: %f\n", final_S);
        if(reg_term != 1) {
                printf("Normalized S: %f\n", final_S/(M*N));
        }else{
                printf("Normalized S: %f\n", final_S/(M*M*N*N));
        }
        printf("lambda*S: %f\n\n", lambda*final_S);
        double time_taken = ((double)t)/CLOCKS_PER_SEC;
        double wall_time = end-start;
        printf("Total CPU time: %lf\n", time_taken);
        printf("Wall time: %lf\n\n\n", wall_time);

        if(strcmp(variables.ofile,"NULL") != 0) {
                FILE *outfile = fopen(variables.ofile, "w");
                if (outfile == NULL)
                {
                        printf("Error opening output file!\n");
                        goToError();
                }

                fprintf(outfile, "Iterations: %d\n", iter);
                fprintf(outfile, "chi2: %f\n", final_chi2);
                fprintf(outfile, "0.5*chi2: %f\n", 0.5*final_chi2);
                fprintf(outfile, "Total visibilities: %d\n", total_visibilities);
                fprintf(outfile, "Reduced-chi2 (Num visibilities): %f\n", (0.5*final_chi2)/total_visibilities);
                fprintf(outfile, "Reduced-chi2 (Weights sum): %f\n", (0.5*final_chi2)/sum_weights);
                fprintf(outfile, "S: %f\n", final_S);
                if(reg_term != 1) {
                        fprintf(outfile, "Normalized S: %f\n", final_S/(M*N));
                }else{
                        fprintf(outfile, "Normalized S: %f\n", final_S/(M*M*N*N));
                }
                fprintf(outfile, "lambda*S: %f\n", lambda*final_S);
                fprintf(outfile, "Wall time: %lf", wall_time);
                fclose(outfile);
        }
        //Pass residuals to host
        printf("Saving final image to disk\n");
        if(IoOrderEnd == NULL) {
                iohandler->IoPrintImage(image->getImage(), mod_in, "", out_image, "JY/PIXEL", iter, 0, fg_scale, M, N, true);
                iohandler->IoPrintImage(image->getImage(), mod_in, "", "alpha.fits", "", iter, 1, 1.0, M, N, true);
        }else{
                (IoOrderEnd)(image->getImage(), iohandler);
        }

        if(print_errors) /* flag for print error image */
        {
                if(this->error == NULL)
                {
                        this->error = Singleton<ErrorFactory>::Instance().CreateError(0);
                }
                /* code to calculate error */
                /* make void * params */
                printf("Calculating Error Images\n");
                this->error->calculateErrorImage(this->image, this->visibilities);
                if(IoOrderError == NULL) {
                        iohandler->IoPrintImage(image->getErrorImage(), mod_in, "", "error_Inu_0.fits", "JY/PIXEL", iter, 0, 1.0, M, N, true);
                        iohandler->IoPrintImage(image->getErrorImage(), mod_in, "", "error_alpha.fits", "", iter, 1, 1.0, M, N, true);
                }else{
                        (IoOrderError)(image->getErrorImage(), iohandler);
                }

        }

        printf("Transferring residuals to host memory\n");
        if(!gridding)
        {
                //Saving residuals to disk
                for(int d=0; d<nMeasurementSets; d++) {
                        residualsToHost(datasets[d].fields, datasets[d].data, num_gpus, firstgpu);
                }
        }else{
                double deltax = RPDEG_D*DELTAX; //radians
                double deltay = RPDEG_D*DELTAY; //radians
                deltau = 1.0 / (M * deltax);
                deltav = 1.0 / (N * deltay);

                printf("Visibilities are gridded, we will need to de-grid to save them in a Measurement Set File\n");
                omp_set_num_threads(num_gpus);
                for(int d=0; d<nMeasurementSets; d++)
                        degridding(datasets[d].fields, datasets[d].data, deltau, deltav, num_gpus, firstgpu, variables.blockSizeV, M, N);

                for(int d=0; d<nMeasurementSets; d++)
                        residualsToHost(datasets[d].fields, datasets[d].data, num_gpus, firstgpu);

        }

        printf("Saving residuals and model to MS...\n");
        for(int d=0; d<nMeasurementSets; d++) {
                if(!save_model_input) {
                        iohandler->IocopyMS(datasets[d].name, datasets[d].oname);
                        iohandler->IowriteMS(datasets[d].oname, "DATA", datasets[d].fields, datasets[d].data, random_probability, false, false, false, verbose_flag);
                        iohandler->IowriteMS(datasets[d].oname, "MODEL", datasets[d].fields, datasets[d].data, random_probability, true, false, false, verbose_flag);
                }else
                        iohandler->IowriteMS(datasets[d].name, "MODEL", datasets[d].fields, datasets[d].data, random_probability, true, false, false, verbose_flag);

        }

        printf("Residuals and model saved.\n");


};

void MFS::unSetDevice()
{
        //Free device and host memory
        printf("Freeing device memory\n");
        if(num_gpus == 1) {
                hipSetDevice(selected);
        }else{
                hipSetDevice(firstgpu);
        }

        for(int d=0; d<nMeasurementSets; d++) {
                for(int f=0; f<datasets[d].data.nfields; f++) {
                        for(int i=0; i<datasets[d].data.total_frequencies; i++) {

                                if(num_gpus > 1) {
                                        hipSetDevice((i%num_gpus) + firstgpu);
                                }
                                for(int s=0; s<datasets[d].data.nstokes; s++) {
                                        hipFree(datasets[d].fields[f].device_visibilities[i][s].uvw);
                                        hipFree(datasets[d].fields[f].device_visibilities[i][s].weight);
                                        hipFree(datasets[d].fields[f].device_visibilities[i][s].Vr);
                                        hipFree(datasets[d].fields[f].device_visibilities[i][s].Vm);
                                        hipFree(datasets[d].fields[f].device_visibilities[i][s].Vo);
                                }

                        }
                }
        }

        printf("Freeing cuFFT plans\n");
        for(int g=0; g<num_gpus; g++) {
                hipSetDevice((g%num_gpus) + firstgpu);
                hipfftDestroy(vars_gpu[g].plan);
        }

        printf("Freeing host memory\n");
        for(int d=0; d<nMeasurementSets; d++) {
                for(int f=0; f<datasets[d].data.nfields; f++) {
                        for(int i=0; i<datasets[d].data.total_frequencies; i++) {
                                for(int s=0; s<datasets[d].data.nstokes; s++) {
                                        if (datasets[d].fields[f].numVisibilitiesPerFreqPerStoke[i][s] > 0) {
                                                datasets[d].fields[f].visibilities[i][s].uvw.clear();
                                                datasets[d].fields[f].visibilities[i][s].weight.clear();
                                                datasets[d].fields[f].visibilities[i][s].Vo.clear();
                                                datasets[d].fields[f].visibilities[i][s].Vm.clear();

                                                if(gridding) {
                                                        datasets[d].fields[f].backup_visibilities[i][s].uvw.clear();
                                                        datasets[d].fields[f].backup_visibilities[i][s].weight.clear();
                                                        datasets[d].fields[f].backup_visibilities[i][s].Vo.clear();
                                                }
                                        }
                                }
                        }
                }
        }



        hipFree(device_Image);

        for(int g=0; g<num_gpus; g++) {
                hipSetDevice((g%num_gpus) + firstgpu);
                hipFree(vars_gpu[g].device_V);
                hipFree(vars_gpu[g].device_I_nu);
        }


        hipSetDevice(firstgpu);


        hipFree(device_noise_image);

        hipFree(device_dphi);
        hipFree(device_dchi2_total);
        hipFree(device_dS);

        hipFree(device_S);

        //Disabling UVA
        if(num_gpus > 1) {
                for(int i=firstgpu+1; i<num_gpus+firstgpu; i++) {
                        hipSetDevice(firstgpu);
                        hipDeviceDisablePeerAccess(i);
                        hipSetDevice(i);
                        hipDeviceDisablePeerAccess(firstgpu);
                }

                for(int i=0; i<num_gpus; i++ ) {
                        hipSetDevice((i%num_gpus) + firstgpu);
                        hipDeviceReset();
                }
        }
        free(host_I);
        free(msinput);
        free(msoutput);
        free(modinput);

        for(int i=0; i< nMeasurementSets; i++) {
                free(datasets[i].name);
                free(datasets[i].oname);
        }

        iohandler->IocloseCanvas(mod_in);
};

namespace {
Synthesizer* CreateMFS()
{
        return new MFS;
}
const int MFSID = 0;
const bool RegisteredMFS = Singleton<SynthesizerFactory>::Instance().RegisterSynthesizer(MFSID, CreateMFS);
};
