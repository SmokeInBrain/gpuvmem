#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------
   Copyright (C) 2016-2017  Miguel Carcamo, Pablo Roman, Simon Casassus,
   Victor Moral, Fernando Rannou - miguel.carcamo@usach.cl

   This program includes Numerical Recipes (NR) based routines whose
   copyright is held by the NR authors. If NR routines are included,
   you are required to comply with the licensing set forth there.

   Part of the program also relies on an an ANSI C library for multi-stream
   random number generation from the related Prentice-Hall textbook
   Discrete-Event Simulation: A First Course by Steve Park and Larry Leemis,
   for more information please contact leemis@math.wm.edu

   For the original parts of this code, the following license applies:

   This program is free software: you can redistribute it and/or modify
   it under the terms of the GNU General Public License as published by
   the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.

   This program is distributed in the hope that it will be useful,
   but WITHOUT ANY WARRANTY; without even the implied warranty of
   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU General Public License for more details.

   You should have received a copy of the GNU General Public License
   along with this program. If not, see <http://www.gnu.org/licenses/>.
 * -------------------------------------------------------------------------
 */
#include "functions.cuh"


extern long M, N, numVisibilities;
extern int iterations, iterthreadsVectorNN, blocksVectorNN, nopositivity, \
           status_mod_in, verbose_flag, apply_noise, adaptive, clip_flag, num_gpus, selected, iter, t_telescope, multigpu, firstgpu, reg_term, print_images, checkpoint, spec_idx, use_mask;

extern hipfftHandle plan1GPU;
extern hipfftComplex *device_V, *device_Inu;

extern float2 *device_dphi, *device_2I;
extern float *device_mask, *device_chi2, *device_dchi2, *device_S, *device_S_alpha, *device_dS, *device_dS_alpha, *device_noise_image;
extern float noise_jypix, fg_scale, noise_cut, MINPIX, \
             minpix, lambda, ftol, random_probability, final_chi2, nu_0, final_H, alpha_start, eta, epsilon;

extern dim3 threadsPerBlockNN, numBlocksNN;

extern float beam_noise, beam_bmaj, beam_bmin, beam_bpa, b_noise_aux, antenna_diameter, pb_factor, pb_cutoff, *host_noise_image;
extern double ra, dec, crpix1, crpix2, DELTAX, DELTAY, deltau, deltav;

extern freqData data;

extern char* mempath, *out_image;

extern fitsfile *mod_in;

extern Field *fields;

extern int flag_opt, valid_pixels;

extern int2 *pixels;

extern VariablesPerField *vars_per_field;

extern varsPerGPU *vars_gpu;

extern char *checkp;

extern double2 *host_M_k;
extern double2 *host_Q_k;

extern float2 *temp;
extern double2 *M_k_out, *Q_k_out;

extern FILE *outfile;
extern FILE *outfile_its;
extern FILE *shutdown_file;

extern int position_in_file;

extern int accepted_afterburndown;

extern int real_iterations;

char *shutdown_file_name;

void sig_handler(int signo)
{
        if(signo == SIGKILL || signo == SIGTERM || signo == SIGINT) {
                shutdown_file= fopen(shutdown_file_name, "w");
                printf("--------------Iteration %d-----------\n", real_iterations);
                fseek(shutdown_file,position_in_file,SEEK_SET);
                fprintf(shutdown_file, "Iterations: %d\n", real_iterations);
                fprintf(shutdown_file, "Accepted after burndown: %d\n", accepted_afterburndown);
                fflush(shutdown_file);
                double2toImage(M_k_out, mod_in, out_image, checkp, 0, M, N, 1.0, accepted_afterburndown, 1);
                double2toImage(Q_k_out, mod_in, out_image, checkp, 1, M, N, 1/(accepted_afterburndown-1), accepted_afterburndown, 1);
                float2toImage(device_2I, mod_in, out_image, checkp, 2, M, N, 1.0, 1);
                fclose(outfile);
                fclose(outfile_its);
                fclose(shutdown_file);
                hipFree(device_2I);
                hipFree(Q_k_out);
                hipFree(M_k_out);
                exit(0);
        }
}

__host__ void goToError()
{
        if(num_gpus > 1) {
                for(int i=firstgpu+1; i<firstgpu + num_gpus; i++) {
                        hipSetDevice(firstgpu);
                        hipDeviceDisablePeerAccess(i);
                        hipSetDevice(i);
                        hipDeviceDisablePeerAccess(firstgpu);
                }

                for(int i=0; i<num_gpus; i++ ) {
                        hipSetDevice((i%num_gpus) + firstgpu);
                        hipDeviceReset();
                }
        }

        printf("An error has ocurred, exiting\n");
        exit(0);

}

__host__ void init_beam(int telescope)
{
        switch(telescope) {
        case 1:
                antenna_diameter = 1.4; /* CBI2 Antenna Diameter */
                pb_factor = 1.22; /* FWHM Factor */
                pb_cutoff = 90.0*RPARCM; /* radians */
                break;
        case 2:
                antenna_diameter = 12.0; /* ALMA Antenna Diameter */
                pb_factor = 1.13; /* FWHM Factor */
                pb_cutoff = 1.0*RPARCM; /* radians */
                break;
        case 3:
                antenna_diameter = 22.0; /* ATCA Antenna Diameter */
                pb_factor = 1.22; /* FWHM Factor */
                pb_cutoff = 1.0*RPARCM; /* radians */
                break;
        case 4:
                antenna_diameter = 25.0; /* VLA Antenna Diameter */
                pb_factor = 1.22; /* FWHM Factor */
                pb_cutoff = 20.0*RPARCM; /* radians */
                break;
        case 5:
                antenna_diameter = 3.5; /* SZA Antenna Diameter */
                pb_factor = 1.22; /* FWHM Factor */
                pb_cutoff = 20.0*RPARCM; /* radians */
                break;
        case 6:
                antenna_diameter = 0.9; /* CBI Antenna Diameter */
                pb_factor = 1.22; /* FWHM Factor */
                pb_cutoff = 20.0*RPARCM; /* radians */
                break;
        default:
                printf("Telescope type not defined\n");
                goToError();
                break;
        }
}


__host__ long NearestPowerOf2(long x)
{
        --x;
        x |= x >> 1;
        x |= x >> 2;
        x |= x >> 4;
        x |= x >> 8;
        x |= x >> 16;
        return ++x;
}


bool isPow2(unsigned int x)
{
        return ((x&(x-1))==0);
}


__host__ void readInputDat(char *file)
{
        FILE *fp;
        char item[50];
        float status;
        if((fp = fopen(file, "r")) == NULL) {
                printf("ERROR. The input file wasn't provided by the user.\n");
                goToError();
        }else{
                while(true) {
                        int ret = fscanf(fp, "%s %e", item, &status);

                        if(ret==EOF) {
                                break;
                        }else{
                                if (strcmp(item,"lambda_entropy")==0) {
                                        if(lambda == -1) {
                                                lambda = status;
                                        }
                                }else if (strcmp(item,"noise_cut")==0) {
                                        if(noise_cut == -1) {
                                                noise_cut = status;
                                        }
                                }else if (strcmp(item,"t_telescope")==0) {
                                        t_telescope = status;
                                }else if(strcmp(item,"minpix")==0) {
                                        if(minpix == -1) {
                                                minpix = status;
                                        }
                                } else if(strcmp(item,"ftol")==0) {
                                        ftol = status;
                                } else if(strcmp(item,"random_probability")==0) {
                                        if(random_probability == -1) {
                                                random_probability = status;
                                        }
                                }else{
                                        printf("Keyword not defined in input\n");
                                        goToError();
                                }
                        }
                }
        }
}

__host__ void print_warranty() {
        printf("THERE IS NO WARRANTY FOR THE PROGRAM, TO THE EXTENT PERMITTED BY \
APPLICABLE LAW.  EXCEPT WHEN OTHERWISE STATED IN WRITING THE COPYRIGHT \
HOLDERS AND/OR OTHER PARTIES PROVIDE THE PROGRAM 'AS IS' WITHOUT WARRANTY \
OF ANY KIND, EITHER EXPRESSED OR IMPLIED, INCLUDING, BUT NOT LIMITED TO, \
THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR \
PURPOSE.  THE ENTIRE RISK AS TO THE QUALITY AND PERFORMANCE OF THE PROGRAM \
IS WITH YOU.  SHOULD THE PROGRAM PROVE DEFECTIVE, YOU ASSUME THE COST OF \
ALL NECESSARY SERVICING, REPAIR OR CORRECTION.\n");
}

__host__ void print_copyright() {
        printf("   TERMS AND CONDITIONS \n"
               " \n"
               "0. Definitions. \n"
               " \n"
               "'This License' refers to version 3 of the GNU General Public License. \n"
               " \n"
               "'Copyright' also means copyright-like laws that apply to other kinds of \n"
               "works, such as semiconductor masks. \n"
               " \n"
               "'The Program' refers to any copyrightable work licensed under this \n"
               "License.  Each licensee is addressed as 'you'.  'Licensees' and \n"
               "'recipients' may be individuals or organizations. \n"
               " \n"
               "To 'modify' a work means to copy from or adapt all or part of the work \n"
               "in a fashion requiring copyright permission, other than the making of an \n"
               "exact copy.  The resulting work is called a 'modified version' of the \n"
               "earlier work or a work 'based on' the earlier work. \n"
               " \n"
               "A 'covered work' means either the unmodified Program or a work based \n"
               "on the Program. \n"
               " \n"
               "To 'propagate' a work means to do anything with it that, without \n"
               "permission, would make you directly or secondarily liable for \n"
               "infringement under applicable copyright law, except executing it on a \n"
               "computer or modifying a private copy.  Propagation includes copying, \n"
               "distribution (with or without modification), making available to the \n"
               "public, and in some countries other activities as well. \n"
               " \n"
               "To 'convey' a work means any kind of propagation that enables other \n"
               "parties to make or receive copies.  Mere interaction with a user through \n"
               "a computer network, with no transfer of a copy, is not conveying. \n"
               " \n"
               "An interactive user interface displays 'Appropriate Legal Notices' \n"
               "to the extent that it includes a convenient and prominently visible \n"
               "feature that (1) displays an appropriate copyright notice, and (2) \n"
               "tells the user that there is no warranty for the work (except to the \n"
               "extent that warranties are provided), that licensees may convey the \n"
               "work under this License, and how to view a copy of this License.  If \n"
               "the interface presents a list of user commands or options, such as a \n"
               "menu, a prominent item in the list meets this criterion. \n"
               " \n"
               "1. Source Code. \n"
               " \n"
               "The 'source code' for a work means the preferred form of the work \n"
               "for making modifications to it.  'Object code' means any non-source \n"
               "form of a work. \n"
               " \n"
               "A 'Standard Interface' means an interface that either is an official \n"
               "standard defined by a recognized standards body, or, in the case of \n"
               "interfaces specified for a particular programming language, one that \n"
               "is widely used among developers working in that language. \n"
               " \n"
               "The 'System Libraries' of an executable work include anything, other \n"
               "than the work as a whole, that (a) is included in the normal form of \n"
               "packaging a Major Component, but which is not part of that Major \n"
               "Component, and (b) serves only to enable use of the work with that \n"
               "Major Component, or to implement a Standard Interface for which an \n"
               "implementation is available to the public in source code form.  A \n"
               "'Major Component', in this context, means a major essential component \n"
               "(kernel, window system, and so on) of the specific operating system \n"
               "(if any) on which the executable work runs, or a compiler used to \n"
               "produce the work, or an object code interpreter used to run it. \n"
               " \n"
               "The 'Corresponding Source' for a work in object code form means all \n"
               "the source code needed to generate, install, and (for an executable \n"
               "work) run the object code and to modify the work, including scripts to \n"
               "control those activities.  However, it does not include the work's \n"
               "System Libraries, or general-purpose tools or generally available free \n"
               "programs which are used unmodified in performing those activities but \n"
               "which are not part of the work.  For example, Corresponding Source \n"
               "includes interface definition files associated with source files for \n"
               "the work, and the source code for shared libraries and dynamically \n"
               "linked subprograms that the work is specifically designed to require, \n"
               "such as by intimate data communication or control flow between those \n"
               "subprograms and other parts of the work. \n"
               " \n"
               "The Corresponding Source need not include anything that users \n"
               "can regenerate automatically from other parts of the Corresponding \n"
               "Source. \n"
               " \n"
               "The Corresponding Source for a work in source code form is that \n"
               "same work. \n"
               " \n"
               "2. Basic Permissions. \n"
               " \n"
               "All rights granted under this License are granted for the term of \n"
               "copyright on the Program, and are irrevocable provided the stated \n"
               "conditions are met.  This License explicitly affirms your unlimited \n"
               "permission to run the unmodified Program.  The output from running a \n"
               "covered work is covered by this License only if the output, given its \n"
               "content, constitutes a covered work.  This License acknowledges your \n"
               "rights of fair use or other equivalent, as provided by copyright law. \n"
               " \n"
               "You may make, run and propagate covered works that you do not \n"
               "convey, without conditions so long as your license otherwise remains \n"
               "in force.  You may convey covered works to others for the sole purpose \n"
               "of having them make modifications exclusively for you, or provide you \n"
               "with facilities for running those works, provided that you comply with \n"
               "the terms of this License in conveying all material for which you do \n"
               "not control copyright.  Those thus making or running the covered works \n"
               "for you must do so exclusively on your behalf, under your direction \n"
               "and control, on terms that prohibit them from making any copies of \n"
               "your copyrighted material outside their relationship with you. \n"
               " \n"
               "Conveying under any other circumstances is permitted solely under \n"
               "the conditions stated below.  Sublicensing is not allowed; section 10 \n"
               "makes it unnecessary. \n"
               " \n"
               "3. Protecting Users' Legal Rights From Anti-Circumvention Law. \n"
               " \n"
               "No covered work shall be deemed part of an effective technological \n"
               "measure under any applicable law fulfilling obligations under article \n"
               "11 of the WIPO copyright treaty adopted on 20 December 1996, or \n"
               "similar laws prohibiting or restricting circumvention of such \n"
               "measures. \n"
               " \n"
               "When you convey a covered work, you waive any legal power to forbid \n"
               "circumvention of technological measures to the extent such circumvention \n"
               "is effected by exercising rights under this License with respect to \n"
               "the covered work, and you disclaim any intention to limit operation or \n"
               "modification of the work as a means of enforcing, against the work's \n"
               "users, your or third parties' legal rights to forbid circumvention of \n"
               "technological measures. \n"
               " \n"
               "4. Conveying Verbatim Copies. \n"
               " \n"
               "You may convey verbatim copies of the Program's source code as you \n"
               "receive it, in any medium, provided that you conspicuously and \n"
               "appropriately publish on each copy an appropriate copyright notice; \n"
               "keep intact all notices stating that this License and any \n"
               "non-permissive terms added in accord with section 7 apply to the code; \n"
               "keep intact all notices of the absence of any warranty; and give all \n"
               "recipients a copy of this License along with the Program. \n"
               " \n"
               "You may charge any price or no price for each copy that you convey, \n"
               "and you may offer support or warranty protection for a fee. \n"
               " \n"
               "5. Conveying Modified Source Versions. \n"
               " \n"
               "You may convey a work based on the Program, or the modifications to \n"
               "produce it from the Program, in the form of source code under the \n"
               "terms of section 4, provided that you also meet all of these conditions: \n"
               " \n"
               "a) The work must carry prominent notices stating that you modified \n"
               "it, and giving a relevant date. \n"
               " \n"
               "b) The work must carry prominent notices stating that it is \n"
               "released under this License and any conditions added under section \n"
               "7.  This requirement modifies the requirement in section 4 to \n"
               "'keep intact all notices'. \n"
               " \n"
               "c) You must license the entire work, as a whole, under this \n"
               "License to anyone who comes into possession of a copy.  This \n"
               "License will therefore apply, along with any applicable section 7 \n"
               "additional terms, to the whole of the work, and all its parts, \n"
               "regardless of how they are packaged.  This License gives no \n"
               "permission to license the work in any other way, but it does not \n"
               "invalidate such permission if you have separately received it. \n"
               " \n"
               "d) If the work has interactive user interfaces, each must display \n"
               "Appropriate Legal Notices; however, if the Program has interactive \n"
               "interfaces that do not display Appropriate Legal Notices, your \n"
               "work need not make them do so. \n"
               " \n"
               "A compilation of a covered work with other separate and independent \n"
               "works, which are not by their nature extensions of the covered work, \n"
               "and which are not combined with it such as to form a larger program, \n"
               "in or on a volume of a storage or distribution medium, is called an \n"
               "'aggregate' if the compilation and its resulting copyright are not \n"
               "used to limit the access or legal rights of the compilation's users \n"
               "beyond what the individual works permit.  Inclusion of a covered work \n"
               "in an aggregate does not cause this License to apply to the other \n"
               "parts of the aggregate. \n"
               " \n"
               "6. Conveying Non-Source Forms. \n"
               " \n"
               "You may convey a covered work in object code form under the terms \n"
               "of sections 4 and 5, provided that you also convey the \n"
               "machine-readable Corresponding Source under the terms of this License, \n"
               "in one of these ways: \n"
               " \n"
               "a) Convey the object code in, or embodied in, a physical product \n"
               "(including a physical distribution medium), accompanied by the \n"
               "Corresponding Source fixed on a durable physical medium \n"
               "customarily used for software interchange. \n"
               " \n"
               "b) Convey the object code in, or embodied in, a physical product \n"
               "(including a physical distribution medium), accompanied by a \n"
               "written offer, valid for at least three years and valid for as \n"
               "long as you offer spare parts or customer support for that product \n"
               "model, to give anyone who possesses the object code either (1) a \n"
               "copy of the Corresponding Source for all the software in the \n"
               "product that is covered by this License, on a durable physical \n"
               "medium customarily used for software interchange, for a price no \n"
               "more than your reasonable cost of physically performing this \n"
               "conveying of source, or (2) access to copy the \n"
               "Corresponding Source from a network server at no charge. \n"
               " \n"
               "c) Convey individual copies of the object code with a copy of the \n"
               "written offer to provide the Corresponding Source.  This \n"
               "alternative is allowed only occasionally and noncommercially, and \n"
               "only if you received the object code with such an offer, in accord \n"
               "with subsection 6b. \n"
               " \n"
               "d) Convey the object code by offering access from a designated \n"
               "place (gratis or for a charge), and offer equivalent access to the \n"
               "Corresponding Source in the same way through the same place at no \n"
               "further charge.  You need not require recipients to copy the \n"
               "Corresponding Source along with the object code.  If the place to \n"
               "copy the object code is a network server, the Corresponding Source \n"
               "may be on a different server (operated by you or a third party) \n"
               "that supports equivalent copying facilities, provided you maintain \n"
               "clear directions next to the object code saying where to find the \n"
               "Corresponding Source.  Regardless of what server hosts the \n"
               "Corresponding Source, you remain obligated to ensure that it is \n"
               "available for as long as needed to satisfy these requirements. \n"
               " \n"
               "e) Convey the object code using peer-to-peer transmission, provided \n"
               "you inform other peers where the object code and Corresponding \n"
               "Source of the work are being offered to the general public at no \n"
               "charge under subsection 6d. \n"
               " \n"
               "A separable portion of the object code, whose source code is excluded \n"
               "from the Corresponding Source as a System Library, need not be \n"
               "included in conveying the object code work. \n"
               " \n"
               "A 'User Product' is either (1) a 'consumer product', which means any \n"
               "tangible personal property which is normally used for personal, family, \n"
               "or household purposes, or (2) anything designed or sold for incorporation \n"
               "into a dwelling.  In determining whether a product is a consumer product, \n"
               "doubtful cases shall be resolved in favor of coverage.  For a particular \n"
               "product received by a particular user, 'normally used' refers to a \n"
               "typical or common use of that class of product, regardless of the status \n"
               "of the particular user or of the way in which the particular user \n"
               "actually uses, or expects or is expected to use, the product.  A product \n"
               "is a consumer product regardless of whether the product has substantial \n"
               "commercial, industrial or non-consumer uses, unless such uses represent \n"
               "the only significant mode of use of the product. \n"
               " \n"
               "'Installation Information' for a User Product means any methods, \n"
               "procedures, authorization keys, or other information required to install \n"
               "and execute modified versions of a covered work in that User Product from \n"
               "a modified version of its Corresponding Source.  The information must \n"
               "suffice to ensure that the continued functioning of the modified object \n"
               "code is in no case prevented or interfered with solely because \n"
               "modification has been made. \n"
               " \n"
               "If you convey an object code work under this section in, or with, or \n"
               "specifically for use in, a User Product, and the conveying occurs as \n"
               "part of a transaction in which the right of possession and use of the \n"
               "User Product is transferred to the recipient in perpetuity or for a \n"
               "fixed term (regardless of how the transaction is characterized), the \n"
               "Corresponding Source conveyed under this section must be accompanied \n"
               "by the Installation Information.  But this requirement does not apply \n"
               "if neither you nor any third party retains the ability to install \n"
               "modified object code on the User Product (for example, the work has \n"
               "been installed in ROM). \n"
               " \n"
               "The requirement to provide Installation Information does not include a \n"
               "requirement to continue to provide support service, warranty, or updates \n"
               "for a work that has been modified or installed by the recipient, or for \n"
               "the User Product in which it has been modified or installed.  Access to a \n"
               "network may be denied when the modification itself materially and \n"
               "adversely affects the operation of the network or violates the rules and \n"
               "protocols for communication across the network. \n"
               " \n"
               "Corresponding Source conveyed, and Installation Information provided, \n"
               "in accord with this section must be in a format that is publicly \n"
               "documented (and with an implementation available to the public in \n"
               "source code form), and must require no special password or key for \n"
               "unpacking, reading or copying. \n"
               " \n"
               "7. Additional Terms. \n"
               " \n"
               "'Additional permissions' are terms that supplement the terms of this \n"
               "License by making exceptions from one or more of its conditions. \n"
               "Additional permissions that are applicable to the entire Program shall \n"
               "be treated as though they were included in this License, to the extent \n"
               "that they are valid under applicable law.  If additional permissions \n"
               "apply only to part of the Program, that part may be used separately \n"
               "under those permissions, but the entire Program remains governed by \n"
               "this License without regard to the additional permissions. \n"
               " \n"
               "When you convey a copy of a covered work, you may at your option \n"
               "remove any additional permissions from that copy, or from any part of \n"
               "it.  (Additional permissions may be written to require their own \n"
               "removal in certain cases when you modify the work.)  You may place \n"
               "additional permissions on material, added by you to a covered work, \n"
               "for which you have or can give appropriate copyright permission. \n"
               " \n"
               "Notwithstanding any other provision of this License, for material you \n"
               "add to a covered work, you may (if authorized by the copyright holders of \n"
               "that material) supplement the terms of this License with terms: \n"
               " \n"
               "a) Disclaiming warranty or limiting liability differently from the \n"
               "terms of sections 15 and 16 of this License; or \n"
               " \n"
               "b) Requiring preservation of specified reasonable legal notices or \n"
               "author attributions in that material or in the Appropriate Legal \n"
               "Notices displayed by works containing it; or \n"
               " \n"
               "c) Prohibiting misrepresentation of the origin of that material, or \n"
               "requiring that modified versions of such material be marked in \n"
               "reasonable ways as different from the original version; or \n"
               " \n"
               "d) Limiting the use for publicity purposes of names of licensors or \n"
               "authors of the material; or \n"
               " \n"
               "e) Declining to grant rights under trademark law for use of some \n"
               "trade names, trademarks, or service marks; or \n"
               " \n"
               "f) Requiring indemnification of licensors and authors of that \n"
               "material by anyone who conveys the material (or modified versions of \n"
               "it) with contractual assumptions of liability to the recipient, for \n"
               "any liability that these contractual assumptions directly impose on \n"
               "those licensors and authors. \n"
               " \n"
               "All other non-permissive additional terms are considered 'further \n"
               "restrictions' within the meaning of section 10.  If the Program as you \n"
               "received it, or any part of it, contains a notice stating that it is \n"
               "governed by this License along with a term that is a further \n"
               "restriction, you may remove that term.  If a license document contains \n"
               "a further restriction but permits relicensing or conveying under this \n"
               "License, you may add to a covered work material governed by the terms \n"
               "of that license document, provided that the further restriction does \n"
               "not survive such relicensing or conveying. \n"
               " \n"
               "If you add terms to a covered work in accord with this section, you \n"
               "must place, in the relevant source files, a statement of the \n"
               "additional terms that apply to those files, or a notice indicating \n"
               "where to find the applicable terms. \n"
               " \n"
               "Additional terms, permissive or non-permissive, may be stated in the \n"
               "form of a separately written license, or stated as exceptions; \n"
               "the above requirements apply either way. \n"
               " \n"
               "8. Termination. \n"
               " \n"
               "You may not propagate or modify a covered work except as expressly \n"
               "provided under this License.  Any attempt otherwise to propagate or \n"
               "modify it is void, and will automatically terminate your rights under \n"
               "this License (including any patent licenses granted under the third \n"
               "paragraph of section 11). \n"
               " \n"
               "However, if you cease all violation of this License, then your \n"
               "license from a particular copyright holder is reinstated (a) \n"
               "provisionally, unless and until the copyright holder explicitly and \n"
               "finally terminates your license, and (b) permanently, if the copyright \n"
               "holder fails to notify you of the violation by some reasonable means \n"
               "prior to 60 days after the cessation. \n"
               " \n"
               "Moreover, your license from a particular copyright holder is \n"
               "reinstated permanently if the copyright holder notifies you of the \n"
               "violation by some reasonable means, this is the first time you have \n"
               "received notice of violation of this License (for any work) from that \n"
               "copyright holder, and you cure the violation prior to 30 days after \n"
               "your receipt of the notice. \n"
               " \n"
               "Termination of your rights under this section does not terminate the \n"
               "licenses of parties who have received copies or rights from you under \n"
               "this License.  If your rights have been terminated and not permanently \n"
               "reinstated, you do not qualify to receive new licenses for the same \n"
               "material under section 10. \n"
               " \n"
               "9. Acceptance Not Required for Having Copies. \n"
               " \n"
               "You are not required to accept this License in order to receive or \n"
               "run a copy of the Program.  Ancillary propagation of a covered work \n"
               "occurring solely as a consequence of using peer-to-peer transmission \n"
               "to receive a copy likewise does not require acceptance.  However, \n"
               "nothing other than this License grants you permission to propagate or \n"
               "modify any covered work.  These actions infringe copyright if you do \n"
               "not accept this License.  Therefore, by modifying or propagating a \n"
               "covered work, you indicate your acceptance of this License to do so. \n"
               " \n"
               "10. Automatic Licensing of Downstream Recipients. \n"
               " \n"
               "Each time you convey a covered work, the recipient automatically \n"
               "receives a license from the original licensors, to run, modify and \n"
               "propagate that work, subject to this License.  You are not responsible \n"
               "for enforcing compliance by third parties with this License. \n"
               " \n"
               "An 'entity transaction' is a transaction transferring control of an \n"
               "organization, or substantially all assets of one, or subdividing an \n"
               "organization, or merging organizations.  If propagation of a covered \n"
               "work results from an entity transaction, each party to that \n"
               "transaction who receives a copy of the work also receives whatever \n"
               "licenses to the work the party's predecessor in interest had or could \n"
               "give under the previous paragraph, plus a right to possession of the \n"
               "Corresponding Source of the work from the predecessor in interest, if \n"
               "the predecessor has it or can get it with reasonable efforts. \n"
               " \n"
               "You may not impose any further restrictions on the exercise of the \n"
               "rights granted or affirmed under this License.  For example, you may \n"
               "not impose a license fee, royalty, or other charge for exercise of \n"
               "rights granted under this License, and you may not initiate litigation \n"
               "(including a cross-claim or counterclaim in a lawsuit) alleging that \n"
               "any patent claim is infringed by making, using, selling, offering for \n"
               "sale, or importing the Program or any portion of it. \n"
               " \n"
               "11. Patents. \n"
               " \n"
               "A 'contributor' is a copyright holder who authorizes use under this \n"
               "License of the Program or a work on which the Program is based.  The \n"
               "work thus licensed is called the contributor's 'contributor version'. \n"
               " \n"
               "A contributor's 'essential patent claims' are all patent claims \n"
               "owned or controlled by the contributor, whether already acquired or \n"
               "hereafter acquired, that would be infringed by some manner, permitted \n"
               "by this License, of making, using, or selling its contributor version, \n"
               "but do not include claims that would be infringed only as a \n"
               "consequence of further modification of the contributor version.  For \n"
               "purposes of this definition, 'control' includes the right to grant \n"
               "patent sublicenses in a manner consistent with the requirements of \n"
               "this License. \n"
               " \n"
               "Each contributor grants you a non-exclusive, worldwide, royalty-free \n"
               "patent license under the contributor's essential patent claims, to \n"
               "make, use, sell, offer for sale, import and otherwise run, modify and \n"
               "propagate the contents of its contributor version. \n"
               " \n"
               "In the following three paragraphs, a 'patent license' is any express \n"
               "agreement or commitment, however denominated, not to enforce a patent \n"
               "(such as an express permission to practice a patent or covenant not to \n"
               "sue for patent infringement).  To 'grant' such a patent license to a \n"
               "party means to make such an agreement or commitment not to enforce a \n"
               "patent against the party. \n"
               " \n"
               "If you convey a covered work, knowingly relying on a patent license, \n"
               "and the Corresponding Source of the work is not available for anyone \n"
               "to copy, free of charge and under the terms of this License, through a \n"
               "publicly available network server or other readily accessible means, \n"
               "then you must either (1) cause the Corresponding Source to be so \n"
               "available, or (2) arrange to deprive yourself of the benefit of the \n"
               "patent license for this particular work, or (3) arrange, in a manner \n"
               "consistent with the requirements of this License, to extend the patent \n"
               "license to downstream recipients.  'Knowingly relying' means you have \n"
               "actual knowledge that, but for the patent license, your conveying the \n"
               "covered work in a country, or your recipient's use of the covered work \n"
               "in a country, would infringe one or more identifiable patents in that \n"
               "country that you have reason to believe are valid. \n"
               " \n"
               "If, pursuant to or in connection with a single transaction or \n"
               "arrangement, you convey, or propagate by procuring conveyance of, a \n"
               "covered work, and grant a patent license to some of the parties \n"
               "receiving the covered work authorizing them to use, propagate, modify \n"
               "or convey a specific copy of the covered work, then the patent license \n"
               "you grant is automatically extended to all recipients of the covered \n"
               "work and works based on it. \n"
               " \n"
               "A patent license is 'discriminatory' if it does not include within \n"
               "the scope of its coverage, prohibits the exercise of, or is \n"
               "conditioned on the non-exercise of one or more of the rights that are \n"
               "specifically granted under this License.  You may not convey a covered \n"
               "work if you are a party to an arrangement with a third party that is \n"
               "in the business of distributing software, under which you make payment \n"
               "to the third party based on the extent of your activity of conveying \n"
               "the work, and under which the third party grants, to any of the \n"
               "parties who would receive the covered work from you, a discriminatory \n"
               "patent license (a) in connection with copies of the covered work \n"
               "conveyed by you (or copies made from those copies), or (b) primarily \n"
               "for and in connection with specific products or compilations that \n"
               "contain the covered work, unless you entered into that arrangement, \n"
               "or that patent license was granted, prior to 28 March 2007. \n"
               " \n"
               "Nothing in this License shall be construed as excluding or limiting \n"
               "any implied license or other defenses to infringement that may \n"
               "otherwise be available to you under applicable patent law. \n"
               " \n"
               "12. No Surrender of Others' Freedom. \n"
               " \n"
               "If conditions are imposed on you (whether by court order, agreement or \n"
               "otherwise) that contradict the conditions of this License, they do not \n"
               "excuse you from the conditions of this License.  If you cannot convey a \n"
               "covered work so as to satisfy simultaneously your obligations under this \n"
               "License and any other pertinent obligations, then as a consequence you may \n"
               "not convey it at all.  For example, if you agree to terms that obligate you \n"
               "to collect a royalty for further conveying from those to whom you convey \n"
               "the Program, the only way you could satisfy both those terms and this \n"
               "License would be to refrain entirely from conveying the Program. \n"
               " \n"
               "13. Use with the GNU Affero General Public License. \n"
               " \n"
               "Notwithstanding any other provision of this License, you have \n"
               "permission to link or combine any covered work with a work licensed \n"
               "under version 3 of the GNU Affero General Public License into a single \n"
               "combined work, and to convey the resulting work.  The terms of this \n"
               "License will continue to apply to the part which is the covered work, \n"
               "but the special requirements of the GNU Affero General Public License, \n"
               "section 13, concerning interaction through a network will apply to the \n"
               "combination as such. \n"
               " \n"
               "14. Revised Versions of this License. \n"
               " \n"
               "The Free Software Foundation may publish revised and/or new versions of \n"
               "the GNU General Public License from time to time.  Such new versions will \n"
               "be similar in spirit to the present version, but may differ in detail to \n"
               "address new problems or concerns. \n"
               " \n"
               "Each version is given a distinguishing version number.  If the \n"
               "Program specifies that a certain numbered version of the GNU General \n"
               "Public License 'or any later version' applies to it, you have the \n"
               "option of following the terms and conditions either of that numbered \n"
               "version or of any later version published by the Free Software \n"
               "Foundation.  If the Program does not specify a version number of the \n"
               "GNU General Public License, you may choose any version ever published \n"
               "by the Free Software Foundation. \n"
               " \n"
               "If the Program specifies that a proxy can decide which future \n"
               "versions of the GNU General Public License can be used, that proxy's \n"
               "public statement of acceptance of a version permanently authorizes you \n"
               "to choose that version for the Program. \n"
               " \n"
               "Later license versions may give you additional or different \n"
               "permissions.  However, no additional obligations are imposed on any \n"
               "author or copyright holder as a result of your choosing to follow a \n"
               "later version. \n"
               " \n"
               "15. Disclaimer of Warranty. \n"
               " \n"
               "THERE IS NO WARRANTY FOR THE PROGRAM, TO THE EXTENT PERMITTED BY \n"
               "APPLICABLE LAW.  EXCEPT WHEN OTHERWISE STATED IN WRITING THE COPYRIGHT \n"
               "HOLDERS AND/OR OTHER PARTIES PROVIDE THE PROGRAM 'AS IS' WITHOUT WARRANTY \n"
               "OF ANY KIND, EITHER EXPRESSED OR IMPLIED, INCLUDING, BUT NOT LIMITED TO, \n"
               "THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR \n"
               "PURPOSE.  THE ENTIRE RISK AS TO THE QUALITY AND PERFORMANCE OF THE PROGRAM \n"
               "IS WITH YOU.  SHOULD THE PROGRAM PROVE DEFECTIVE, YOU ASSUME THE COST OF \n"
               "ALL NECESSARY SERVICING, REPAIR OR CORRECTION. \n"
               " \n"
               "16. Limitation of Liability. \n"
               " \n"
               "IN NO EVENT UNLESS REQUIRED BY APPLICABLE LAW OR AGREED TO IN WRITING \n"
               "WILL ANY COPYRIGHT HOLDER, OR ANY OTHER PARTY WHO MODIFIES AND/OR CONVEYS \n"
               "THE PROGRAM AS PERMITTED ABOVE, BE LIABLE TO YOU FOR DAMAGES, INCLUDING ANY \n"
               "GENERAL, SPECIAL, INCIDENTAL OR CONSEQUENTIAL DAMAGES ARISING OUT OF THE \n"
               "USE OR INABILITY TO USE THE PROGRAM (INCLUDING BUT NOT LIMITED TO LOSS OF \n"
               "DATA OR DATA BEING RENDERED INACCURATE OR LOSSES SUSTAINED BY YOU OR THIRD \n"
               "PARTIES OR A FAILURE OF THE PROGRAM TO OPERATE WITH ANY OTHER PROGRAMS), \n"
               "EVEN IF SUCH HOLDER OR OTHER PARTY HAS BEEN ADVISED OF THE POSSIBILITY OF \n"
               "SUCH DAMAGES. \n"
               " \n"
               "17. Interpretation of Sections 15 and 16. \n"
               " \n"
               "If the disclaimer of warranty and limitation of liability provided \n"
               "above cannot be given local legal effect according to their terms, \n"
               "reviewing courts shall apply local law that most closely approximates \n"
               "an absolute waiver of all civil liability in connection with the \n"
               "Program, unless a warranty or assumption of liability accompanies a \n"
               "copy of the Program in return for a fee. \n"
               " \n"
               " END OF TERMS AND CONDITIONS \n\n");
}

__host__ void print_help() {
        printf("Example: ./bin/gpuvmem options [ arguments ...]\n");
        printf("    -h  --help             Shows this\n");
        printf( "   -X  --blockSizeX       Block X Size for Image (Needs to be pow of 2) (Mandatory)\n");
        printf( "   -Y  --blockSizeY       Block Y Size for Image (Needs to be pow of 2) (Mandatory)\n");
        printf( "   -V  --blockSizeV       Block Size for Visibilities (Needs to be pow of 2) (Mandatory)\n");
        printf( "   -i  --input            The name of the input file of visibilities(MS) (Mandatory)\n");
        printf( "   -o  --output           The name of the output file of residual visibilities(MS) (Mandatory)\n");
        printf( "   -O  --output-image     The name of the output image FITS file (Mandatory)\n");
        printf("    -I  --inputdat         The name of the input file of parameters (Mandatory)\n");
        printf("    -m  --modin            mod_in_0 FITS (I_nu_0) image file (Mandatory)\n");
        printf("    -F  --nu_0             Frequency of reference (Mandatory)\n");
        printf("    -a  --alpha            Alpha spectral index image\n");
        printf("    -x  --minpix           Minimum positive value of a pixel (Optional)\n");
        printf("    -n  --noise            Noise Parameter (Optional)\n");
        printf("    -N  --noise-cut        Noise-cut Parameter (Optional)\n");
        printf("    -l  --lambda           Lambda Regularization Parameter (Optional)\n");
        printf("    -E  --epsilon          Epsilon Regularization Parameter (Optional)\n");
        printf("    -g  --gridding         Use gridding to decrease the number of visibilities. This is done in CPU (Need to select the CPU threads that will grid the input visibilities)\n");
        printf("    -r  --randoms          Percentage of data used when random sampling (Default = 1.0, optional)\n");
        printf("    -P  --prior            Prior used to regularize the solution (Default = 0 = Entropy)\n");
        printf("    -e  --eta              Variable that controls the minimum image value (Default eta = -1.0)\n");
        printf("    -p  --path             MEM path to save FITS images. With last / included. (Example ./../mem/)\n");
        printf("    -f  --file             Output file where final objective function values are saved (Optional)\n");
        printf("    -M  --multigpu         Number of GPUs to use multiGPU image synthesis (Default OFF => 0)\n");
        printf("    -R  --robust-parameter Robust weighting parameter when gridding. -2.0 for uniform weighting, 2.0 for natural weighting and 0.0 for a tradeoff between these two. (Default R = 2.0).\n");
        printf("    -s  --select           If multigpu option is OFF, then select the GPU ID of the GPU you will work on. (Default = 0)\n");
        printf("    -t  --iterations       Number of iterations for optimization (Default = 500)\n");
        printf("    -B  --burndown_steps   Burndown iterations\n");
        printf("    -c  --copyright        Shows copyright conditions\n");
        printf("    -w  --warranty         Shows no warranty details\n");
        printf("        --nopositivity     Run gpuvmem using chi2 with no posititivy restriction\n");
        printf("        --apply-noise      Apply random gaussian noise to visibilities\n");
        printf("        --clipping         Clips the image to positive values\n");
        printf("        --print-images     Prints images per iteration\n");
        printf("        --checkpoint       Start from a certain iteration\n");
        printf("        --spec_idx       Spectral index calculation\n");
        printf("        --adaptive         MCMC Noise for each pixel is estimated using the variance of the samples\n");
        printf("        --use-mask         MCMC Noise depends of the signal of the object, more noise will be injected where is no signal\n");
        printf("        --verbose          Shows information through all the execution\n");
}

__host__ char *strip(const char *string, const char *chars)
{
        char * newstr = (char*)malloc(strlen(string) + 1);
        int counter = 0;

        for (; *string; string++) {
                if (!strchr(chars, *string)) {
                        newstr[ counter ] = *string;
                        ++counter;
                }
        }

        newstr[counter] = 0;
        return newstr;
}

void swap (int *a, int *b)
{
        int temp = *a;
        *a = *b;
        *b = temp;
}

void randomize(int2 arr[], int n)
{
        // Use a different seed value so that we don't get same
        // result each time we run this program
        SelectStream(3);

        // Start from the last element and swap one by one. We don't
        // need to run for the first element that's why i > 0
        for (int i = n-1; i > 0; i--)
        {
                // Pick a random index from 0 to i

                int j = Uniform(0, (float)i);

                // Swap arr[i] with the element at random index
                swap(&arr[i].x, &arr[j].x);
                swap(&arr[i].y, &arr[j].y);
        }
}

__host__ Vars getOptions(int argc, char **argv) {
        Vars variables;
        variables.multigpu = "NULL";
        variables.ofile = "NULL";
        variables.path = "mem/";
        variables.output_image = "mod_out";
        variables.select = 0;
        variables.blockSizeX = -1;
        variables.blockSizeY = -1;
        variables.blockSizeV = -1;
        variables.nu_0 = -1;
        variables.it_max = 500;
        variables.burndown_steps = 0;
        variables.noise = -1;
        variables.epsilon = 0.0;
        variables.lambda = -1;
        variables.randoms = -1;
        variables.noise_cut = -1;
        variables.minpix = -1;
        variables.reg_term = 0;
        variables.alpha_name = "NULL";
        variables.eta = -1.0;
        variables.gridding = 0;
        variables.robust_param = 2.0;
        variables.current_k = 1;
        variables.noise_factorInu0 = 1.0;


        long next_op;
        const char* const short_op = "hcwi:o:O:I:m:x:n:N:l:r:f:M:s:p:P:X:R:Y:V:t:F:a:e:E:B:A:g:k:z:";

        const struct option long_op[] = { //Flag for help, copyright and warranty
                {"help", 0, NULL, 'h' },
                {"warranty", 0, NULL, 'w' },
                {"copyright", 0, NULL, 'c' },
                /* These options set a flag. */
                {"verbose", 0, &verbose_flag, 1},
                {"apply-noise", 0, &apply_noise, 1},
                {"nopositivity", 0, &nopositivity, 1},
                {"clipping", 0, &clip_flag, 1},
                {"checkpoint", 0, &checkpoint, 1},
                {"print-images", 0, &print_images, 1},
                {"adaptive", 0, &adaptive, 1},
                {"use-mask", 0, &use_mask, 1},
                {"spec_idx", 0, &spec_idx, 1},
                /* These options don’t set a flag. */
                {"input", 1, NULL, 'i' }, {"output", 1, NULL, 'o'}, {"output-image", 1, NULL, 'O'},
                {"inputdat", 1, NULL, 'I'}, {"modin", 1, NULL, 'm' }, {"noise", 0, NULL, 'n' },
                {"lambda", 0, NULL, 'l' }, {"multigpu", 0, NULL, 'M'}, {"select", 1, NULL, 's'},
                {"path", 1, NULL, 'p'}, {"prior", 0, NULL, 'P'}, {"eta", 0, NULL, 'e'},
                {"blockSizeX", 1, NULL, 'X'}, {"blockSizeY", 1, NULL, 'Y'}, {"robust-parameter", 0, NULL, 'R'}, {"blockSizeV", 1, NULL, 'V'},
                {"iterations", 0, NULL, 't'}, {"burndown_steps", 1, NULL, 'B'}, {"noise-cut", 0, NULL, 'N' }, {"minpix", 0, NULL, 'x' },
                {"randoms", 0, NULL, 'r' }, {"nu_0", 1, NULL, 'F' }, {"file", 0, NULL, 'f' }, {"current_k", 1, NULL, 'k' },
                {"epsilon", 0, NULL, 'E' }, {"alpha_name", 1, NULL, 'a' }, {"alpha_value", 1, NULL, 'A' }, {"gridding", 0, NULL, 'g' },
                {"noise-factor", 0, NULL, 'z' },
                { NULL, 0, NULL, 0 }
        };

        if (argc == 1) {
                printf(
                        "ERROR. THE PROGRAM HAS BEEN EXECUTED WITHOUT THE NEEDED PARAMETERS OR OPTIONS\n");
                print_help();
                exit(EXIT_SUCCESS);
        }
        int option_index = 0;
        while (1) {
                next_op = getopt_long(argc, argv, short_op, long_op, &option_index);
                if (next_op == -1) {
                        break;
                }

                switch (next_op) {
                case 0:
                        /* If this option set a flag, do nothing else now. */
                        if (long_op[option_index].flag != 0)
                                break;
                        printf ("option %s", long_op[option_index].name);
                        if (optarg)
                                printf (" with arg %s", optarg);
                        printf ("\n");
                        break;
                case 'h':
                        print_help();
                        exit(EXIT_SUCCESS);
                case 'w':
                        print_warranty();
                        exit(EXIT_SUCCESS);
                case 'c':
                        print_copyright();
                        exit(EXIT_SUCCESS);
                case 'i':
                        variables.input = (char*) malloc((strlen(optarg)+1)*sizeof(char));
                        strcpy(variables.input, optarg);
                        break;
                case 'o':
                        variables.output = (char*) malloc((strlen(optarg)+1)*sizeof(char));
                        strcpy(variables.output, optarg);
                        break;
                case 'O':
                        variables.output_image = (char*) malloc((strlen(optarg)+1)*sizeof(char));
                        strcpy(variables.output_image, optarg);
                        break;
                case 'I':
                        variables.inputdat = (char*) malloc((strlen(optarg)+1)*sizeof(char));
                        strcpy(variables.inputdat, optarg);
                        break;
                case 'm':
                        variables.modin = (char*) malloc((strlen(optarg)+1)*sizeof(char));
                        strcpy(variables.modin, optarg);
                        break;
                case 'x':
                        variables.minpix = atof(optarg);
                        break;
                case 'e':
                        variables.eta = atof(optarg);
                        break;
                case 'E':
                        variables.epsilon = atof(optarg);
                        break;
                case 'F':
                        variables.nu_0 = atof(optarg);
                        break;
                case 'R':
                        variables.robust_param = atof(optarg);
                        break;
                case 'a':
                        variables.alpha_name = (char*) malloc((strlen(optarg)+1)*sizeof(char));
                        strcpy(variables.alpha_name, optarg);
                        break;
                case 'A':
                        variables.alpha_value = atof(optarg);
                        break;
                case 'n':
                        variables.noise = atof(optarg);
                        break;
                case 'N':
                        variables.noise_cut = atof(optarg);
                        break;
                case 'l':
                        variables.lambda = atof(optarg);
                        break;
                case 'p':
                        variables.path = (char*) malloc((strlen(optarg)+1)*sizeof(char));
                        strcpy(variables.path, optarg);
                        break;
                case 'P':
                        variables.reg_term = atoi(optarg);;
                        break;
                case 'M':
                        variables.multigpu = optarg;
                        break;
                case 'r':
                        variables.randoms = atof(optarg);
                        break;
                case 'f':
                        variables.ofile = (char*) malloc((strlen(optarg)+1)*sizeof(char));
                        strcpy(variables.ofile, optarg);
                        break;
                case 's':
                        variables.select = atoi(optarg);
                        break;
                case 'X':
                        variables.blockSizeX = atoi(optarg);
                        break;
                case 'Y':
                        variables.blockSizeY = atoi(optarg);
                        break;
                case 'V':
                        variables.blockSizeV = atoi(optarg);
                        break;
                case 'k':
                        variables.current_k = atoi(optarg);
                        break;
                case 't':
                        variables.it_max = atoi(optarg);
                        break;
                case 'B':
                        variables.burndown_steps = atoi(optarg);
                        break;
                case 'g':
                        variables.gridding = atoi(optarg);
                        break;
                case 'z':
                        variables.noise_factorInu0 = atof(optarg);
                        break;
                case '?':
                        print_help();
                        exit(EXIT_FAILURE);
                case -1:
                        break;
                default:
                        print_help();
                        exit(EXIT_FAILURE);
                }
        }

        if(variables.blockSizeX == -1 && variables.blockSizeY == -1 && variables.blockSizeV == -1 ||
           strcmp(strip(variables.input, " "),"") == 0 && strcmp(strip(variables.output, " "),"") == 0 && strcmp(strip(variables.output_image, " "),"") == 0 && strcmp(strip(variables.inputdat, " "),"") == 0 ||
           strcmp(strip(variables.modin, " "),"") == 0 && strcmp(strip(variables.path, " "),"") == 0 || variables.nu_0 == -1 || strcmp(strip(variables.alpha_name, " "),"") == 0) {
                print_help();
                exit(EXIT_FAILURE);
        }

        if(!isPow2(variables.blockSizeX) && !isPow2(variables.blockSizeY) && !isPow2(variables.blockSizeV)) {
                print_help();
                exit(EXIT_FAILURE);
        }

        if(variables.randoms > 1.0) {
                print_help();
                exit(EXIT_FAILURE);
        }

        if(variables.reg_term > 2) {
                print_help();
                exit(EXIT_FAILURE);
        }

        if(variables.gridding < 0) {
                print_help();
                exit(EXIT_FAILURE);
        }

        if(strcmp(variables.multigpu,"NULL")!=0 && variables.select != 0) {
                print_help();
                exit(EXIT_FAILURE);
        }
        return variables;
}

template <bool nIsPow2>
__global__ void deviceReduceKernel(float *g_idata, float *g_odata, unsigned int n)
{
        extern __shared__ float sdata[];

        // perform first level of reduction,
        // reading from global memory, writing to shared memory
        unsigned int tid = threadIdx.x;
        unsigned int blockSize = blockDim.x;
        unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
        unsigned int gridSize = blockSize*2*gridDim.x;

        float mySum = 0.f;

        // we reduce multiple elements per thread.  The number is determined by the
        // number of active thread blocks (via gridDim).  More blocks will result
        // in a larger gridSize and therefore fewer elements per thread
        while (i < n)
        {
                mySum += g_idata[i];

                // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
                if (nIsPow2 || i + blockSize < n)
                        mySum += g_idata[i+blockSize];

                i += gridSize;
        }

        // each thread puts its local sum into shared memory
        sdata[tid] = mySum;
        __syncthreads();


        // do reduction in shared mem
        if ((blockSize >= 512) && (tid < 256))
        {
                sdata[tid] = mySum = mySum + sdata[tid + 256];
        }

        __syncthreads();

        if ((blockSize >= 256) &&(tid < 128))
        {
                sdata[tid] = mySum = mySum + sdata[tid + 128];
        }

        __syncthreads();

        if ((blockSize >= 128) && (tid <  64))
        {
                sdata[tid] = mySum = mySum + sdata[tid +  64];
        }

        __syncthreads();

#if (__CUDA_ARCH__ >= 300 )
        if ( tid < 32 )
        {
                // Fetch final intermediate sum from 2nd warp
                if (blockSize >=  64) mySum += sdata[tid + 32];
                // Reduce final warp using shuffle
                for (int offset = warpSize/2; offset > 0; offset /= 2)
                {
                        mySum += __shfl_down(mySum, offset);
                }
        }
#else
        // fully unroll reduction within a single warp
        if ((blockSize >=  64) && (tid < 32))
        {
                sdata[tid] = mySum = mySum + sdata[tid + 32];
        }

        __syncthreads();

        if ((blockSize >=  32) && (tid < 16))
        {
                sdata[tid] = mySum = mySum + sdata[tid + 16];
        }

        __syncthreads();

        if ((blockSize >=  16) && (tid <  8))
        {
                sdata[tid] = mySum = mySum + sdata[tid +  8];
        }

        __syncthreads();

        if ((blockSize >=   8) && (tid <  4))
        {
                sdata[tid] = mySum = mySum + sdata[tid +  4];
        }

        __syncthreads();

        if ((blockSize >=   4) && (tid <  2))
        {
                sdata[tid] = mySum = mySum + sdata[tid +  2];
        }

        __syncthreads();

        if ((blockSize >=   2) && ( tid <  1))
        {
                sdata[tid] = mySum = mySum + sdata[tid +  1];
        }

        __syncthreads();
#endif

        // write result for this block to global mem
        if (tid == 0) g_odata[blockIdx.x] = mySum;
}





__host__ float deviceReduce(float *in, long N)
{
        float *device_out;
        gpuErrchk(hipMalloc(&device_out, sizeof(float)*1024));
        gpuErrchk(hipMemset(device_out, 0, sizeof(float)*1024));

        int threads = 512;
        int blocks = min((int(NearestPowerOf2(N)) + threads - 1) / threads, 1024);
        int smemSize = (threads <= 32) ? 2 * threads * sizeof(float) : threads * sizeof(float);

        bool isPower2 = isPow2(N);
        if(isPower2) {
                deviceReduceKernel<true><<<blocks, threads, smemSize>>>(in, device_out, N);
                gpuErrchk(hipDeviceSynchronize());
        }else{
                deviceReduceKernel<false><<<blocks, threads, smemSize>>>(in, device_out, N);
                gpuErrchk(hipDeviceSynchronize());
        }

        float *h_odata = (float *) malloc(blocks*sizeof(float));
        float sum = 0.0;

        gpuErrchk(hipMemcpy(h_odata, device_out, blocks * sizeof(float),hipMemcpyDeviceToHost));
        for (int i=0; i<blocks; i++)
        {
                sum += h_odata[i];
        }
        hipFree(device_out);
        free(h_odata);
        return sum;
}

__global__ void hermitianSymmetry(double3 *UVW, hipfftComplex *Vo, float freq, int numVisibilities)
{
        int i = threadIdx.x + blockDim.x * blockIdx.x;

        if (i < numVisibilities) {
                if(UVW[i].x < 0.0) {
                        UVW[i].x *= -1.0;
                        UVW[i].y *= -1.0;
                        Vo[i].y *= -1.0;
                }
                UVW[i].x *= freq / LIGHTSPEED;
                UVW[i].y *= freq / LIGHTSPEED;
                UVW[i].z *= freq / LIGHTSPEED;
        }
}


__device__ float AiryDiskBeam(float distance, float lambda, float antenna_diameter, float pb_factor)
{
        float atten;
        float r = pb_factor * lambda / antenna_diameter;
        float bessel_arg = PI*distance/(r/RZ);
        float bessel_func = j1f(bessel_arg);
        if(distance == 0.0f) {
                atten = 1.0f;
        }else{
                atten = 4.0f * (bessel_func/bessel_arg) * (bessel_func/bessel_arg);
        }
        return atten;
}

__device__ float EllipticGaussianKernel(float amplitude, int x_c, int y_c, float bmaj, float bmin, float bpa, double DELTAX, double DELTAY)
{
        int j = threadIdx.x + blockDim.x * blockIdx.x;
        int i = threadIdx.y + blockDim.y * blockIdx.y;

        float x = (j - x_c) * DELTAX * RPDEG_D;
        float y = (i - y_c) * DELTAY * RPDEG_D;
        float cos_bpa = cosf(bpa);
        float sin_bpa = sinf(bpa);
        float sin_bpa_2 = sinf(2.0*bpa);
        float a = (cos_bpa*cos_bpa)/(2.0*bmaj*bmaj) + (sin_bpa*sin_bpa)/(2.0*bmin*bmin);
        float b = sin_bpa_2/(2.0*bmaj*bmaj) - sin_bpa_2/(2.0*bmin*bmin);
        float c = (sin_bpa*sin_bpa)/(2.0*bmaj*bmaj) + (cos_bpa*cos_bpa)/(2.0*bmin*bmin);
        float G = amplitude*expf(-a*x*x - b*x*y - c*y*y);
        return G;
}

__device__ float GaussianBeam(float distance, float lambda, float antenna_diameter, float pb_factor)
{
        float fwhm = pb_factor * lambda / antenna_diameter;
        float c = 4.0*logf(2.0);
        float r = distance/fwhm;
        float atten = expf(-c*r*r);
        return atten;
}

__device__ float attenuation(float antenna_diameter, float pb_factor, float pb_cutoff, float freq, float xobs, float yobs, double DELTAX, double DELTAY)
{

        int j = threadIdx.x + blockDim.x * blockIdx.x;
        int i = threadIdx.y + blockDim.y * blockIdx.y;

        float atten_result, atten;

        int x0 = xobs;
        int y0 = yobs;
        float x = (j - x0) * DELTAX * RPDEG_D;
        float y = (i - y0) * DELTAY * RPDEG_D;

        float arc = sqrtf(x*x+y*y);
        float lambda = LIGHTSPEED/freq;
        atten = GaussianBeam(arc, lambda, antenna_diameter, pb_factor);

        if(arc <= pb_cutoff) {
                atten_result = atten;
        }else{
                atten_result = 0.0f;
        }

        return atten_result;
}



__global__ void total_attenuation(float *total_atten, float antenna_diameter, float pb_factor, float pb_cutoff, float freq, float xobs, float yobs, double DELTAX, double DELTAY, long N)
{
        int j = threadIdx.x + blockDim.x * blockIdx.x;
        int i = threadIdx.y + blockDim.y * blockIdx.y;

        float attenPerFreq = attenuation(antenna_diameter, pb_factor, pb_cutoff, freq, xobs, yobs, DELTAX, DELTAY);

        total_atten[N*i+j] += attenPerFreq;
}

__global__ void mean_attenuation(float *total_atten, int channels, long N)
{
        int j = threadIdx.x + blockDim.x * blockIdx.x;
        int i = threadIdx.y + blockDim.y * blockIdx.y;

        total_atten[N*i+j] /= channels;
}

__global__ void weight_image(float *weight_image, float *total_atten, float noise_jypix, long N)
{
        int j = threadIdx.x + blockDim.x * blockIdx.x;
        int i = threadIdx.y + blockDim.y * blockIdx.y;

        float atten = total_atten[N*i+j];
        weight_image[N*i+j] += (atten / noise_jypix) * (atten / noise_jypix);
}

__global__ void noise_image(float *noise_image, float *weight_image, float noise_jypix, long N)
{
        int j = threadIdx.x + blockDim.x * blockIdx.x;
        int i = threadIdx.y + blockDim.y * blockIdx.y;

        float noiseval;
        noiseval = sqrtf(1.0/weight_image[N*i+j]);
        noise_image[N*i+j] = noiseval;
}

__global__ void apply_beam(float antenna_diameter, float pb_factor, float pb_cutoff, hipfftComplex *image, long N, float xobs, float yobs, float fg_scale, float freq, float DELTAX, float DELTAY)
{
        int j = threadIdx.x + blockDim.x * blockIdx.x;
        int i = threadIdx.y + blockDim.y * blockIdx.y;

        float atten = attenuation(antenna_diameter, pb_factor, pb_cutoff, freq, xobs, yobs, DELTAX, DELTAY);

        image[N*i+j].x = image[N*i+j].x * atten;
        image[N*i+j].y = 0.0;
}


/*--------------------------------------------------------------------
 * Phase rotate the visibility data in "image" to refer phase to point
 * (x,y) instead of (0,0).
 * Multiply pixel V(i,j) by exp(-2 pi i (x/ni + y/nj))
 *--------------------------------------------------------------------*/
__global__ void phase_rotate(hipfftComplex *data, long M, long N, float xphs, float yphs)
{

        int j = threadIdx.x + blockDim.x * blockIdx.x;
        int i = threadIdx.y + blockDim.y * blockIdx.y;

        float u,v, phase, c, s, re, im;
        float du = xphs/(float)M;
        float dv = yphs/(float)N;

        if(j < M/2) {
                u = du * j;
        }else{
                u = du * (j-M);
        }

        if(i < N/2) {
                v = dv * i;
        }else{
                v = dv * (i-N);
        }

        phase = 2.0*(u+v);
    #if (__CUDA_ARCH__ >= 300 )
        sincospif(phase, &s, &c);
    #else
        c = cospif(phase);
        s = sinpif(phase);
    #endif
        re = data[N*i+j].x;
        im = data[N*i+j].y;
        data[N*i+j].x = re * c - im * s;
        data[N*i+j].y = re * s + im * c;
}


/*
 * Interpolate in the visibility array to find the visibility at (u,v);
 */
__global__ void vis_mod(hipfftComplex *Vm, hipfftComplex *V, double3 *UVW, float *weight, double deltau, double deltav, long numVisibilities, long N)
{
        int i = threadIdx.x + blockDim.x * blockIdx.x;
        int i1, i2, j1, j2;
        double du, dv;
        double2 uv;
        hipfftComplex v11, v12, v21, v22;
        float Zreal;
        float Zimag;

        if (i < numVisibilities) {

                uv.x = UVW[i].x/fabs(deltau);
                uv.y = UVW[i].y/deltav;

                if (fabs(uv.x) <= (N/2)+0.5 && fabs(uv.y) <= (N/2)+0.5) {

                        if(uv.x < 0.0)
                                uv.x = round(uv.x+N);


                        if(uv.y < 0.0)
                                uv.y = round(uv.y+N);


                        i1 = (int)uv.x;
                        i2 = (i1+1)%N;
                        du = uv.x - i1;

                        j1 = (int)uv.y;
                        j2 = (j1+1)%N;
                        dv = uv.y - j1;

                        if (i1 >= 0 && i1 < N && i2 >= 0 && i2 < N && j1 >= 0 && j1 < N && j2 >= 0 && j2 < N) {
                                /* Bilinear interpolation */
                                v11 = V[N*j1 + i1]; /* [i1, j1] */
                                v12 = V[N*j2 + i1]; /* [i1, j2] */
                                v21 = V[N*j1 + i2]; /* [i2, j1] */
                                v22 = V[N*j2 + i2]; /* [i2, j2] */

                                Zreal = (1-du)*(1-dv)*v11.x + (1-du)*dv*v12.x + du*(1-dv)*v21.x + du*dv*v22.x;
                                Zimag = (1-du)*(1-dv)*v11.y + (1-du)*dv*v12.y + du*(1-dv)*v21.y + du*dv*v22.y;

                                Vm[i].x = Zreal;
                                Vm[i].y = Zimag;
                        }else{
                                weight[i] = 0.0f;
                        }
                }else{
                        //Vm[i].x = 0.0f;
                        //Vm[i].y = 0.0f;
                        weight[i] = 0.0f;
                }

        }

}

__global__ void residual(hipfftComplex *Vr, hipfftComplex *Vm, hipfftComplex *Vo, long numVisibilities){
        int i = threadIdx.x + blockDim.x * blockIdx.x;
        if (i < numVisibilities) {
                Vr[i].x = Vm[i].x - Vo[i].x;
                Vr[i].y = Vm[i].y - Vo[i].y;
        }
}
__global__ void clipWNoise(float *noise, hipfftComplex *I, long N, float noise_cut, float MINPIX, float eta)

{
        int j = threadIdx.x + blockDim.x * blockIdx.x;
        int i = threadIdx.y + blockDim.y * blockIdx.y;

        if(noise[N*i+j] > noise_cut) {
                if(eta > 0.0) {
                        I[N*i+j].x = 0.0;
                }
                else{
                        I[N*i+j].x = -1.0 * eta * MINPIX;
                }

        }

        I[N*i+j].y = 0;
}

__global__ void clip2IWNoise(float *noise, float2 *I, long N, float noise_cut, float minpix, float alpha_start, float fg_scale, float eta)
{
        int j = threadIdx.x + blockDim.x * blockIdx.x;
        int i = threadIdx.y + blockDim.y * blockIdx.y;

        if(noise[N*i+j] > noise_cut) {
                if(eta > 0.0) {
                        I[N*i+j].x = 0.0;
                }
                else{
                        I[N*i+j].x = -1.0f * eta * minpix * fg_scale;
                }

                I[N*i+j].y = 0.0f;

        }

}

__global__ void clip2I(float2 *I, long N, float minpix, float fg_scale, float eta)
{
        int j = threadIdx.x + blockDim.x * blockIdx.x;
        int i = threadIdx.y + blockDim.y * blockIdx.y;

        //I_nu_0
        if(I[N*i+j].x < -1.0f * eta * minpix * fg_scale) {
                I[N*i+j].x = -1.0f * eta * minpix * fg_scale;
        }
        /*/ /alpha
           if(I[N*i+j].y < minpix_alpha) {
                I[N*i+j].y = minpix_alpha;
           } */
}

__global__ void clip(hipfftComplex *I, long N, float MINPIX)
{
        int j = threadIdx.x + blockDim.x * blockIdx.x;
        int i = threadIdx.y + blockDim.y * blockIdx.y;

        //T
        if(I[N*i+j].x < MINPIX && MINPIX >= 0.0) {
                I[N*i+j].x = MINPIX;
        }
        I[N*i+j].y = 0.0f;
}

__global__ void newP(float2 *p, float2 *xi, float xmin, long N, float minpix, float fg_scale, float eta)
{
        int j = threadIdx.x + blockDim.x * blockIdx.x;
        int i = threadIdx.y + blockDim.y * blockIdx.y;

        xi[N*i+j].x *= xmin;
        xi[N*i+j].y *= xmin;
        //I_nu_0
        if(p[N*i+j].x + xi[N*i+j].x > -1.0*eta*minpix*fg_scale) {
                p[N*i+j].x += xi[N*i+j].x;
        }else{
                p[N*i+j].x = -1.0*eta*minpix*fg_scale;
                xi[N*i+j].x = 0.0;
        }

        p[N*i+j].y += xi[N*i+j].y;
        /*/ /alpha
           if(p[N*i+j].y + xi[N*i+j].y > minpix_alpha) {
                p[N*i+j].y += xi[N*i+j].y;
           }else{
                p[N*i+j].y = minpix_alpha;
                xi[N*i+j].y = 0.0;
           } */

}

__global__ void newPNoPositivity(float2 *p, float2 *xi, float xmin, long N)
{
        int j = threadIdx.x + blockDim.x * blockIdx.x;
        int i = threadIdx.y + blockDim.y * blockIdx.y;

        xi[N*i+j].x *= xmin;
        xi[N*i+j].y *= xmin;

        p[N*i+j].x += xi[N*i+j].x;
        p[N*i+j].y += xi[N*i+j].y;
}

__global__ void evaluateXt(float2 *xt, float2 *pcom, float2 *xicom, float x, long N, float minpix, float fg_scale, float eta)
{
        int j = threadIdx.x + blockDim.x * blockIdx.x;
        int i = threadIdx.y + blockDim.y * blockIdx.y;
        //I_nu_0
        if(pcom[N*i+j].x + x * xicom[N*i+j].x > -1.0*eta*minpix*fg_scale) {
                xt[N*i+j].x = pcom[N*i+j].x + x * xicom[N*i+j].x;
        }else{
                xt[N*i+j].x = -1.0*eta*minpix*fg_scale;
        }

        xt[N*i+j].y = pcom[N*i+j].y + x * xicom[N*i+j].y;
        //alpha
        /*if(pcom[N*i+j].y + x * xicom[N*i+j].y > minpix_alpha){
           xt[N*i+j].y = pcom[N*i+j].y + x * xicom[N*i+j].y;
           }else{
            xt[N*i+j].y = minpix_alpha;
           }*/
}


__global__ void evaluateXtNoPositivity(float2 *xt, float2 *pcom, float2 *xicom, float x, long N)
{
        int j = threadIdx.x + blockDim.x * blockIdx.x;
        int i = threadIdx.y + blockDim.y * blockIdx.y;
        //I_nu_0
        xt[N*i+j].x = pcom[N*i+j].x + x * xicom[N*i+j].x;
        //alpha
        xt[N*i+j].y = pcom[N*i+j].y + x * xicom[N*i+j].y;
}


__global__ void chi2Vector(float *chi2, hipfftComplex *Vr, float *w, long numVisibilities)
{
        int i = threadIdx.x + blockDim.x * blockIdx.x;

        if (i < numVisibilities) {
                chi2[i] =  w[i] * ((Vr[i].x * Vr[i].x) + (Vr[i].y * Vr[i].y));
        }

}

__global__ void LVectorAlpha(float *L, float *noise, float2 *I, long N, float noise_cut)
{
        int j = threadIdx.x + blockDim.x * blockIdx.x;
        int i = threadIdx.y + blockDim.y * blockIdx.y;

        float Dx, Dy;

        if(noise[N*i+j] <= noise_cut) {
                if((i>1 && i<N-1) && (j>1 && j<N-1)) {
                        Dx = I[N*i+(j-1)].y - 2 * I[N*i+j].y + I[N*i+(j+1)].y;
                        Dy = I[N*(i-1)+j].y - 2 * I[N*i+j].y + I[N*(i+1)+j].y;
                        L[N*i+j] = 0.5 * (Dx + Dy) * (Dx + Dy);
                }else{
                        L[N*i+j] = I[N*i+j].y;
                }
        }
}

__global__ void SVector(float *S, float *noise, float2 *I, long N, float noise_cut, float MINPIX, float fg_scale, float eta)
{
        int j = threadIdx.x + blockDim.x * blockIdx.x;
        int i = threadIdx.y + blockDim.y * blockIdx.y;

        float entropy = 0.0;
        float I_code = I[N*i+j].x / fg_scale;
        if(noise[N*i+j] <= noise_cut) {
                entropy = I_code * logf((I_code / MINPIX) + (eta + 1.0));
        }

        S[N*i+j] = entropy;
}

__global__ void QPVector(float *Q, float *noise, float2 *I, long N, float noise_cut)
{
        int j = threadIdx.x + blockDim.x * blockIdx.x;
        int i = threadIdx.y + blockDim.y * blockIdx.y;

        float qp = 0.0;
        if(noise[N*i+j] <= noise_cut) {
                if((i>0 && i<N) && (j>0 && j<N)) {
                        qp = (I[N*i+j].x - I[N*i+(j-1)].x) * (I[N*i+j].x - I[N*i+(j-1)].x) +
                             (I[N*i+j].x - I[N*i+(j+1)].x) * (I[N*i+j].x - I[N*i+(j+1)].x) +
                             (I[N*i+j].x - I[N*(i-1)+j].x) * (I[N*i+j].x - I[N*(i-1)+j].x) +
                             (I[N*i+j].x - I[N*(i+1)+j].x) * (I[N*i+j].x - I[N*(i+1)+j].x);
                        qp /= 2.0;
                }else{
                        qp = I[N*i+j].x;
                }
        }

        Q[N*i+j] = qp;
}

__global__ void TVVector(float *TV, float *noise, float2 *I, long N, float noise_cut)
{
        int j = threadIdx.x + blockDim.x * blockIdx.x;
        int i = threadIdx.y + blockDim.y * blockIdx.y;

        float tv = 0.0;
        if(noise[N*i+j] <= noise_cut) {
                if(i!= N-1 || j!=N-1) {
                        float dx = I[N*i+j].x - I[N*i+(j+1)].x;
                        float dy = I[N*i+j].x - I[N*(i+1)+j].x;
                        tv = sqrtf((dx * dx) + (dy * dy));
                }else{
                        tv = I[N*i+j].x;
                }
        }

        TV[N*i+j] = tv;
}

__global__ void searchDirection(float2 *g, float2 *xi, float2 *h, long N)
{
        int j = threadIdx.x + blockDim.x * blockIdx.x;
        int i = threadIdx.y + blockDim.y * blockIdx.y;

        g[N*i+j].x = -xi[N*i+j].x;
        g[N*i+j].y = -xi[N*i+j].y;

        xi[N*i+j].x = h[N*i+j].x = g[N*i+j].x;
        xi[N*i+j].y = h[N*i+j].y = g[N*i+j].y;
}

__global__ void newXi(float2 *g, float2 *xi, float2 *h, float gam, long N)
{
        int j = threadIdx.x + blockDim.x * blockIdx.x;
        int i = threadIdx.y + blockDim.y * blockIdx.y;

        g[N*i+j].x = -xi[N*i+j].x;
        g[N*i+j].y = -xi[N*i+j].y;

        xi[N*i+j].x = h[N*i+j].x = g[N*i+j].x + gam * h[N*i+j].x;
        xi[N*i+j].y = h[N*i+j].y = g[N*i+j].y + gam * h[N*i+j].y;
}

__global__ void getGGandDGG(float *gg, float *dgg, float2 *xi, float2 *g, long N)
{
        int j = threadIdx.x + blockDim.x * blockIdx.x;
        int i = threadIdx.y + blockDim.y * blockIdx.y;

        float gg_Inu_0, gg_alpha;
        float dgg_Inu_0, dgg_alpha;

        gg_Inu_0 = g[N*i+j].x * g[N*i+j].x;
        gg_alpha = g[N*i+j].y * g[N*i+j].y;

        dgg_Inu_0 = (xi[N*i+j].x + g[N*i+j].x) * xi[N*i+j].x;
        dgg_alpha = (xi[N*i+j].y + g[N*i+j].y) * xi[N*i+j].y;

        gg[N*i+j] = gg_Inu_0 + gg_alpha;
        dgg[N*i+j] = dgg_Inu_0 + dgg_alpha;
}

__global__ void restartDPhi(float2 *dphi, float *dS, float *dL, long N)
{
        int j = threadIdx.x + blockDim.x * blockIdx.x;
        int i = threadIdx.y + blockDim.y * blockIdx.y;
        //I_nu_0
        dphi[N*i+j].x = 0.0;
        //alpha
        dphi[N*i+j].y = 0.0;

        dS[N*i+j] = 0.0;

        dL[N*i+j] = 0.0;

}


__global__ void calculateInu(hipfftComplex *I_nu, float2 *image2, float nu, float nu_0, float fg_scale, float minpix, float eta, long N)
{
        int j = threadIdx.x + blockDim.x * blockIdx.x;
        int i = threadIdx.y + blockDim.y * blockIdx.y;

        float I_nu_0, alpha, nudiv_pow_alpha, nudiv;

        nudiv = nu/nu_0;

        I_nu_0 = image2[N*i+j].x;
        alpha = image2[N*i+j].y;
        //alpha = 3.5;

        nudiv_pow_alpha = powf(nudiv, alpha);

        I_nu[N*i+j].x = I_nu_0 * nudiv_pow_alpha;

        I_nu[N*i+j].y = 0.0f;
}

__global__ void random_init(unsigned int seed, hiprandState_t* states, long N)
{
        int j = threadIdx.x + blockDim.x * blockIdx.x;
        int i = threadIdx.y + blockDim.y * blockIdx.y;
        hiprand_init(seed, /* the seed can be the same for each thread, here we pass the time from CPU */
                    (N*i+j), /* the sequence number should be different for each core */
                    0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
                    &states[N*i+j]);
}

__global__ void calculateTheta(float2 *theta, hiprandState_t* states, double2 *total, double2 *total2, int samples, long N){
        int j = threadIdx.x + blockDim.x * blockIdx.x;
        int i = threadIdx.y + blockDim.y * blockIdx.y;

        float2 random_number;
        float2 stddev, avg;

        avg.x = total[N*i+j].x / samples;
        avg.y = total[N*i+j].y / samples;

        stddev.x = (2.38/(N*sqrtf(2))) * sqrtf((total2[N*i+j].x / samples) - (avg.x * avg.x));
        stddev.y = (2.38/(N*sqrtf(2))) * sqrtf((total2[N*i+j].y / samples) - (avg.y * avg.y));

        random_number.x = hiprand_normal(&states[N*i+j]) * stddev.x;
        random_number.y = hiprand_normal(&states[N*i+j]) * stddev.y;

        theta[N*i+j].x =  random_number.y;
        theta[N*i+j].x =  random_number.y;
}

__global__ void changeI(float2 *I, float2 *temp, float2 *theta, hiprandState_t* states, long N)
{
        int j = threadIdx.x + blockDim.x * blockIdx.x;
        int i = threadIdx.y + blockDim.y * blockIdx.y;

        float2 nrandom;

        nrandom.x = hiprand_normal(&states[N*i+j]) * theta[N*i+j].x;
        nrandom.y = hiprand_normal(&states[N*i+j]) * theta[N*i+j].y;

        temp[N*i+j].x = I[N*i+j].x + nrandom.x;
        temp[N*i+j].y = I[N*i+j].y + nrandom.y;

}


__host__ void do_gridding(Field *fields, freqData *data, double deltau, double deltav, int M, int N, float robust)
{
        int local_max = 0;
        int max = 0;
        float pow2_factor, S2, w_avg;
        for(int f=0; f < data->nfields; f++) {
                for(int i=0; i < data->total_frequencies; i++) {
            #pragma omp parallel for schedule(static,1)
                        for(int z=0; z < fields[f].numVisibilitiesPerFreq[i]; z++) {
                                int j, k;
                                double3 uvw;
                                float w;
                                hipfftComplex Vo;

                                uvw  = fields[f].visibilities[i].uvw[z];
                                w = fields[f].visibilities[i].weight[z];
                                Vo = fields[f].visibilities[i].Vo[z];


                                // Visibilities from metres to klambda
                                uvw.x *= fields[f].visibilities[i].freq / LIGHTSPEED;
                                uvw.y *= fields[f].visibilities[i].freq / LIGHTSPEED;
                                uvw.z *= fields[f].visibilities[i].freq / LIGHTSPEED;

                                //Apply hermitian symmetry (it will be applied afterwards)
                                if(uvw.x < 0.0) {
                                        uvw.x *= -1.0;
                                        uvw.y *= -1.0;
                                        Vo.y *= -1.0;
                                }

                                j = round(uvw.x / fabs(deltau) + N/2);
                                k = round(uvw.y / fabs(deltav) + M/2);

                                if(k < M && j < N)
                                {
                    #pragma omp critical
                                        {
                                                fields[f].gridded_visibilities[i].Vo[N*k+j].x += w * Vo.x;
                                                fields[f].gridded_visibilities[i].Vo[N*k+j].y += w * Vo.y;
                                                fields[f].gridded_visibilities[i].weight[N*k+j] += w;
                                        }
                                }
                        }

                        int visCounter = 0;
                        float gridWeightSum = 0.0f;

                        for(int k=0; k<M; k++) {
                                for (int j = 0; j < N; j++) {
                                        float weight = fields[f].gridded_visibilities[i].weight[N*k+j];
                                        if(weight > 0.0f) {
                                                gridWeightSum += weight;
                                                visCounter++;
                                        }
                                }
                        }

                        // Briggs/Robust formula
                        pow2_factor = pow(10.0, -2.0*robust);
                        w_avg = gridWeightSum/visCounter;
                        S2 = 5.0f * 5.0f * pow2_factor / w_avg;

            #pragma omp parallel for schedule(static,1)
                        for(int k=0; k<M; k++) {
                                for(int j=0; j<N; j++) {
                                        double deltau_meters = fabs(deltau) * (LIGHTSPEED/fields[f].visibilities[i].freq);
                                        double deltav_meters = fabs(deltav) * (LIGHTSPEED/fields[f].visibilities[i].freq);

                                        double u_meters = (j - (N/2)) * deltau_meters;
                                        double v_meters = (k - (M/2)) * deltav_meters;

                                        fields[f].gridded_visibilities[i].uvw[N*k+j].x = u_meters;
                                        fields[f].gridded_visibilities[i].uvw[N*k+j].y = v_meters;

                                        float weight = fields[f].gridded_visibilities[i].weight[N*k+j];
                                        if(weight > 0.0f) {
                                                fields[f].gridded_visibilities[i].Vo[N*k+j].x /= weight;
                                                fields[f].gridded_visibilities[i].Vo[N*k+j].y /= weight;
                                                fields[f].gridded_visibilities[i].weight[N*k+j] /= (1 + weight * S2);
                                        }else{
                                                fields[f].gridded_visibilities[i].weight[N*k+j] = 0.0f;
                                        }
                                }
                        }

                        fields[f].visibilities[i].uvw = (double3*)realloc(fields[f].visibilities[i].uvw, visCounter*sizeof(double3));

                        fields[f].visibilities[i].Vo = (hipfftComplex*)realloc(fields[f].visibilities[i].Vo, visCounter*sizeof(hipfftComplex));

                        fields[f].visibilities[i].Vm = (hipfftComplex*)malloc(visCounter*sizeof(hipfftComplex));
                        memset(fields[f].visibilities[i].Vm, 0, visCounter*sizeof(hipfftComplex));

                        fields[f].visibilities[i].weight = (float*)realloc(fields[f].visibilities[i].weight, visCounter*sizeof(float));

                        int l = 0;
                        for(int k=0; k<M; k++) {
                                for(int j=0; j<N; j++) {
                                        float weight = fields[f].gridded_visibilities[i].weight[N*k+j];
                                        if(weight > 0.0f) {
                                                fields[f].visibilities[i].uvw[l].x = fields[f].gridded_visibilities[i].uvw[N*k+j].x;
                                                fields[f].visibilities[i].uvw[l].y = fields[f].gridded_visibilities[i].uvw[N*k+j].y;
                                                fields[f].visibilities[i].Vo[l].x = fields[f].gridded_visibilities[i].Vo[N*k+j].x;
                                                fields[f].visibilities[i].Vo[l].y = fields[f].gridded_visibilities[i].Vo[N*k+j].y;
                                                fields[f].visibilities[i].weight[l] = fields[f].gridded_visibilities[i].weight[N*k+j];
                                                l++;
                                        }
                                }
                        }

                        free(fields[f].gridded_visibilities[i].uvw);
                        free(fields[f].gridded_visibilities[i].Vo);
                        free(fields[f].gridded_visibilities[i].weight);

                        if(fields[f].numVisibilitiesPerFreq[i] > 0) {
                                fields[f].numVisibilitiesPerFreq[i] = visCounter;
                        }
                }

                local_max = *std::max_element(fields[f].numVisibilitiesPerFreq,fields[f].numVisibilitiesPerFreq+data->total_frequencies);
                if(local_max > max) {
                        max = local_max;
                }
        }


        data->max_number_visibilities_in_channel = max;
}

__host__ float calculateNoise(Field *fields, freqData data, int *total_visibilities, int blockSizeV, int gridding)
{
        //Declaring block size and number of blocks for visibilities
        float sum_weights = 0.0;
        long UVpow2;

        for(int f=0; f<data.nfields; f++) {
                for(int i=0; i< data.total_frequencies; i++) {
                        //Calculating beam noise
                        for(int j=0; j< fields[f].numVisibilitiesPerFreq[i]; j++) {
                                if(fields[f].visibilities[i].weight[j] > 0.0) {
                                        sum_weights += fields[f].visibilities[i].weight[j];
                                }
                        }
                        *total_visibilities += fields[f].numVisibilitiesPerFreq[i];
                        fields[f].visibilities[i].numVisibilities = fields[f].numVisibilitiesPerFreq[i];
                        UVpow2 = NearestPowerOf2(fields[f].visibilities[i].numVisibilities);
                        fields[f].visibilities[i].threadsPerBlockUV = blockSizeV;
                        fields[f].visibilities[i].numBlocksUV = UVpow2/fields[f].visibilities[i].threadsPerBlockUV;
                }
        }

        float sum_inverse_weight = 1.0/sum_weights;

        if(verbose_flag) {
                float aux_noise = sqrt(sum_inverse_weight);
                printf("Calculated NOISE %e\n", aux_noise);
        }

        if(beam_noise == -1 || gridding > 0)
        {
                beam_noise = sqrt(sum_inverse_weight);
                if(verbose_flag) {
                        printf("No NOISE keyword detected in header or you might be using gridding\n");
                        printf("Using NOISE: %e ...\n", beam_noise);
                }
        }else{
                printf("Using header keyword NOISE anyway...\n");
                printf("Keyword NOISE = %e\n", beam_noise);
        }

        return beam_noise;
}


__global__ void changeGibbsEllipticalMaskAlpha(float2 *temp, float2 *theta, float *mask, float normal_I_nu_0, float normal_alpha, float bmaj, float bmin, float bpa, float factor_beam, float factor_noise, float sigma, int2 pix, float DELTAX, float DELTAY, int N)
{
        float2 nrandom;
        int j = threadIdx.x + blockDim.x * blockIdx.x;
        int i = threadIdx.y + blockDim.y * blockIdx.y;

        int c_i = pix.x;
        int c_j = pix.y;

        int idx = N*pix.x + pix.y;

        float bmaj_rad = factor_beam * bmaj * fabs(DELTAX) * RPDEG_D;
        float bmin_rad = factor_beam * bmin * fabs(DELTAY) * RPDEG_D;

        float pix_val = EllipticGaussianKernel(1.0, c_j, c_i, bmaj_rad, bmin_rad, bpa, DELTAX, DELTAY);

        nrandom.x = normal_I_nu_0 * theta[idx].x;
        nrandom.y = normal_alpha * theta[idx].y;

        if(mask[idx] <= 5.0f * sigma)
                temp[N*i+j].y += factor_noise * nrandom.y * pix_val;
        else
                temp[N*i+j].y += nrandom.y * pix_val;

        temp[N*i+j].x += nrandom.x * pix_val;
}

__global__ void changeGibbsEllipticalGaussian(float2 *temp, float2 *theta, float normal_I_nu_0, float bmaj, float bmin, float bpa, float factor, int2 pix, double DELTAX, double DELTAY, int N)
{
        float2 nrandom;
        int j = threadIdx.x + blockDim.x * blockIdx.x;
        int i = threadIdx.y + blockDim.y * blockIdx.y;

        int c_i = pix.x;
        int c_j = pix.y;

        int idx = N*pix.x + pix.y;

        float bmaj_rad = factor * bmaj * fabs(DELTAX) * RPDEG_D;
        float bmin_rad = factor * bmin * fabs(DELTAY) * RPDEG_D;

        float pix_val = EllipticGaussianKernel(1.0, c_j, c_i, bmaj_rad, bmin_rad, bpa, DELTAX, DELTAY);

        nrandom.x = normal_I_nu_0 * theta[idx].x;

        temp[N*i+j].x += nrandom.x * pix_val;

}

__global__ void changeGibbsEllipticalGaussianSpecIdx(float2 *temp, float2 *theta, float normal_I_nu_0, float normal_alpha, float bmaj, float bmin, float bpa, float factor, int2 pix, double DELTAX, double DELTAY, int N)
{
        float2 nrandom;
        int j = threadIdx.x + blockDim.x * blockIdx.x;
        int i = threadIdx.y + blockDim.y * blockIdx.y;

        int c_i = pix.x;
        int c_j = pix.y;

        int idx = N*pix.x + pix.y;

        float bmaj_rad = factor * bmaj * fabs(DELTAX) * RPDEG_D;
        float bmin_rad = factor * bmin * fabs(DELTAY) * RPDEG_D;

        float pix_val = EllipticGaussianKernel(1.0, c_j, c_i, bmaj_rad, bmin_rad, bpa, DELTAX, DELTAY);

        nrandom.x = normal_I_nu_0 * theta[idx].x;
        nrandom.y = normal_alpha * theta[idx].y;

        temp[N*i+j].x += nrandom.x * pix_val;
        temp[N*i+j].y += nrandom.y * pix_val;

}

__global__ void sumI(double2 *M_k, double2 *Q_k, float2 *I, int k, long N)
{
        int j = threadIdx.x + blockDim.x * blockIdx.x;
        int i = threadIdx.y + blockDim.y * blockIdx.y;

        double dif_I_nu_0 = I[N*i+j].x - M_k[N*i+j].x;
        double dif_alpha = I[N*i+j].y - M_k[N*i+j].y;

        double dif_I_nu_02 = dif_I_nu_0 * dif_I_nu_0;
        double dif_alpha2 = dif_alpha * dif_alpha;

        M_k[N*i+j].x = M_k[N*i+j].x + (dif_I_nu_0/k);
        M_k[N*i+j].y = M_k[N*i+j].y + (dif_alpha/k);

        Q_k[N*i+j].x = Q_k[N*i+j].x + ((k-1) * dif_I_nu_02 / k);
        Q_k[N*i+j].y = Q_k[N*i+j].y + ((k-1) * dif_alpha2 / k);
}

__global__ void floatToDoubleKernel(double2 *a, float2 *b, long N)
{
        int j = threadIdx.x + blockDim.x * blockIdx.x;
        int i = threadIdx.y + blockDim.y * blockIdx.y;

        a[N*i+j].x = b[N*i+j].x;
        a[N*i+j].y = b[N*i+j].y;
}

__global__ void updateTheta(float2 *theta, double2 *Q_k, float s_d, int samples, long N)
{
        int j = threadIdx.x + blockDim.x * blockIdx.x;
        int i = threadIdx.y + blockDim.y * blockIdx.y;

        double2 cov;

        cov.x = Q_k[N*i+j].x/(samples-1);
        cov.y = Q_k[N*i+j].y/(samples-1);

        theta[N*i+j].x = sqrt(s_d * cov.x + s_d * 1E-8);
        theta[N*i+j].y = sqrt(s_d * cov.y + s_d * 1E-8);
}


__host__ float chiCuadrado(float2 *I)
{
        if(num_gpus == 1) {
                hipSetDevice(selected);
        }else{
                hipSetDevice(firstgpu);
        }

        float resultPhi = 0.0;
        float resultchi2  = 0.0;
        float resultS  = 0.0;
        float resultS_alpha = 0.0;

        //if(clip_flag){
        clip2I<<<numBlocksNN, threadsPerBlockNN>>>(I, N, MINPIX, fg_scale, eta);
        gpuErrchk(hipDeviceSynchronize());
        //}

        clip2IWNoise<<<numBlocksNN, threadsPerBlockNN>>>(device_noise_image, I, N, noise_cut, MINPIX, alpha_start, fg_scale, eta);
        gpuErrchk(hipDeviceSynchronize());

        if(epsilon) {
                LVectorAlpha<<<numBlocksNN, threadsPerBlockNN>>>(device_S_alpha, device_noise_image, I, N, noise_cut);
                gpuErrchk(hipDeviceSynchronize());
        }

        if(iter>0 && lambda!=0.0) {
                switch(reg_term) {
                case 0:
                        SVector<<<numBlocksNN, threadsPerBlockNN>>>(device_S, device_noise_image, I, N, noise_cut, MINPIX, fg_scale, eta);
                        gpuErrchk(hipDeviceSynchronize());
                        break;
                case 1:
                        QPVector<<<numBlocksNN, threadsPerBlockNN>>>(device_S, device_noise_image, I, N, noise_cut);
                        gpuErrchk(hipDeviceSynchronize());
                        break;
                case 2:
                        TVVector<<<numBlocksNN, threadsPerBlockNN>>>(device_S, device_noise_image, I, N, noise_cut);
                        gpuErrchk(hipDeviceSynchronize());
                        break;
                default:
                        printf("Selected prior is not defined\n");
                        goToError();
                        break;
                }
        }


        if(num_gpus == 1) {
                hipSetDevice(selected);
                for(int f=0; f<data.nfields; f++) {
                        for(int i=0; i<data.total_frequencies; i++) {

                                if(fields[f].numVisibilitiesPerFreq[i] != 0) {

                                        calculateInu<<<numBlocksNN, threadsPerBlockNN>>>(device_Inu, I, fields[f].visibilities[i].freq, nu_0, fg_scale, MINPIX, eta, N);
                                        gpuErrchk(hipDeviceSynchronize());

                                        /*if(clip_flag){
                                           clip<<<numBlocksNN, threadsPerBlockNN>>>(device_Inu, N, MINPIX);
                                           gpuErrchk(hipDeviceSynchronize());
                                           }*/

                                        apply_beam<<<numBlocksNN, threadsPerBlockNN>>>(antenna_diameter, pb_factor, pb_cutoff, device_Inu, N, fields[f].ref_xobs, fields[f].ref_yobs, fg_scale, fields[f].visibilities[i].freq, DELTAX, DELTAY);
                                        gpuErrchk(hipDeviceSynchronize());

                                        //FFT 2D
                                        if ((hipfftExecC2C(plan1GPU, (hipfftComplex*)device_Inu, (hipfftComplex*)device_V, HIPFFT_FORWARD)) != HIPFFT_SUCCESS) {
                                                printf("CUFFT exec error\n");
                                                goToError();
                                        }
                                        gpuErrchk(hipDeviceSynchronize());

                                        //PHASE_ROTATE
                                        phase_rotate<<<numBlocksNN, threadsPerBlockNN>>>(device_V, M, N, fields[f].phs_xobs, fields[f].phs_yobs);
                                        gpuErrchk(hipDeviceSynchronize());

                                        //RESIDUAL CALCULATION
                                        vis_mod<<<fields[f].visibilities[i].numBlocksUV, fields[f].visibilities[i].threadsPerBlockUV>>>(fields[f].device_visibilities[i].Vm, device_V, fields[f].device_visibilities[i].uvw, fields[f].device_visibilities[i].weight, deltau, deltav, fields[f].numVisibilitiesPerFreq[i], N);
                                        gpuErrchk(hipDeviceSynchronize());

                                        residual<<<fields[f].visibilities[i].numBlocksUV, fields[f].visibilities[i].threadsPerBlockUV>>>(fields[f].device_visibilities[i].Vr, fields[f].device_visibilities[i].Vm, fields[f].device_visibilities[i].Vo, fields[f].numVisibilitiesPerFreq[i]);
                                        gpuErrchk(hipDeviceSynchronize());


                                        ////chi 2 VECTOR
                                        chi2Vector<<<fields[f].visibilities[i].numBlocksUV, fields[f].visibilities[i].threadsPerBlockUV>>>(device_chi2, fields[f].device_visibilities[i].Vr, fields[f].device_visibilities[i].weight, fields[f].numVisibilitiesPerFreq[i]);
                                        gpuErrchk(hipDeviceSynchronize());

                                        //REDUCTIONS
                                        //chi2
                                        resultchi2  += deviceReduce(device_chi2, fields[f].numVisibilitiesPerFreq[i]);
                                }
                        }
                }
        }else{
                for(int f=0; f<data.nfields; f++) {
                        #pragma omp parallel for schedule(static,1)
                        for (int i = 0; i < data.total_frequencies; i++)
                        {
                                float partial_chi2 = 0.0;

                                unsigned int j = omp_get_thread_num();
                                //unsigned int num_cpu_threads = omp_get_num_threads();
                                // set and check the CUDA device for this CPU thread
                                int gpu_id = -1;
                                hipSetDevice((i%num_gpus) + firstgpu); // "% num_gpus" allows more CPU threads than GPU devices
                                hipGetDevice(&gpu_id);
                                if(fields[f].numVisibilitiesPerFreq[i] != 0) {

                                        gpuErrchk(hipMemset(vars_gpu[i%num_gpus].device_chi2, 0, sizeof(float)*data.max_number_visibilities_in_channel));

                                        calculateInu<<<numBlocksNN, threadsPerBlockNN>>>(vars_gpu[i%num_gpus].device_Inu, I, fields[f].visibilities[i].freq, nu_0, fg_scale, MINPIX, eta, N);
                                        gpuErrchk(hipDeviceSynchronize());

                                        /*if(clip_flag){
                                           clip<<<numBlocksNN, threadsPerBlockNN>>>(vars_gpu[i%num_gpus].device_Inu, N, MINPIX);
                                           gpuErrchk(hipDeviceSynchronize());
                                           }*/

                                        apply_beam<<<numBlocksNN, threadsPerBlockNN>>>(antenna_diameter, pb_factor, pb_cutoff, vars_gpu[i%num_gpus].device_Inu, N, fields[f].ref_xobs, fields[f].ref_yobs, fg_scale, fields[f].visibilities[i].freq, DELTAX, DELTAY);
                                        gpuErrchk(hipDeviceSynchronize());

                                        //FFT 2D
                                        if ((hipfftExecC2C(vars_gpu[i%num_gpus].plan, (hipfftComplex*)vars_gpu[i%num_gpus].device_Inu, (hipfftComplex*)vars_gpu[i%num_gpus].device_V, HIPFFT_FORWARD)) != HIPFFT_SUCCESS) {
                                                printf("CUFFT exec error\n");
                                                //return -1 ;
                                                goToError();
                                        }
                                        gpuErrchk(hipDeviceSynchronize());

                                        //PHASE_ROTATE
                                        phase_rotate<<<numBlocksNN, threadsPerBlockNN>>>(vars_gpu[i%num_gpus].device_V, M, N, fields[f].phs_xobs, fields[f].phs_yobs);
                                        gpuErrchk(hipDeviceSynchronize());

                                        //RESIDUAL CALCULATION
                                        vis_mod<<<fields[f].visibilities[i].numBlocksUV, fields[f].visibilities[i].threadsPerBlockUV>>>(fields[f].device_visibilities[i].Vm, vars_gpu[i%num_gpus].device_V, fields[f].device_visibilities[i].uvw, fields[f].device_visibilities[i].weight, deltau, deltav, fields[f].numVisibilitiesPerFreq[i], N);
                                        gpuErrchk(hipDeviceSynchronize());


                                        residual<<<fields[f].visibilities[i].numBlocksUV, fields[f].visibilities[i].threadsPerBlockUV>>>(fields[f].device_visibilities[i].Vr, fields[f].device_visibilities[i].Vm, fields[f].device_visibilities[i].Vo, fields[f].numVisibilitiesPerFreq[i]);
                                        gpuErrchk(hipDeviceSynchronize());

                                        ////chi2 VECTOR
                                        chi2Vector<<<fields[f].visibilities[i].numBlocksUV, fields[f].visibilities[i].threadsPerBlockUV>>>(vars_gpu[i%num_gpus].device_chi2, fields[f].device_visibilities[i].Vr, fields[f].device_visibilities[i].weight, fields[f].numVisibilitiesPerFreq[i]);
                                        gpuErrchk(hipDeviceSynchronize());


                                        partial_chi2 = deviceReduce(vars_gpu[i%num_gpus].device_chi2, fields[f].numVisibilitiesPerFreq[i]);

                                        //REDUCTIONS
                                        //chi2
                                        #pragma omp critical
                                        {
                                                resultchi2  += partial_chi2;
                                        }
                                }
                        }
                }
        }

        if(num_gpus == 1) {
                hipSetDevice(selected);
        }else{
                hipSetDevice(firstgpu);
        }

        resultS = deviceReduce(device_S, M*N);

        if(epsilon) {
                resultS_alpha = deviceReduce(device_S_alpha, M*N);
        }

        resultPhi = (0.5 * resultchi2) + (lambda * resultS) + (epsilon * resultS_alpha);

        final_chi2 = resultchi2;
        final_H = resultS;
        /*printf("chi2 value = %.5f\n", resultchi2);
           printf("H value = %.5f\n", resultH);
           printf("(1/2) * chi2 value = %.5f\n", 0.5*resultchi2);
           printf("lambda * H value = %.5f\n", lambda*resultH);
           printf("Phi value = %.5f\n\n", resultPhi);*/

        return resultPhi;
}


__host__ void MetropolisHasting(float2 *I, float2 *theta, int iterations, int *burndown_steps, int *accepted)
{
        PlantSeeds(-1);
        if(num_gpus == 1) {
                hipSetDevice(selected);
        }else{
                hipSetDevice(firstgpu);
        }
        hiprandState_t *states_0;
        hiprandState_t *states_1;
        //*states2;
        hipMalloc((void**)&states_0, M*N*sizeof(hiprandState_t));
        hipMalloc((void**)&states_1, M*N*sizeof(hiprandState_t));
        //hipMalloc((void**)&states2, N*N*sizeof(hiprandState_t));
        float chi2_t_0, chi2_t_1;
        float delta_chi2 = 0.0f;
        float p = 0.0f;
        float un_rand = 0.0f;
        float s_d = (2.4*2.4)/valid_pixels;
        accepted_afterburndown = *accepted;

        /**************** GPU MEMORY FOR TOTAL OUT I_nu_0 and alpha ***************/

        gpuErrchk(hipMalloc((void**)&M_k_out, sizeof(double2)*M*N));
        gpuErrchk(hipMalloc((void**)&Q_k_out, sizeof(double2)*M*N));

        if(checkpoint) {
                gpuErrchk(hipMemcpy(Q_k_out, host_Q_k, sizeof(double2)*M*N, hipMemcpyHostToDevice));
                gpuErrchk(hipMemcpy(M_k_out, host_M_k, sizeof(double2)*M*N, hipMemcpyHostToDevice));
        }else{
                gpuErrchk(hipMemset(Q_k_out, 0, sizeof(double2)*M*N));
                floatToDoubleKernel<<<numBlocksNN, threadsPerBlockNN>>>(M_k_out, I, N);
                gpuErrchk(hipDeviceSynchronize());
        }


        /**************** GPU MEMORY FOR TEMPORAL I_nu_0 and alpha ***************/
        gpuErrchk(hipMalloc((void**)&temp, sizeof(float2)*M*N));
        gpuErrchk(hipMemset(temp, 0, sizeof(float2)*M*N));


        random_init<<<numBlocksNN, threadsPerBlockNN>>>(time(0), states_0, N);
        gpuErrchk(hipDeviceSynchronize());
        random_init<<<numBlocksNN, threadsPerBlockNN>>>(time(0), states_1, N);
        gpuErrchk(hipDeviceSynchronize());
        //random_init<<<numBlocksNN, threadsPerBlockNN>>>(time(0), states2, N);
        //gpuErrchk(hipDeviceSynchronize());
        size_t file_chi2_n = snprintf(NULL, 0, "%schi2.txt", checkp) + 1;
        size_t file_iter_n = snprintf(NULL, 0, "%siter.txt", checkp) + 1;
        size_t file_shutdown_n = snprintf(NULL, 0, "%sshutdown.txt", checkp) + 1;
        char *chi2_file_name = (char*)malloc(sizeof(char)*file_chi2_n);
        char *iter_file_name = (char*)malloc(sizeof(char)*file_iter_n);
        shutdown_file_name = (char*)malloc(sizeof(char)*file_shutdown_n);

        snprintf(chi2_file_name, file_chi2_n*sizeof(char), "%schi2.txt", checkp, 0);
        snprintf(iter_file_name, file_iter_n*sizeof(char), "%siter.txt", checkp, 0);
        snprintf(shutdown_file_name, file_shutdown_n*sizeof(char), "%sshutdown.txt", checkp, 0);

        if(checkpoint)
                outfile = fopen(chi2_file_name, "a");
        else
                outfile = fopen(chi2_file_name, "w");

        outfile_its = fopen(iter_file_name, "w");

        position_in_file = ftell(outfile_its);
        randomize(pixels, valid_pixels);

        float n_I_nu_0;
        float n_alpha;

        for(real_iterations = 1; real_iterations <= iterations; real_iterations++) {

                if(adaptive && real_iterations >= *burndown_steps + 100) {
                        //__global__ void updateTheta(float2 *theta, double2 *total, double2 *total2, float s_d, int samples, long N)
                        updateTheta<<<numBlocksNN, threadsPerBlockNN>>>(theta, Q_k_out, s_d, accepted_afterburndown, N);
                        gpuErrchk(hipDeviceSynchronize());
                }

                if(real_iterations == *burndown_steps) {
                        if(checkpoint) {
                                gpuErrchk(hipMemcpy(M_k_out, host_M_k, sizeof(double2)*M*N, hipMemcpyHostToDevice));
                        }
                        else
                        {
                                floatToDoubleKernel<<<numBlocksNN, threadsPerBlockNN>>>(M_k_out, I, N);
                                gpuErrchk(hipDeviceSynchronize());
                        }
                }

                for(int j = 0; j < valid_pixels; j++) {

                        gpuErrchk(hipMemcpy(temp, I, M*N*sizeof(float2), hipMemcpyDeviceToDevice));

                        if(use_mask) {
                                SelectStream(0);
                                n_I_nu_0 = Normal(0.0, 1.0);
                                SelectStream(1);
                                n_alpha = Normal(0.0, 1.0);
                                changeGibbsEllipticalMaskAlpha<<<numBlocksNN, threadsPerBlockNN>>>(temp, theta, device_mask, n_I_nu_0, n_alpha, beam_bmaj, beam_bmin, beam_bpa, (1.0/3.0), 10.0, noise_jypix, pixels[j], DELTAX, DELTAY, N);
                                gpuErrchk(hipDeviceSynchronize());
                        }else{
                                //changeGibbs<<<1, 1>>>(temp, theta, states, pixels[j], N);
                                if(spec_idx)
                                {
                                        SelectStream(0);
                                        n_I_nu_0 = Normal(0.0, 1.0);
                                        SelectStream(1);
                                        n_alpha = Normal(0.0, 1.0);
                                        changeGibbsEllipticalGaussianSpecIdx<<<numBlocksNN, threadsPerBlockNN>>>(temp, theta, n_I_nu_0, n_alpha, beam_bmaj, beam_bmin, beam_bpa, (1.0/3.0), pixels[j], DELTAX, DELTAY, N);
                                        gpuErrchk(hipDeviceSynchronize());
                                }else{
                                        SelectStream(0);
                                        n_I_nu_0 = Normal(0.0, 1.0);
                                        changeGibbsEllipticalGaussian<<<numBlocksNN, threadsPerBlockNN>>>(temp, theta, n_I_nu_0, beam_bmaj, beam_bmin, beam_bpa, (1.0/3.0), pixels[j], DELTAX, DELTAY, N);
                                        gpuErrchk(hipDeviceSynchronize());
                                }

                        }

                        chi2_t_0 = chiCuadrado(I);
                        chi2_t_1 = chiCuadrado(temp);
                        //printf("chi2_t0: %f\n", chi2_t_0);
                        //printf("chi2_t1: %f\n", chi2_t_1);
                        fprintf(outfile, "%f\n", chi2_t_0);
                        delta_chi2 = chi2_t_1 - chi2_t_0;
                        if(delta_chi2 <= 0) {
                                //printf("Acccepted Delta chi2: %f\n", delta_chi2);
                                gpuErrchk(hipMemcpy(I, temp, sizeof(float2)*M*N, hipMemcpyDeviceToDevice));
                                /*if(print_images && i%3 == 0)
                                   float2toImage(I, mod_in, out_image, mempath, i, M, N, 1);*/
                                if(real_iterations >= *burndown_steps) {
                                        accepted_afterburndown++;
                                        sumI<<<numBlocksNN, threadsPerBlockNN>>>(M_k_out, Q_k_out, I, accepted_afterburndown, N);
                                        gpuErrchk(hipDeviceSynchronize());
                                }

                        }
                        else{
                                //printf("Not Accepted Delta chi2: %f\n", delta_chi2);
                                SelectStream(2);
                                un_rand = Random();
                                if(-log(un_rand) > delta_chi2) {
                                        gpuErrchk(hipMemcpy(I, temp, sizeof(float2)*M*N, hipMemcpyDeviceToDevice));
                                        if(real_iterations >= *burndown_steps) {
                                                accepted_afterburndown++;
                                                sumI<<<numBlocksNN, threadsPerBlockNN>>>(M_k_out, Q_k_out, I, accepted_afterburndown, N);
                                                gpuErrchk(hipDeviceSynchronize());
                                        }
                                }
                        }
                }

                printf("--------------Iteration %d-----------\n", real_iterations);
                printf("From %d valid pixels, accepted :%d\n", valid_pixels, accepted_afterburndown);
                fseek(outfile_its,position_in_file,SEEK_SET);
                fprintf(outfile_its, "Iterations: %d\n", real_iterations);
                fprintf(outfile_its, "Accepted after burndown: %d\n", accepted_afterburndown);
                fflush(outfile_its);
                double2toImage(M_k_out, mod_in, out_image, checkp, 0, M, N, 1.0, accepted_afterburndown, 1);
                double2toImage(Q_k_out, mod_in, out_image, checkp, 1, M, N, 1.0/(accepted_afterburndown-1), accepted_afterburndown, 1);
                float2toImage(I, mod_in, out_image, checkp, 2, M, N, 1.0, 1);
                randomize(pixels, valid_pixels);
        }

        printf("ACCEPTED AFTER BURNDOWN: %d\n", accepted_afterburndown);
        fprintf(outfile_its, "%d\n", accepted_afterburndown);

        double2toImage(M_k_out, mod_in, out_image, checkp, 0, M, N, 1.0, accepted_afterburndown, 1);
        double2toImage(Q_k_out, mod_in, out_image, checkp, 1, M, N, 1.0/(accepted_afterburndown-1), accepted_afterburndown, 1);
        float2toImage(I, mod_in, out_image, checkp, 2, M, N, 1.0, 1);

        signal(SIGINT, sig_handler);
        signal(SIGTERM, sig_handler);
        signal(SIGKILL, sig_handler);

        fclose(outfile);
        fclose(outfile_its);
        hipFree(temp);
        hipFree(states_0);
        hipFree(states_1);

}


__host__ void Metropolis(float2 *I, float2 *theta, int iterations, int burndown_steps, int accepted)
{
        PlantSeeds(-1);
        if(num_gpus == 1) {
                hipSetDevice(selected);
        }else{
                hipSetDevice(firstgpu);
        }
        hiprandState_t *states_0;
        hiprandState_t *states_1;
        //*states2;
        hipMalloc((void**)&states_0, M*N*sizeof(hiprandState_t));
        hipMalloc((void**)&states_1, M*N*sizeof(hiprandState_t));
        //hipMalloc((void**)&states2, N*N*sizeof(hiprandState_t));
        float chi2_t_0, chi2_t_1;
        float delta_chi2 = 0.0f;
        float p = 0.0f;
        float un_rand = 0.0f;
        float s_d = (2.4*2.4)/valid_pixels;
        accepted_afterburndown = accepted;


        /**************** GPU MEMORY FOR TOTAL OUT I_nu_0 and alpha ***************/

        gpuErrchk(hipMalloc((void**)&M_k_out, sizeof(double2)*M*N));
        gpuErrchk(hipMalloc((void**)&Q_k_out, sizeof(double2)*M*N));

        if(checkpoint) {
                gpuErrchk(hipMemcpy(Q_k_out, host_Q_k, sizeof(double2)*M*N, hipMemcpyHostToDevice));
                gpuErrchk(hipMemcpy(M_k_out, host_M_k, sizeof(double2)*M*N, hipMemcpyHostToDevice));
        }else{
                gpuErrchk(hipMemset(Q_k_out, 0, sizeof(double2)*M*N));
                floatToDoubleKernel<<<numBlocksNN, threadsPerBlockNN>>>(M_k_out, I, N);
                gpuErrchk(hipDeviceSynchronize());
        }


        /**************** GPU MEMORY FOR TEMPORAL I_nu_0 and alpha ***************/
        gpuErrchk(hipMalloc((void**)&temp, sizeof(float2)*M*N));
        gpuErrchk(hipMemset(temp, 0, sizeof(float2)*M*N));

        random_init<<<numBlocksNN, threadsPerBlockNN>>>(time(0), states_0, N);
        gpuErrchk(hipDeviceSynchronize());
        random_init<<<numBlocksNN, threadsPerBlockNN>>>(time(0), states_1, N);
        gpuErrchk(hipDeviceSynchronize());
        //random_init<<<numBlocksNN, threadsPerBlockNN>>>(time(0), states2, N);
        //gpuErrchk(hipDeviceSynchronize());
        size_t file_chi2_n = snprintf(NULL, 0, "%schi2.txt", checkp) + 1;
        size_t file_iter_n = snprintf(NULL, 0, "%siter.txt", checkp) + 1;
        size_t file_shutdown_n = snprintf(NULL, 0, "%sshutdown.txt", checkp) + 1;
        char *chi2_file_name = (char*)malloc(sizeof(char)*file_chi2_n);
        char *iter_file_name = (char*)malloc(sizeof(char)*file_iter_n);
        shutdown_file_name = (char*)malloc(sizeof(char)*file_shutdown_n);

        snprintf(chi2_file_name, file_chi2_n*sizeof(char), "%schi2.txt", checkp, 0);
        snprintf(iter_file_name, file_iter_n*sizeof(char), "%siter.txt", checkp, 0);
        snprintf(shutdown_file_name, file_shutdown_n*sizeof(char), "%sshutdown.txt", checkp, 0);

        if(checkpoint)
                outfile = fopen(chi2_file_name, "a");
        else
                outfile = fopen(chi2_file_name, "w");

        outfile_its = fopen(iter_file_name, "w");

        position_in_file = ftell(outfile_its);
        randomize(pixels, valid_pixels);

        float n_I_nu_0;
        float n_alpha;
        for(real_iterations = 1; real_iterations<= iterations; real_iterations++) {
                if(adaptive && real_iterations >= burndown_steps + 100) {
                        //__global__ void updateTheta(float2 *theta, double2 *total, double2 *total2, float s_d, int samples, long N)
                        updateTheta<<<numBlocksNN, threadsPerBlockNN>>>(theta, Q_k_out, s_d, accepted_afterburndown, N);
                        gpuErrchk(hipDeviceSynchronize());
                }

                if(real_iterations == burndown_steps) {
                        if(checkpoint) {
                                gpuErrchk(hipMemcpy(M_k_out, host_M_k, sizeof(double2)*M*N, hipMemcpyHostToDevice));
                        }
                        else
                        {
                                floatToDoubleKernel<<<numBlocksNN, threadsPerBlockNN>>>(M_k_out, I, N);
                                gpuErrchk(hipDeviceSynchronize());
                        }
                }

                for(int j = 0; j < valid_pixels; j++) {

                        //if(i>0){
                        //  calculateTheta<<<numBlocksNN, threadsPerBlockNN>>>(theta, states2, total, total2, accepted+1, N);
                        //  gpuErrchk(hipDeviceSynchronize());
                        //}
                        gpuErrchk(hipMemcpy(temp, I, M*N*sizeof(float2), hipMemcpyDeviceToDevice));

                        if(use_mask) {
                                SelectStream(0);
                                n_I_nu_0 = Normal(0.0, 1.0);
                                SelectStream(1);
                                n_alpha = Normal(0.0, 1.0);
                                changeGibbsEllipticalMaskAlpha<<<numBlocksNN, threadsPerBlockNN>>>(temp, theta, device_mask, n_I_nu_0, n_alpha, beam_bmaj, beam_bmin, beam_bpa, (1.0/3.0), 10.0, noise_jypix, pixels[j], DELTAX, DELTAY, N);
                                gpuErrchk(hipDeviceSynchronize());
                        }else{
                                //changeGibbs<<<1, 1>>>(temp, theta, states, pixels[j], N);
                                if(spec_idx)
                                {
                                        SelectStream(0);
                                        n_I_nu_0 = Normal(0.0, 1.0);
                                        SelectStream(1);
                                        n_alpha = Normal(0.0, 1.0);
                                        changeGibbsEllipticalGaussianSpecIdx<<<numBlocksNN, threadsPerBlockNN>>>(temp, theta, n_I_nu_0, n_alpha, beam_bmaj, beam_bmin, beam_bpa, (1.0/3.0), pixels[j], DELTAX, DELTAY, N);
                                        gpuErrchk(hipDeviceSynchronize());
                                }else{
                                        SelectStream(0);
                                        n_I_nu_0 = Normal(0.0, 1.0);
                                        changeGibbsEllipticalGaussian<<<numBlocksNN, threadsPerBlockNN>>>(temp, theta, n_I_nu_0, beam_bmaj, beam_bmin, beam_bpa, (1.0/3.0), pixels[j], DELTAX, DELTAY, N);
                                        gpuErrchk(hipDeviceSynchronize());
                                }

                        }


                        chi2_t_0 = chiCuadrado(I);
                        chi2_t_1 = chiCuadrado(temp);
                        //printf("chi2_t0: %f\n", chi2_t_0);
                        //printf("chi2_t1: %f\n", chi2_t_1);
                        fprintf(outfile, "%f\n", chi2_t_0);
                        fflush(outfile);
                        delta_chi2 = chi2_t_1 - chi2_t_0;
                        if(delta_chi2 <= 0.0f) {
                                //printf("Accepted Delta chi2: %f\n", delta_chi2);
                                gpuErrchk(hipMemcpy(I, temp, sizeof(float2)*M*N, hipMemcpyDeviceToDevice));
                                /*if(print_images && i%3 == 0)
                                   float2toImage(I, mod_in, out_image, mempath, i, M, N, 1);*/
                                if(real_iterations >= burndown_steps) {
                                        accepted_afterburndown++;
                                        sumI<<<numBlocksNN, threadsPerBlockNN>>>(M_k_out, Q_k_out, I, accepted_afterburndown, N);
                                        gpuErrchk(hipDeviceSynchronize());
                                }

                        }
                        /*else{
                                //printf("Not Accepted Delta chi2: %f\n", delta_chi2);
                                //l = exp(-delta_chi2);
                                //p = chi2_t_0 / chi2_t_1;
                                un_rand = Random();
                                if(-log(un_rand) > delta_chi2) {
                                        //printf("*****  P = %f > %f   *******\n", -log(un_rand), delta_chi2);
                                        gpuErrchk(hipMemcpy2D(I, sizeof(float2), temp, sizeof(float2), sizeof(float2), M*N, hipMemcpyDeviceToDevice));
                                        if(print_images && i%3 == 0)
                                           float2toImage(I, mod_in, out_image, mempath, i, M, N, 1);
                                        if(real_iterations >= burndown_steps) {
                                                sumI<<<numBlocksNN, threadsPerBlockNN>>>(total_out, total2_out, I, N);
                                                gpuErrchk(hipDeviceSynchronize());
                                                accepted_afterburndown++;
                                        }
                                        second_pass++;

                                }
                           }*/
                }

                printf("--------------Iteration %d-----------\n", real_iterations);
                printf("From %d valid pixels, accepted :%d\n", valid_pixels, accepted_afterburndown);
                fseek(outfile_its,position_in_file,SEEK_SET);
                fprintf(outfile_its, "Iterations: %d\n", real_iterations);
                fprintf(outfile_its, "Accepted after burndown: %d\n", accepted_afterburndown);
                fflush(outfile_its);
                double2toImage(M_k_out, mod_in, out_image, checkp, 0, M, N, 1.0, accepted_afterburndown, 1);
                double2toImage(Q_k_out, mod_in, out_image, checkp, 1, M, N, 1.0/(accepted_afterburndown-1), accepted_afterburndown, 1);
                float2toImage(I, mod_in, out_image, checkp, 2, M, N, 1.0, 1);
                randomize(pixels, valid_pixels);
        }

        //avgI<<<numBlocksNN, threadsPerBlockNN>>>(total_out, total2_out, accepted_afterburndown, N);
        //gpuErrchk(hipDeviceSynchronize());
        printf("ACCEPTED AFTER BURNDOWN: %d\n", accepted_afterburndown);
        fprintf(outfile_its, "%d\n", accepted_afterburndown);

        double2toImage(M_k_out, mod_in, out_image, checkp, 0, M, N, 1.0, accepted_afterburndown, 1);
        double2toImage(Q_k_out, mod_in, out_image, checkp, 1, M, N, 1.0/(accepted_afterburndown-1), accepted_afterburndown, 1);
        float2toImage(I, mod_in, out_image, checkp, 2, M, N, 1.0, 1);

        signal(SIGINT, sig_handler);
        signal(SIGTERM, sig_handler);
        signal(SIGKILL, sig_handler);

        fclose(outfile);
        fclose(outfile_its);
        hipFree(temp);
        hipFree(states_0);
        hipFree(states_1);

}
